/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025, NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../linear_programming/utilities/pdlp_test_utilities.cuh"
#include "mip_utils.cuh"

#include <cuopt/linear_programming/solve.hpp>
#include <mps_parser/parser.hpp>
#include <utilities/common_utils.hpp>
#include <utilities/error.hpp>

#include <raft/core/handle.hpp>
#include <raft/util/cudart_utils.hpp>

#include <gtest/gtest.h>

#include <cstdint>
#include <sstream>
#include <string>
#include <vector>

namespace cuopt::linear_programming::test {

struct result_map_t {
  std::string file;
  double cost;
};

void test_miplib_file(result_map_t test_instance)
{
  const raft::handle_t handle_{};

  auto path = make_path_absolute(test_instance.file);
  cuopt::mps_parser::mps_data_model_t<int, double> problem =
    cuopt::mps_parser::parse_mps<int, double>(path, false);
  handle_.sync_stream();
  mip_solver_settings_t<int, double> settings;
  // set the time limit depending on we are in assert mode or not
#ifdef ASSERT_MODE
  constexpr double test_time_limit = 60.;
#else
  constexpr double test_time_limit = 30.;
#endif

  settings.time_limit                  = test_time_limit;
  mip_solution_t<int, double> solution = solve_mip(&handle_, problem, settings);
  EXPECT_EQ(solution.get_termination_status(), mip_termination_status_t::FeasibleFound);
  double obj_val = solution.get_objective_value();
  // for now keep a 100% error rate
  EXPECT_NEAR(test_instance.cost, obj_val, test_instance.cost);
  test_variable_bounds(problem, solution.get_solution(), settings);
  // TODO test integrality as well
}

TEST(mip_solve, run_small_tests)
{
  std::vector<result_map_t> test_instances = {
    {"mip/50v-10.mps", 11311031.}, {"mip/neos5.mps", 15.}, {"mip/swath1.mps", 1300.}};
  for (const auto& test_instance : test_instances) {
    test_miplib_file(test_instance);
  }
}

}  // namespace cuopt::linear_programming::test
