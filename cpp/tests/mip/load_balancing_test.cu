#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../linear_programming/utilities/pdlp_test_utilities.cuh"
#include "mip_utils.cuh"

#include <raft/sparse/detail/cusparse_wrappers.h>
#include <linear_programming/initial_scaling_strategy/initial_scaling.cuh>
#include <linear_programming/utilities/problem_checking.cuh>
#include <mip/presolve/bounds_presolve.cuh>
#include <mip/presolve/load_balanced_bounds_presolve.cuh>
#include <mip/problem/load_balanced_problem.cuh>
#include <mps_parser/parser.hpp>
#include <raft/core/handle.hpp>
#include <raft/util/cudart_utils.hpp>
#include <utilities/common_utils.hpp>
#include <utilities/error.hpp>
#include <utilities/timer.hpp>

#include <rmm/mr/device/cuda_async_memory_resource.hpp>

#include <gtest/gtest.h>

#include <cstdint>
#include <sstream>
#include <string>
#include <vector>

namespace cuopt::linear_programming::test {

inline auto make_async() { return std::make_shared<rmm::mr::cuda_async_memory_resource>(); }

void init_handler(const raft::handle_t* handle_ptr)
{
  // Init cuBlas / cuSparse context here to avoid having it during solving time
  RAFT_CUBLAS_TRY(raft::linalg::detail::cublassetpointermode(
    handle_ptr->get_cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE, handle_ptr->get_stream()));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsesetpointermode(
    handle_ptr->get_cusparse_handle(), HIPSPARSE_POINTER_MODE_DEVICE, handle_ptr->get_stream()));
}

std::tuple<std::vector<int>, std::vector<double>, std::vector<double>> select_k_random(
  detail::problem_t<int, double>& problem, int sample_size)
{
  auto seed = std::random_device{}();
  std::cerr << "Tested with seed " << seed << "\n";
  problem.compute_n_integer_vars();
  auto v_lb       = host_copy(problem.variable_lower_bounds);
  auto v_ub       = host_copy(problem.variable_upper_bounds);
  auto int_var_id = host_copy(problem.integer_indices);
  int_var_id.erase(std::remove_if(int_var_id.begin(),
                                  int_var_id.end(),
                                  [v_lb, v_ub](auto id) {
                                    return !(std::isfinite(v_lb[id]) && std::isfinite(v_ub[id]));
                                  }),
                   int_var_id.end());
  sample_size = std::min(sample_size, static_cast<int>(int_var_id.size()));
  std::vector<int> random_int_vars;
  std::mt19937 m{seed};
  std::sample(
    int_var_id.begin(), int_var_id.end(), std::back_inserter(random_int_vars), sample_size, m);
  std::vector<double> probe_0(sample_size);
  std::vector<double> probe_1(sample_size);
  for (int i = 0; i < static_cast<int>(random_int_vars.size()); ++i) {
    if (i % 2) {
      probe_0[i] = v_lb[random_int_vars[i]];
      probe_1[i] = v_ub[random_int_vars[i]];
    } else {
      probe_1[i] = v_lb[random_int_vars[i]];
      probe_0[i] = v_ub[random_int_vars[i]];
    }
  }
  return std::make_tuple(std::move(random_int_vars), std::move(probe_0), std::move(probe_1));
}

std::pair<std::vector<thrust::pair<int, double>>, std::vector<thrust::pair<int, double>>>
convert_probe_tuple(std::tuple<std::vector<int>, std::vector<double>, std::vector<double>>& probe)
{
  std::vector<thrust::pair<int, double>> probe_first;
  std::vector<thrust::pair<int, double>> probe_second;
  for (size_t i = 0; i < std::get<0>(probe).size(); ++i) {
    probe_first.emplace_back(thrust::make_pair(std::get<0>(probe)[i], std::get<1>(probe)[i]));
    probe_second.emplace_back(thrust::make_pair(std::get<0>(probe)[i], std::get<2>(probe)[i]));
  }
  return std::make_pair(std::move(probe_first), std::move(probe_second));
}

std::tuple<std::vector<double>, std::vector<double>, std::vector<double>, std::vector<double>>
bounds_probe_results(detail::bound_presolve_t<int, double>& bnd_prb_0,
                     detail::bound_presolve_t<int, double>& bnd_prb_1,
                     detail::problem_t<int, double>& problem,
                     const std::pair<std::vector<thrust::pair<int, double>>,
                                     std::vector<thrust::pair<int, double>>>& probe)
{
  auto& probe_first  = std::get<0>(probe);
  auto& probe_second = std::get<1>(probe);
  rmm::device_uvector<double> b_lb_0(problem.n_variables, problem.handle_ptr->get_stream());
  rmm::device_uvector<double> b_ub_0(problem.n_variables, problem.handle_ptr->get_stream());
  rmm::device_uvector<double> b_lb_1(problem.n_variables, problem.handle_ptr->get_stream());
  rmm::device_uvector<double> b_ub_1(problem.n_variables, problem.handle_ptr->get_stream());
  bnd_prb_0.solve(problem, probe_first);
  bnd_prb_0.set_updated_bounds(problem.handle_ptr, make_span(b_lb_0), make_span(b_ub_0));
  bnd_prb_1.solve(problem, probe_second);
  bnd_prb_1.set_updated_bounds(problem.handle_ptr, make_span(b_lb_1), make_span(b_ub_1));

  auto h_lb_0 = host_copy(b_lb_0);
  auto h_ub_0 = host_copy(b_ub_0);
  auto h_lb_1 = host_copy(b_lb_1);
  auto h_ub_1 = host_copy(b_ub_1);
  return std::make_tuple(
    std::move(h_lb_0), std::move(h_ub_0), std::move(h_lb_1), std::move(h_ub_1));
}

void test_multi_probe(std::string path)
{
  auto memory_resource = make_async();
  rmm::mr::set_current_device_resource(memory_resource.get());
  const raft::handle_t handle_{};
  cuopt::mps_parser::mps_data_model_t<int, double> mps_problem =
    cuopt::mps_parser::parse_mps<int, double>(path, false);
  handle_.sync_stream();
  auto op_problem = mps_data_model_to_optimization_problem(&handle_, mps_problem);
  problem_checking_t<int, double>::check_problem_representation(op_problem);
  detail::problem_t<int, double> problem(op_problem);
  mip_solver_settings_t<int, double> default_settings{};
  detail::pdhg_solver_t<int, double> pdhg_solver(problem.handle_ptr, problem);
  detail::pdlp_initial_scaling_strategy_t<int, double> scaling(&handle_,
                                                               problem,
                                                               10,
                                                               1.0,
                                                               pdhg_solver,
                                                               problem.reverse_coefficients,
                                                               problem.reverse_offsets,
                                                               problem.reverse_constraints,
                                                               true);
  detail::mip_solver_t<int, double> solver(problem, default_settings, scaling, cuopt::timer_t(0));
  detail::load_balanced_problem_t<int, double> lb_problem(problem);
  detail::load_balanced_bounds_presolve_t<int, double> lb_prs(lb_problem, solver.context);

  detail::bound_presolve_t<int, double> bnd_prb(solver.context);

  auto probe_tuple       = select_k_random(problem, 100);
  auto bounds_probe_vals = convert_probe_tuple(probe_tuple);
  {
    auto& probe_first = std::get<0>(bounds_probe_vals);
    bnd_prb.solve(problem, probe_first);
    rmm::device_uvector<double> b_lb(problem.n_variables, problem.handle_ptr->get_stream());
    rmm::device_uvector<double> b_ub(problem.n_variables, problem.handle_ptr->get_stream());
    bnd_prb.set_updated_bounds(problem.handle_ptr, make_span(b_lb), make_span(b_ub));

    auto h_lb = host_copy(b_lb);
    auto h_ub = host_copy(b_ub);

    lb_prs.solve(probe_first);

    auto bnds = host_copy(lb_prs.vars_bnd);
    for (int i = 0; i < (int)h_lb.size(); ++i) {
      EXPECT_DOUBLE_EQ(bnds[2 * i], h_lb[i]);
      EXPECT_DOUBLE_EQ(bnds[2 * i + 1], h_ub[i]);
    }
  }
}

TEST(presolve, multi_probe)
{
  std::vector<std::string> test_instances = {
    "mip/50v-10-free-bound.mps", "mip/neos5-free-bound.mps", "mip/neos5.mps"};
  for (const auto& test_instance : test_instances) {
    auto path = make_path_absolute(test_instance);
    test_multi_probe(path);
  }
}

}  // namespace cuopt::linear_programming::test
