#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <mip/mip_constants.hpp>
#include <mip/relaxed_lp/relaxed_lp.cuh>
#include <utilities/copy_helpers.hpp>
#include <utilities/seed_generator.cuh>
#include "constraint_prop.cuh"
#include "simple_rounding.cuh"

#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/partition.h>
#include <thrust/sort.h>

namespace cuopt::linear_programming::detail {

template <typename i_t, typename f_t>
repair_stats_t constraint_prop_t<i_t, f_t>::repair_stats;

template <typename i_t, typename f_t>
constraint_prop_t<i_t, f_t>::constraint_prop_t(mip_solver_context_t<i_t, f_t>& context_)
  : context(context_),
    temp_problem(*context.problem_ptr),
    temp_sol(*context.problem_ptr),
    bounds_update(context),
    bounds_repair(*context.problem_ptr, bounds_update),
    conditional_bounds_update(*context.problem_ptr),
    set_vars(context.problem_ptr->n_variables, context.problem_ptr->handle_ptr->get_stream()),
    unset_vars(context.problem_ptr->n_variables, context.problem_ptr->handle_ptr->get_stream()),
    lb_restore(context.problem_ptr->n_variables, context.problem_ptr->handle_ptr->get_stream()),
    ub_restore(context.problem_ptr->n_variables, context.problem_ptr->handle_ptr->get_stream()),
    assignment_restore(context.problem_ptr->n_variables,
                       context.problem_ptr->handle_ptr->get_stream()),
    rng(cuopt::seed_generator::get_seed(), 0, 0)
{
}

constexpr int n_subsections          = 3 * 7;
constexpr size_t size_of_subsections = n_subsections + 1;

template <typename i_t, typename f_t>
__device__ void assign_offsets(
  raft::device_span<i_t> offsets, i_t category, i_t idx, f_t frac_1, f_t frac_2)
{
  if (frac_1 <= 0.02 && frac_2 > 0.02) {
    offsets[category * 7 + 1] = idx + 1;
  } else if (frac_1 <= 0.05 && frac_2 > 0.05) {
    offsets[category * 7 + 2] = idx + 1;
  } else if (frac_1 <= 0.1 && frac_2 > 0.1) {
    offsets[category * 7 + 3] = idx + 1;
  } else if (frac_1 <= 0.2 && frac_2 > 0.2) {
    offsets[category * 7 + 4] = idx + 1;
  } else if (frac_1 <= 0.3 && frac_2 > 0.3) {
    offsets[category * 7 + 5] = idx + 1;
  } else if (frac_1 <= 0.4 && frac_2 > 0.4) {
    offsets[category * 7 + 6] = idx + 1;
  }
}

template <typename i_t, typename f_t>
void sort_subsections(raft::device_span<i_t> vars,
                      rmm::device_uvector<f_t>& random_vector,
                      rmm::device_uvector<i_t>& offsets,
                      const raft::handle_t* handle_ptr)
{
  size_t temp_storage_bytes = 0;
  rmm::device_uvector<std::byte> d_temp_storage(0, handle_ptr->get_stream());
  rmm::device_uvector<f_t> input_random_vec(random_vector, handle_ptr->get_stream());
  rmm::device_uvector<i_t> input_vars(vars.size(), handle_ptr->get_stream());
  raft::copy(input_vars.data(), vars.data(), vars.size(), handle_ptr->get_stream());
  hipcub::DeviceSegmentedSort::SortPairs(d_temp_storage.data(),
                                      temp_storage_bytes,
                                      input_random_vec.data(),
                                      random_vector.data(),
                                      input_vars.data(),
                                      vars.data(),
                                      vars.size(),
                                      n_subsections,
                                      offsets.data(),
                                      offsets.data() + 1,
                                      handle_ptr->get_stream());

  // Allocate temporary storage
  d_temp_storage.resize(temp_storage_bytes, handle_ptr->get_stream());

  // Run sorting operation
  hipcub::DeviceSegmentedSort::SortPairs(d_temp_storage.data(),
                                      temp_storage_bytes,
                                      input_random_vec.data(),
                                      random_vector.data(),
                                      input_vars.data(),
                                      vars.data(),
                                      vars.size(),
                                      n_subsections,
                                      offsets.data(),
                                      offsets.data() + 1,
                                      handle_ptr->get_stream());
  handle_ptr->sync_stream();
}

template <typename i_t, typename f_t>
__global__ void compute_implied_slack_consumption_per_var(
  typename problem_t<i_t, f_t>::view_t pb,
  raft::device_span<i_t> var_indices,
  raft::device_span<f_t> min_activity,
  raft::device_span<f_t> max_activity,
  raft::device_span<f_t> implied_var_slack_consumption,
  bool is_problem_ii,
  typename mip_solver_settings_t<i_t, f_t>::tolerances_t tols)
{
  i_t var_idx = var_indices[blockIdx.x];
  cuopt_assert(pb.is_integer_var(var_idx), "Variable must be integer!");
  i_t var_offset                       = pb.reverse_offsets[var_idx];
  i_t var_degree                       = pb.reverse_offsets[var_idx + 1] - var_offset;
  f_t th_var_implied_slack_consumption = 0.;
  f_t lb                               = pb.variable_lower_bounds[var_idx];
  f_t ub                               = pb.variable_upper_bounds[var_idx];
  for (i_t i = threadIdx.x; i < var_degree; i += blockDim.x) {
    auto a        = pb.reverse_coefficients[var_offset + i];
    auto cnst_idx = pb.reverse_constraints[var_offset + i];
    auto min_a    = min_activity[cnst_idx];
    auto max_a    = max_activity[cnst_idx];
    auto cnstr_ub = pb.constraint_upper_bounds[cnst_idx];
    auto cnstr_lb = pb.constraint_lower_bounds[cnst_idx];
    // don't consider constraints that are infeasible
    if ((min_a >= cnstr_ub + tols.absolute_tolerance) ||
        (max_a <= cnstr_lb - tols.absolute_tolerance)) {
      continue;
    }

    auto slack_min_act = cnstr_ub - min_a;
    auto slack_max_act = cnstr_lb - max_a;
#pragma unroll
    for (auto act : {slack_min_act, slack_max_act}) {
      f_t slack_consumption_ratio;
      if (is_problem_ii && abs(act) < tols.absolute_tolerance) {
        slack_consumption_ratio = 1000.;
      } else {
        slack_consumption_ratio = (a / act) * (a / act);
      }
      th_var_implied_slack_consumption += slack_consumption_ratio;
    }
  }
  __shared__ f_t shmem[raft::WarpSize];
  f_t block_var_implied_slack_consumption =
    raft::blockReduce(th_var_implied_slack_consumption, (char*)shmem);
  if (threadIdx.x == 0) {
    implied_var_slack_consumption[blockIdx.x] = block_var_implied_slack_consumption;
  }
}

// sort by the implied percent of slack consumption
// across all constraints, sum the square roots of implied slack consumption percent
template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::sort_by_implied_slack_consumption(solution_t<i_t, f_t>& sol,
                                                                    raft::device_span<i_t> vars,
                                                                    bool problem_ii)
{
  CUOPT_LOG_TRACE("Sorting vars by importance");
  rmm::device_uvector<f_t> implied_slack_consumption_per_var(vars.size(),
                                                             sol.handle_ptr->get_stream());
  const i_t block_dim = 128;
  compute_implied_slack_consumption_per_var<i_t, f_t>
    <<<vars.size(), block_dim, 0, sol.handle_ptr->get_stream()>>>(
      sol.problem_ptr->view(),
      vars,
      make_span(bounds_update.upd.min_activity),
      make_span(bounds_update.upd.max_activity),
      make_span(implied_slack_consumption_per_var),
      problem_ii,
      context.settings.get_tolerances());
  thrust::sort_by_key(sol.handle_ptr->get_thrust_policy(),
                      implied_slack_consumption_per_var.begin(),
                      implied_slack_consumption_per_var.end(),
                      vars.data(),
                      thrust::greater<f_t>{});
  sol.handle_ptr->sync_stream();
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::sort_by_interval_and_frac(solution_t<i_t, f_t>& sol,
                                                            raft::device_span<i_t> vars,
                                                            std::mt19937 rng)
{
  // we can't call this function when the problem is ii. it causes false offset computations
  // TODO add assert that the problem is not ii
  auto assgn = make_span(sol.assignment);
  thrust::stable_sort(sol.handle_ptr->get_thrust_policy(),
                      vars.begin(),
                      vars.end(),
                      [lb = sol.problem_ptr->variable_lower_bounds.data(),
                       ub = sol.problem_ptr->variable_upper_bounds.data(),
                       assgn] __device__(i_t v_idx_1, i_t v_idx_2) {
                        f_t bounds_interval_1 = ub[v_idx_1] - lb[v_idx_1];
                        f_t bounds_interval_2 = ub[v_idx_2] - lb[v_idx_2];
                        // if bounds interval are equal (binary and ternary) check fraction
                        // if both bounds intervals are greater than 2. then do fraction
                        if ((bounds_interval_1 == bounds_interval_2) ||
                            (bounds_interval_1 > 2 && bounds_interval_2 > 2)) {
                          f_t frac_1 = get_fractionality_of_val(assgn[v_idx_1]);
                          f_t frac_2 = get_fractionality_of_val(assgn[v_idx_2]);
                          return frac_1 < frac_2;
                        } else {
                          return bounds_interval_1 < bounds_interval_2;
                        }
                      });
  // now do the suffling, for that we need to assign some random values to rnd array
  // we will sort this rnd array and the vars in subsections, so that each subsection will be
  // shuffled in total we will have 3(binary, ternary and rest) x 7 intervals = 21 subsections.
  // first extract these subsections from the data
  rmm::device_uvector<i_t> subsection_offsets(size_of_subsections, sol.handle_ptr->get_stream());
  thrust::fill(
    sol.handle_ptr->get_thrust_policy(), subsection_offsets.begin(), subsection_offsets.end(), -1);
  subsection_offsets.set_element(0, 0, sol.handle_ptr->get_stream());
  subsection_offsets.set_element(n_subsections, vars.size(), sol.handle_ptr->get_stream());
  thrust::for_each(sol.handle_ptr->get_thrust_policy(),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator((i_t)vars.size() - 1),
                   [lb      = make_span(sol.problem_ptr->variable_lower_bounds),
                    ub      = make_span(sol.problem_ptr->variable_upper_bounds),
                    offsets = make_span(subsection_offsets),
                    vars,
                    assgn] __device__(i_t idx) {
                     i_t var_1             = vars[idx];
                     i_t var_2             = vars[idx + 1];
                     f_t bounds_interval_1 = ub[var_1] - lb[var_1];
                     f_t bounds_interval_2 = ub[var_2] - lb[var_2];
                     f_t frac_1            = get_fractionality_of_val(assgn[var_1]);
                     f_t frac_2            = get_fractionality_of_val(assgn[var_2]);
                     if (bounds_interval_1 == 1 && bounds_interval_2 == 1) {
                       i_t category = 0;
                       assign_offsets<i_t, f_t>(offsets, category, idx, frac_1, frac_2);
                     } else if (bounds_interval_1 == 1 && bounds_interval_2 == 2) {
                       offsets[7] = idx + 1;
                     } else if (bounds_interval_1 == 2 && bounds_interval_2 == 2) {
                       i_t category = 1;
                       assign_offsets<i_t, f_t>(offsets, category, idx, frac_1, frac_2);
                     } else if (bounds_interval_1 == 2 && bounds_interval_2 > 2) {
                       offsets[14] = idx + 1;
                     } else {
                       i_t category = 2;
                       assign_offsets<i_t, f_t>(offsets, category, idx, frac_1, frac_2);
                     }
                   });
  // if there are any empty sections fill their offsets as the previous offset
  thrust::for_each(sol.handle_ptr->get_thrust_policy(),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(1),
                   [offsets = subsection_offsets.data()] __device__(i_t idx) {
                     i_t last_existing_offset = 0;
                     for (i_t i = n_subsections; i > 0; --i) {
                       if (offsets[i] == -1) {
                         offsets[i] = last_existing_offset;
                       } else {
                         last_existing_offset = offsets[i];
                       }
                     }
                   });
  auto random_vector = get_random_uniform_vector<i_t, f_t>((i_t)vars.size(), rng);
  rmm::device_uvector<f_t> device_random_vector(random_vector.size(), sol.handle_ptr->get_stream());
  raft::copy(device_random_vector.data(),
             random_vector.data(),
             random_vector.size(),
             sol.handle_ptr->get_stream());
  sort_subsections<i_t, f_t>(vars, device_random_vector, subsection_offsets, sol.handle_ptr);
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::sort_by_frac(solution_t<i_t, f_t>& sol,
                                               raft::device_span<i_t> vars)
{
  auto assgn = make_span(sol.assignment);
  thrust::sort(sol.handle_ptr->get_thrust_policy(),
               vars.begin(),
               vars.end(),
               [assgn] __device__(i_t v_idx_1, i_t v_idx_2) {
                 f_t frac_1 = get_fractionality_of_val(assgn[v_idx_1]);
                 f_t frac_2 = get_fractionality_of_val(assgn[v_idx_2]);
                 return frac_1 < frac_2;
               });
}

template <typename i_t, typename f_t>
struct find_set_int_t {
  // This functor should be called only on integer variables
  f_t eps;
  raft::device_span<f_t> var_lb;
  raft::device_span<f_t> var_ub;
  raft::device_span<f_t> assignment;
  find_set_int_t(f_t eps_,
                 raft::device_span<f_t> lb_,
                 raft::device_span<f_t> ub_,
                 raft::device_span<f_t> assignment_)
    : eps(eps_), var_lb(lb_), var_ub(ub_), assignment(assignment_)
  {
  }

  HDI bool operator()(i_t idx)
  {
    auto var_val = assignment[idx];
    bool is_set  = is_integer<f_t>(var_val);
    return is_set;
  }
};

template <typename i_t, typename f_t>
struct find_unset_int_t {
  // This functor should be called only on integer variables
  f_t eps;
  raft::device_span<f_t> var_lb;
  raft::device_span<f_t> var_ub;
  raft::device_span<f_t> assignment;
  find_unset_int_t(f_t eps_,
                   raft::device_span<f_t> lb_,
                   raft::device_span<f_t> ub_,
                   raft::device_span<f_t> assignment_)
    : eps(eps_), var_lb(lb_), var_ub(ub_), assignment(assignment_)
  {
  }

  HDI bool operator()(i_t idx)
  {
    auto var_val = assignment[idx];
    bool is_set  = is_integer<f_t>(var_val);
    return !is_set;
  }
};

// TODO verify this logic
template <typename i_t, typename f_t>
__device__ bool round_val_on_singleton_and_crossing(
  f_t& assign, f_t v_lb, f_t v_ub, f_t o_lb, f_t o_ub)
{
  if (v_lb == v_ub) {
    assign = floor(v_lb + 0.5);
    return true;
  } else if (v_ub <= o_lb && v_lb <= o_ub) {
    assign = floor(v_lb + 0.5);
    return true;
  } else if (v_ub <= o_lb && v_lb >= o_ub) {
    if (!isfinite(o_lb)) {
      assign = ceil(o_ub - 0.5);
    } else if (!isfinite(o_ub)) {
      assign = floor(o_lb + 0.5);
    } else {
      assign = round((o_lb + o_ub) / 2);
    }
    return true;
  } else if (v_lb >= o_ub && v_ub >= o_lb) {
    assign = ceil(v_ub - 0.5);
    return true;
  }
  // if all cases fail
  else if (v_lb > v_ub) {
    if (!isfinite(o_lb)) {
      assign = ceil(o_ub - 0.5);
    } else if (!isfinite(o_ub)) {
      assign = floor(o_lb + 0.5);
    } else {
      assign = round((o_lb + o_ub) / 2);
    }
    return true;
  }
  return false;
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::collapse_crossing_bounds(problem_t<i_t, f_t>& problem,
                                                           problem_t<i_t, f_t>& orig_problem,
                                                           const raft::handle_t* handle_ptr)
{
  auto lb          = make_span(problem.variable_lower_bounds);
  auto ub          = make_span(problem.variable_upper_bounds);
  auto original_lb = make_span(orig_problem.variable_lower_bounds);
  auto original_ub = make_span(orig_problem.variable_upper_bounds);
  thrust::for_each(
    handle_ptr->get_thrust_policy(),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator((i_t)lb.size()),
    [lb,
     ub,
     original_lb,
     original_ub,
     variable_types = make_span(problem.variable_types),
     int_tol        = problem.tolerances.integrality_tolerance] __device__(i_t idx) {
      auto v_lb = lb[idx];
      auto v_ub = ub[idx];
      auto o_lb = original_lb[idx];
      auto o_ub = original_ub[idx];
      if (v_lb > v_ub) {
        f_t val_to_collapse;
        if (variable_types[idx] == var_t::INTEGER) {
          round_val_on_singleton_and_crossing<i_t, f_t>(val_to_collapse, v_lb, v_ub, o_lb, o_ub);
        } else {
          if (isfinite(o_lb) && isfinite(o_ub)) {
            val_to_collapse = (o_lb + o_ub) / 2;
          } else {
            val_to_collapse = isfinite(o_lb) ? o_lb : o_ub;
          }
        }

        cuopt_assert(o_lb - int_tol <= val_to_collapse && val_to_collapse <= o_ub + int_tol,
                     "Out of original bounds!");
        lb[idx] = val_to_collapse;
        ub[idx] = val_to_collapse;
      }
    });
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::set_bounds_on_fixed_vars(solution_t<i_t, f_t>& sol)
{
  auto assgn = make_span(sol.assignment);
  auto lb    = make_span(sol.problem_ptr->variable_lower_bounds);
  auto ub    = make_span(sol.problem_ptr->variable_upper_bounds);
  thrust::for_each(sol.handle_ptr->get_thrust_policy(),
                   sol.problem_ptr->integer_indices.begin(),
                   sol.problem_ptr->integer_indices.end(),
                   [pb = sol.problem_ptr->view(), assgn, lb, ub] __device__(i_t idx) {
                     auto var_val = assgn[idx];
                     if (pb.is_integer(var_val)) {
                       lb[idx] = var_val;
                       ub[idx] = var_val;
                     }
                   });
}

template <typename i_t, typename f_t>
struct is_bound_fixed_t {
  // This functor should be called only on integer variables
  f_t eps;
  raft::device_span<f_t> lb;
  raft::device_span<f_t> ub;
  raft::device_span<f_t> original_lb;
  raft::device_span<f_t> original_ub;
  raft::device_span<f_t> assignment;
  is_bound_fixed_t(f_t eps_,
                   raft::device_span<f_t> lb_,
                   raft::device_span<f_t> ub_,
                   raft::device_span<f_t> original_lb_,
                   raft::device_span<f_t> original_ub_,
                   raft::device_span<f_t> assignment_)
    : eps(eps_),
      lb(lb_),
      ub(ub_),
      original_lb(original_lb_),
      original_ub(original_ub_),
      assignment(assignment_)
  {
  }

  HDI bool operator()(i_t idx)
  {
    auto v_lb = lb[idx];
    auto v_ub = ub[idx];
    auto o_lb = original_lb[idx];
    auto o_ub = original_ub[idx];
    bool is_singleton =
      round_val_on_singleton_and_crossing<i_t, f_t>(assignment[idx], v_lb, v_ub, o_lb, o_ub);
    return is_singleton;
  }
};

template <typename i_t, typename f_t>
struct fix_bounds_t {
  f_t eps;
  raft::device_span<f_t> lb;
  raft::device_span<f_t> ub;
  raft::device_span<f_t> assign;

  fix_bounds_t(f_t eps_,
               raft::device_span<f_t> lb_,
               raft::device_span<f_t> ub_,
               raft::device_span<f_t> assign_)
    : eps(eps_), lb(lb_), ub(ub_), assign(assign_)
  {
  }

  HDI void operator()(i_t idx)
  {
    auto val = assign[idx];
    lb[idx]  = round(val) - eps;
    ub[idx]  = round(val) + eps;
  }
};

template <typename i_t, typename f_t>
struct greater_than_threshold_t {
  f_t threshold;
  raft::device_span<f_t> assignment;

  greater_than_threshold_t(f_t t, raft::device_span<f_t> assignment_)
    : threshold(t), assignment(assignment_)
  {
  }

  __host__ __device__ bool operator()(const i_t& x) const { return assignment[x] > threshold; }
};

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::copy_bounds(rmm::device_uvector<f_t>& output_lb,
                                              rmm::device_uvector<f_t>& output_ub,
                                              rmm::device_uvector<f_t>& output_assignment,
                                              const rmm::device_uvector<f_t>& input_lb,
                                              const rmm::device_uvector<f_t>& input_ub,
                                              const rmm::device_uvector<f_t>& input_assignment,
                                              const raft::handle_t* handle_ptr)
{
  raft::copy(output_lb.data(), input_lb.data(), input_lb.size(), handle_ptr->get_stream());
  raft::copy(output_ub.data(), input_ub.data(), input_ub.size(), handle_ptr->get_stream());
  raft::copy(output_assignment.data(),
             input_assignment.data(),
             input_assignment.size(),
             handle_ptr->get_stream());
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::save_bounds(solution_t<i_t, f_t>& sol)
{
  copy_bounds(lb_restore,
              ub_restore,
              assignment_restore,
              sol.problem_ptr->variable_lower_bounds,
              sol.problem_ptr->variable_upper_bounds,
              sol.assignment,
              sol.handle_ptr);
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::restore_bounds(solution_t<i_t, f_t>& sol)
{
  copy_bounds(sol.problem_ptr->variable_lower_bounds,
              sol.problem_ptr->variable_upper_bounds,
              sol.assignment,
              lb_restore,
              ub_restore,
              assignment_restore,
              sol.handle_ptr);
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::restore_original_bounds(solution_t<i_t, f_t>& sol,
                                                          solution_t<i_t, f_t>& orig_sol)
{
  copy_bounds(sol.problem_ptr->variable_lower_bounds,
              sol.problem_ptr->variable_upper_bounds,
              sol.assignment,
              orig_sol.problem_ptr->variable_lower_bounds,
              orig_sol.problem_ptr->variable_upper_bounds,
              orig_sol.assignment,
              orig_sol.handle_ptr);
}

template <typename i_t, typename f_t>
std::vector<thrust::pair<i_t, f_t>> constraint_prop_t<i_t, f_t>::generate_bulk_rounding_vector(
  const solution_t<i_t, f_t>& sol,
  const solution_t<i_t, f_t>& orig_sol,
  const std::vector<i_t>& host_vars_to_set,
  const std::optional<std::vector<thrust::pair<f_t, f_t>>> probing_candidates)
{
  const f_t int_tol = orig_sol.problem_ptr->tolerances.integrality_tolerance;
  std::string log_str{"Setting var:\t"};
  std::vector<thrust::pair<i_t, f_t>> var_val_pairs;
  var_val_pairs.reserve(host_vars_to_set.size());
  for (i_t i = 0; i < (i_t)host_vars_to_set.size(); ++i) {
    auto unset_var_idx = host_vars_to_set[i];
    f_t first_probe, second_probe;
    if (probing_candidates.has_value()) {
      // for now get the first one
      thrust::tie(first_probe, second_probe) = probing_candidates.value()[unset_var_idx];
    } else {
      std::tie(first_probe, std::ignore, second_probe) =
        probing_values(sol, orig_sol, unset_var_idx);
    }
    cuopt_assert(orig_sol.problem_ptr->is_integer(first_probe), "Probing value must be an integer");
    cuopt_assert(orig_sol.problem_ptr->is_integer(second_probe),
                 "Probing value must be an integer");
    f_t val_to_round = first_probe;
    // check probing cache if some implied bounds exists
    if (use_probing_cache &&
        bounds_update.probing_cache.contains(*sol.problem_ptr, unset_var_idx)) {
      // check if there are any conflicting bounds
      val_to_round =
        bounds_update.probing_cache.get_least_conflicting_rounding(*sol.problem_ptr,
                                                                   bounds_update.host_lb,
                                                                   bounds_update.host_ub,
                                                                   unset_var_idx,
                                                                   first_probe,
                                                                   second_probe,
                                                                   int_tol);
    }
    cuopt_assert(orig_sol.problem_ptr->variable_lower_bounds.element(
                   unset_var_idx, sol.handle_ptr->get_stream()) <= val_to_round + int_tol &&
                   val_to_round - int_tol <= orig_sol.problem_ptr->variable_upper_bounds.element(
                                               unset_var_idx, sol.handle_ptr->get_stream()),
                 "Variable out of original bounds!");
    var_val_pairs.emplace_back(unset_var_idx, val_to_round);
    log_str.append(std::to_string(unset_var_idx) + ", ");
  }
  CUOPT_LOG_TRACE("%s", log_str.c_str());
  return var_val_pairs;
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::update_host_assignment(const solution_t<i_t, f_t>& sol)
{
  raft::copy(curr_host_assignment.data(),
             sol.assignment.data(),
             sol.problem_ptr->n_variables,
             sol.handle_ptr->get_stream());
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::set_host_bounds(const solution_t<i_t, f_t>& sol)
{
  cuopt_assert(sol.problem_ptr->variable_lower_bounds.size() == bounds_update.host_lb.size(),
               "size of variable lower bound mismatch");
  raft::copy(bounds_update.host_lb.data(),
             sol.problem_ptr->variable_lower_bounds.data(),
             sol.problem_ptr->variable_lower_bounds.size(),
             sol.handle_ptr->get_stream());
  cuopt_assert(sol.problem_ptr->variable_upper_bounds.size() == bounds_update.host_ub.size(),
               "size of variable upper bound mismatch");
  raft::copy(bounds_update.host_ub.data(),
             sol.problem_ptr->variable_upper_bounds.data(),
             sol.problem_ptr->variable_upper_bounds.size(),
             sol.handle_ptr->get_stream());
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::restore_original_bounds_on_unfixed(
  problem_t<i_t, f_t>& problem,
  problem_t<i_t, f_t>& original_problem,
  const raft::handle_t* handle_ptr)
{
  thrust::for_each(handle_ptr->get_thrust_policy(),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(problem.n_variables),
                   [p_v = problem.view(), op_v = original_problem.view()] __device__(i_t var_idx) {
                     if (!p_v.integer_equal(p_v.variable_lower_bounds[var_idx],
                                            p_v.variable_upper_bounds[var_idx]) ||
                         !p_v.is_integer_var(var_idx)) {
                       p_v.variable_lower_bounds[var_idx] = op_v.variable_lower_bounds[var_idx];
                       p_v.variable_upper_bounds[var_idx] = op_v.variable_upper_bounds[var_idx];
                     }
                   });
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::run_repair_procedure(problem_t<i_t, f_t>& problem,
                                                       problem_t<i_t, f_t>& original_problem,
                                                       timer_t& timer,
                                                       const raft::handle_t* handle_ptr)
{
  bounds_update.set_updated_bounds(problem);
  repair_stats.repair_attempts++;
  f_t repair_start_time                = timer.remaining_time();
  i_t n_of_repairs_needed_for_feasible = 0;
  do {
    n_of_repairs_needed_for_feasible++;
    if (timer.check_time_limit()) {
      CUOPT_LOG_DEBUG("Time limit is reached in repair loop!");
      f_t repair_end_time = timer.remaining_time();
      repair_stats.total_time_spent_on_repair += repair_start_time - repair_end_time;
      return false;
    }
    repair_stats.total_repair_loops++;
    collapse_crossing_bounds(problem, original_problem, handle_ptr);
    bool bounds_repaired =
      bounds_repair.repair_problem(problem, original_problem, timer, handle_ptr);
    if (bounds_repaired) {
      repair_stats.intermediate_repair_success++;
      CUOPT_LOG_DEBUG("Bounds repair success, running bounds prop to verify feasibility!");
    }
    f_t bounds_prop_start_time = timer.remaining_time();
    // restore all bounds to the original bounds and run bounds prop on
    // it. note that the number of fixed vars will still be the same, as repair only shifts the vars
    restore_original_bounds_on_unfixed(problem, original_problem, handle_ptr);
    bounds_update.settings.iteration_limit = 100;
    auto term_crit                         = bounds_update.solve(problem);
    bounds_update.settings.iteration_limit = 20;
    if (timer.check_time_limit()) {
      CUOPT_LOG_DEBUG("Time limit is reached in repair loop!");
      f_t repair_end_time = timer.remaining_time();
      repair_stats.total_time_spent_on_repair += repair_start_time - repair_end_time;
      return false;
    }
    if (termination_criterion_t::NO_UPDATE != term_crit) {
      bounds_update.set_updated_bounds(problem);
    }

    f_t bounds_prop_end_time = timer.remaining_time();
    repair_stats.total_time_spent_bounds_prop_after_repair +=
      bounds_prop_start_time - bounds_prop_end_time;
  } while (bounds_update.infeas_constraints_count > 0);
  repair_stats.repair_success++;
  CUOPT_LOG_DEBUG("Repair success: n_of_repair_calls needed: %d", n_of_repairs_needed_for_feasible);
  f_t repair_end_time = timer.remaining_time();
  repair_stats.total_time_spent_on_repair += repair_start_time - repair_end_time;
  return true;
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::is_problem_ii(problem_t<i_t, f_t>& problem)
{
  bounds_update.calculate_activity_on_problem_bounds(problem);
  bounds_update.calculate_infeasible_redundant_constraints(problem);
  bool problem_ii = bounds_update.infeas_constraints_count > 0;
  return problem_ii;
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::find_integer(
  solution_t<i_t, f_t>& sol,
  solution_t<i_t, f_t>& orig_sol,
  f_t lp_run_time_after_feasible,
  timer_t& timer,
  std::optional<std::vector<thrust::pair<f_t, f_t>>> probing_candidates)
{
  using crit_t             = termination_criterion_t;
  auto& unset_integer_vars = unset_vars;
  std::mt19937 rng(cuopt::seed_generator::get_seed());
  lb_restore.resize(sol.problem_ptr->n_variables, sol.handle_ptr->get_stream());
  ub_restore.resize(sol.problem_ptr->n_variables, sol.handle_ptr->get_stream());
  assignment_restore.resize(sol.problem_ptr->n_variables, sol.handle_ptr->get_stream());
  unset_integer_vars.resize(sol.problem_ptr->n_integer_vars, sol.handle_ptr->get_stream());
  curr_host_assignment.resize(sol.problem_ptr->n_variables);
  // round vals that are close enough
  bounds_update.settings.time_limit      = max_timer.remaining_time();
  bounds_update.settings.iteration_limit = 20;
  bounds_update.resize(*sol.problem_ptr);
  if (max_timer.check_time_limit()) {
    CUOPT_LOG_DEBUG("Time limit is reached before bounds prop rounding!");
    sol.round_nearest();
    expand_device_copy(orig_sol.assignment, sol.assignment, sol.handle_ptr->get_stream());
    cuopt_func_call(orig_sol.test_variable_bounds());
    return orig_sol.compute_feasibility();
  }
  raft::copy(unset_integer_vars.data(),
             sol.problem_ptr->integer_indices.data(),
             sol.problem_ptr->n_integer_vars,
             sol.handle_ptr->get_stream());
  CUOPT_LOG_DEBUG("Bounds propagation rounding: unset vars %lu", unset_integer_vars.size());
  if (unset_integer_vars.size() == 0) {
    CUOPT_LOG_ERROR("No integer variables provided in the bounds prop rounding");
    expand_device_copy(orig_sol.assignment, sol.assignment, sol.handle_ptr->get_stream());
    cuopt_func_call(orig_sol.test_variable_bounds());
    return orig_sol.compute_feasibility();
  }
  // this is needed for the sort inside of the loop
  bool problem_ii = is_problem_ii(*sol.problem_ptr);
  // if the problem is ii, run the bounds prop in the beginning
  if (problem_ii) {
    bool bounds_repaired =
      bounds_repair.repair_problem(*sol.problem_ptr, *orig_sol.problem_ptr, timer, sol.handle_ptr);
    if (bounds_repaired) {
      CUOPT_LOG_DEBUG("Initial ii is repaired by bounds repair!");
    } else {
      auto term_crit = bounds_update.solve(*sol.problem_ptr);
      if (termination_criterion_t::NO_UPDATE != term_crit) {
        bounds_update.set_updated_bounds(*sol.problem_ptr);
      }
      rounding_ii = true;
    }
  }
  // do the sort if the problem is not ii. crossing bounds might cause some issues on the sort order
  else {
    // this is a sort to have initial shuffling, so that stable sort within will keep the order and
    // some randomness will be achieved
    sort_by_interval_and_frac(sol, make_span(unset_integer_vars), rng);
  }
  set_host_bounds(sol);
  size_t set_count      = 0;
  bool timeout_happened = false;
  bool repair_tried     = false;
  while (set_count < unset_integer_vars.size()) {
    update_host_assignment(sol);
    if (max_timer.check_time_limit()) {
      CUOPT_LOG_DEBUG("Second time limit is reached returning nearest rounding!");
      sol.round_nearest();
      timeout_happened = true;
      break;
    }
    if (!rounding_ii && timer.check_time_limit()) {
      CUOPT_LOG_DEBUG("First time limit is reached! Continuing without backtracking and repair!");
      rounding_ii = true;
      // this is for not trying the repair procedure again
      repair_tried = true;
    }
    const i_t n_curr_unset = unset_integer_vars.size() - set_count;
    if (!recovery_mode || rounding_ii) {
      if (n_curr_unset > 36) {
        bounds_prop_interval = sqrt(n_curr_unset);
      } else {
        bounds_prop_interval = 1;
      }
    }
    i_t n_vars_to_set = recovery_mode ? 1 : bounds_prop_interval;
    // if we are not at the last stage or if we are in recovery mode, don't sort
    if (n_vars_to_set != 1) {
      sort_by_implied_slack_consumption(
        sol, make_span(unset_integer_vars, set_count, unset_integer_vars.size()), problem_ii);
    }
    std::vector<i_t> host_vars_to_set(n_vars_to_set);
    raft::copy(host_vars_to_set.data(),
               unset_integer_vars.data() + set_count,
               n_vars_to_set,
               sol.handle_ptr->get_stream());
    auto var_val_pairs =
      generate_bulk_rounding_vector(sol, orig_sol, host_vars_to_set, probing_candidates);
    probe(sol, orig_sol.problem_ptr, var_val_pairs, &set_count, unset_integer_vars);
    if (!repair_tried && rounding_ii && !timeout_happened) {
      timer_t repair_timer{std::min(timer.remaining_time() / 5, timer.elapsed_time() / 3)};
      save_bounds(sol);
      // update bounds and run repair procedure
      bool bounds_repaired =
        run_repair_procedure(*sol.problem_ptr, *orig_sol.problem_ptr, repair_timer, sol.handle_ptr);
      if (!bounds_repaired) {
        restore_bounds(sol);
        repair_tried = true;
      } else {
        CUOPT_LOG_DEBUG(
          "Bounds are repaired! Deactivating recovery mode in bounds prop. n_curr_unset %d  "
          "bounds_prop_interval %d",
          n_curr_unset,
          bounds_prop_interval);
        recovery_mode      = false;
        rounding_ii        = false;
        n_iter_in_recovery = 0;
        // test that bounds are really repaired and no ii cstr is present
        cuopt_assert(!is_problem_ii(*sol.problem_ptr),
                     "Problem must not be ii after repair success");
        // during repair procedure some variables might be collapsed
        auto iter = thrust::stable_partition(
          sol.handle_ptr->get_thrust_policy(),
          unset_vars.begin() + set_count,
          unset_vars.end(),
          is_bound_fixed_t<i_t, f_t>{orig_sol.problem_ptr->tolerances.integrality_tolerance,
                                     make_span(sol.problem_ptr->variable_lower_bounds),
                                     make_span(sol.problem_ptr->variable_upper_bounds),
                                     make_span(orig_sol.problem_ptr->variable_lower_bounds),
                                     make_span(orig_sol.problem_ptr->variable_upper_bounds),
                                     make_span(sol.assignment)});
        i_t n_fixed_vars = (iter - (unset_vars.begin() + set_count));
        CUOPT_LOG_TRACE("After repair procedure, number of additional fixed vars %d", n_fixed_vars);
        set_count += n_fixed_vars;
      }
    }
    if (recovery_mode && bounds_update.infeas_constraints_count > 0) {
      // if bounds are not repaired, restore previous bounds
      CUOPT_LOG_DEBUG("Problem is ii in constraint prop. n_curr_unset %d  bounds_prop_interval %d",
                      n_curr_unset,
                      bounds_prop_interval);
      rounding_ii   = true;
      recovery_mode = false;
    }
    if (recovery_mode && (++n_iter_in_recovery == bounds_prop_interval)) {
      CUOPT_LOG_DEBUG(
        "Deactivating recovery mode in bounds prop. n_curr_unset %d  bounds_prop_interval %d",
        n_curr_unset,
        bounds_prop_interval);
      recovery_mode      = false;
      n_iter_in_recovery = 0;
    }
    // we use this to utilize the caching
    // we update from the problem bounds and not the final bounds of bounds update
    // because we might be in a recovery mode where we want to continue with the bounds before bulk
    // which is the unchanged problem bounds
    bounds_update.update_host_bounds(sol.handle_ptr,
                                     make_span(sol.problem_ptr->variable_lower_bounds),
                                     make_span(sol.problem_ptr->variable_upper_bounds));
  }
  CUOPT_LOG_DEBUG("Bounds propagation rounding end: ii constraint count %d",
                  bounds_update.infeas_constraints_count);
  cuopt_assert(sol.test_number_all_integer(), "All integers must be rounded");
  expand_device_copy(orig_sol.assignment, sol.assignment, sol.handle_ptr->get_stream());
  cuopt_func_call(orig_sol.test_variable_bounds());
  // if the constraint is not ii, run LP
  if (bounds_update.infeas_constraints_count == 0 && !timeout_happened) {
    run_lp_with_vars_fixed(*orig_sol.problem_ptr,
                           orig_sol,
                           orig_sol.problem_ptr->integer_indices,
                           context.settings.get_tolerances(),
                           context.lp_state,
                           lp_run_time_after_feasible,
                           true);
  }
  bool res_feasible = orig_sol.compute_feasibility();
  orig_sol.handle_ptr->sync_stream();
  return res_feasible;
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::apply_round(
  solution_t<i_t, f_t>& sol,
  f_t lp_run_time_after_feasible,
  timer_t& timer,
  std::optional<std::vector<thrust::pair<f_t, f_t>>> probing_candidates)
{
  raft::common::nvtx::range fun_scope("constraint prop round");

  // this is second timer that can continue but without recovery mode
  const f_t max_time_for_bounds_prop = 5.;
  max_timer                          = timer_t{max_time_for_bounds_prop};
  if (check_brute_force_rounding(sol)) { return true; }
  recovery_mode      = false;
  rounding_ii        = false;
  n_iter_in_recovery = 0;
  sol.compute_constraints();
  problem_t<i_t, f_t> p(*sol.problem_ptr);
  temp_sol.resize_copy(sol);
  temp_sol.problem_ptr       = &p;
  f_t bounds_prop_start_time = max_timer.remaining_time();
  cuopt_func_call(temp_sol.test_variable_bounds(false));
  bool sol_found =
    find_integer(temp_sol, sol, lp_run_time_after_feasible, timer, probing_candidates);
  f_t bounds_prop_end_time = max_timer.remaining_time();
  repair_stats.total_time_spent_on_bounds_prop += bounds_prop_start_time - bounds_prop_end_time;

  CUOPT_LOG_DEBUG(
    "repair_success %lu repair_attempts %lu intermediate_repair_success %lu total_repair_loops %lu "
    "total_time_spent_on_repair %f total_time_spent_bounds_prop_after_repair %f "
    "total_time_spent_on_bounds_prop %f",
    repair_stats.repair_success,
    repair_stats.repair_attempts,
    repair_stats.intermediate_repair_success,
    repair_stats.total_repair_loops,
    repair_stats.total_time_spent_on_repair,
    repair_stats.total_time_spent_bounds_prop_after_repair,
    repair_stats.total_time_spent_on_bounds_prop);
  if (!sol_found) {
    sol.compute_feasibility();
    return false;
  }
  return sol.compute_feasibility();
}

template <typename i_t, typename f_t>
std::tuple<f_t, f_t, f_t> constraint_prop_t<i_t, f_t>::probing_values(
  const solution_t<i_t, f_t>& sol, const solution_t<i_t, f_t>& orig_sol, i_t idx)
{
  auto v_lb    = bounds_update.host_lb[idx];
  auto v_ub    = bounds_update.host_ub[idx];
  auto var_val = curr_host_assignment[idx];

  const f_t int_tol  = sol.problem_ptr->tolerances.integrality_tolerance;
  auto eps           = int_tol;
  auto within_bounds = (v_lb - eps <= var_val) && (var_val <= v_ub + eps);
  // if it is a collapsed var, return immediately one of the bounds
  if (orig_sol.problem_ptr->integer_equal(v_lb, v_ub)) {
    return std::make_tuple(v_lb, var_val, v_lb);
  }
  if (within_bounds) {
    // the value might have been brought within the bounds when it was out of bounds
    f_t first_round_val = round_nearest(var_val, v_lb, v_ub, int_tol, rng);
    f_t second_round_val;
    auto v_f = std::floor(first_round_val - int_tol);
    auto v_c = std::ceil(first_round_val + int_tol);
    if (first_round_val - var_val >= 0) {
      second_round_val = v_f;
      bool floor_within_bounds =
        (v_lb - eps <= second_round_val) && (second_round_val <= v_ub + eps);
      if (!floor_within_bounds) { second_round_val = v_c; }
    } else {
      second_round_val = v_c;
      bool ceil_within_bounds =
        (v_lb - eps <= second_round_val) && (second_round_val <= v_ub + eps);
      if (!ceil_within_bounds) { second_round_val = v_f; }
    }

    cuopt_assert(v_lb <= first_round_val && first_round_val <= v_ub, "probing value out of bounds");
    cuopt_assert(v_lb <= second_round_val && second_round_val <= v_ub,
                 "probing value out of bounds");
    return std::make_tuple(first_round_val, var_val, second_round_val);
  } else {
    auto orig_v_lb =
      orig_sol.problem_ptr->variable_lower_bounds.element(idx, sol.handle_ptr->get_stream());
    auto orig_v_ub =
      orig_sol.problem_ptr->variable_upper_bounds.element(idx, sol.handle_ptr->get_stream());
    cuopt_assert(v_lb >= orig_v_lb, "Current lb should be greater than original lb");
    cuopt_assert(v_ub <= orig_v_ub, "Current ub should be smaller than original ub");
    v_lb = std::max(v_lb, orig_v_lb);
    v_ub = std::min(v_ub, orig_v_ub);
    // the bounds might cross, so correct them here
    if (v_lb > v_ub) {
      v_lb = orig_v_lb;
      v_ub = orig_v_lb;
    }
    auto v_f = std::floor(var_val);
    auto v_c = std::ceil(var_val);
    if (std::ceil(v_lb) == std::floor(v_ub)) {
      return std::make_tuple(std::ceil(v_lb), var_val, std::floor(v_ub));
    } else if (v_f < std::ceil(v_lb)) {
      v_f = std::ceil(v_lb);
      v_c = v_f + 1;
    } else if (v_c > std::floor(v_ub)) {
      v_c = std::floor(v_ub);
      v_f = v_c - 1;
    }
    cuopt_assert(orig_v_lb <= v_f && v_f <= orig_v_ub, "probing value out of bounds");
    cuopt_assert(orig_v_lb <= v_c && v_c <= orig_v_ub, "probing value out of bounds");
    return std::make_tuple(v_f, var_val, v_c);
  }
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::probe(
  solution_t<i_t, f_t>& sol,
  problem_t<i_t, f_t>* original_problem,
  const std::vector<thrust::pair<i_t, f_t>>& var_probe_val_pairs,
  size_t* set_count_ptr,
  rmm::device_uvector<i_t>& unset_vars)
{
  const f_t int_tol          = sol.problem_ptr->tolerances.integrality_tolerance;
  auto set_count             = *set_count_ptr;
  const bool use_host_bounds = true;
  bounds_update.solve(*sol.problem_ptr, var_probe_val_pairs, use_host_bounds);
  // if we are ii at this point, backtrack the number of variables we have set in this given
  // interval then start setting one by one
  // if we determined that the rounding is ii then don't do any recovery and finish ronuding
  // quickly
  bool bounds_update_ii = bounds_update.infeas_constraints_count > 0;
  if (!recovery_mode && !rounding_ii && bounds_update_ii && bounds_prop_interval != 1) {
    CUOPT_LOG_DEBUG("Activating recovery mode in bounds prop: bounds_prop_interval %d n_ii cstr %d",
                    bounds_prop_interval,
                    bounds_update.infeas_constraints_count);
    // do backtracking
    recovery_mode                          = true;
    bounds_update.infeas_constraints_count = 0;
    n_iter_in_recovery                     = 0;
    return false;
  }
  bounds_update.set_updated_bounds(*sol.problem_ptr);

  // which other variables were affected?
  auto iter = thrust::stable_partition(
    sol.handle_ptr->get_thrust_policy(),
    unset_vars.begin() + set_count,
    unset_vars.end(),
    is_bound_fixed_t<i_t, f_t>{int_tol,
                               make_span(sol.problem_ptr->variable_lower_bounds),
                               make_span(sol.problem_ptr->variable_upper_bounds),
                               make_span(original_problem->variable_lower_bounds),
                               make_span(original_problem->variable_upper_bounds),
                               make_span(sol.assignment)});
  i_t n_fixed_vars = (iter - (unset_vars.begin() + set_count));
  cuopt_assert(n_fixed_vars >= var_probe_val_pairs.size(), "Error in number of vars fixed!");
  set_count += n_fixed_vars;
  CUOPT_LOG_TRACE("Set var count increased from %d to %d", *set_count_ptr, set_count);
  *set_count_ptr = set_count;
  return bounds_update.infeas_constraints_count == 0;
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::relax_crossing_bound_vars(solution_t<i_t, f_t>& sol,
                                                            raft::device_span<f_t> lower_bounds,
                                                            raft::device_span<f_t> upper_bounds)
{
  const f_t int_tol = sol.problem_ptr->tolerances.integrality_tolerance;

  thrust::for_each(sol.handle_ptr->get_thrust_policy(),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(sol.problem_ptr->n_variables),
                   [ass_ptr = sol.assignment.data(),
                    int_tol,
                    lower_bounds,
                    upper_bounds,
                    var_type_ptr = sol.problem_ptr->variable_types.data()] __device__(i_t idx) {
                     if (lower_bounds[idx] - int_tol > upper_bounds[idx]) {
                       printf("relaxing var with %d with bounds %f and %f var type %d\n",
                              idx,
                              lower_bounds[idx],
                              upper_bounds[idx],
                              (i_t)var_type_ptr[idx]);
                       // set one of finite bounds
                       if (!isfinite(lower_bounds[idx])) {
                         ass_ptr[idx] = ceil(upper_bounds[idx] - 0.5);
                       } else if (!isfinite(upper_bounds[idx])) {
                         ass_ptr[idx] = floor(lower_bounds[idx] + 0.5);
                       } else {
                         ass_ptr[idx] = round((lower_bounds[idx] + upper_bounds[idx]) / 2);
                       }
                     }
                   });
}

#if MIP_INSTANTIATE_FLOAT
template class constraint_prop_t<int, float>;
#endif

#if MIP_INSTANTIATE_DOUBLE
template class constraint_prop_t<int, double>;
#endif

}  // namespace cuopt::linear_programming::detail
