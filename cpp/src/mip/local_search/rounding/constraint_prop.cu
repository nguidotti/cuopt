#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <mip/mip_constants.hpp>
#include <mip/relaxed_lp/relaxed_lp.cuh>
#include <utilities/copy_helpers.hpp>
#include <utilities/seed_generator.cuh>
#include "constraint_prop.cuh"
#include "simple_rounding.cuh"

#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/partition.h>
#include <thrust/sort.h>

namespace cuopt::linear_programming::detail {

template <typename i_t, typename f_t>
repair_stats_t constraint_prop_t<i_t, f_t>::repair_stats;

template <typename i_t, typename f_t>
constraint_prop_t<i_t, f_t>::constraint_prop_t(mip_solver_context_t<i_t, f_t>& context_)
  : context(context_),
    temp_problem(*context.problem_ptr),
    temp_sol(*context.problem_ptr),
    bounds_update(context),
    multi_probe(context),
    bounds_repair(*context.problem_ptr, bounds_update),
    conditional_bounds_update(*context.problem_ptr),
    set_vars(context.problem_ptr->n_variables, context.problem_ptr->handle_ptr->get_stream()),
    unset_vars(context.problem_ptr->n_variables, context.problem_ptr->handle_ptr->get_stream()),
    lb_restore(context.problem_ptr->n_variables, context.problem_ptr->handle_ptr->get_stream()),
    ub_restore(context.problem_ptr->n_variables, context.problem_ptr->handle_ptr->get_stream()),
    assignment_restore(context.problem_ptr->n_variables,
                       context.problem_ptr->handle_ptr->get_stream()),
    rng(cuopt::seed_generator::get_seed(), 0, 0)
{
}

constexpr int n_subsections          = 3 * 7;
constexpr size_t size_of_subsections = n_subsections + 1;

template <typename i_t, typename f_t>
__device__ void assign_offsets(
  raft::device_span<i_t> offsets, i_t category, i_t idx, f_t frac_1, f_t frac_2)
{
  if (frac_1 <= 0.02 && frac_2 > 0.02) {
    offsets[category * 7 + 1] = idx + 1;
  } else if (frac_1 <= 0.05 && frac_2 > 0.05) {
    offsets[category * 7 + 2] = idx + 1;
  } else if (frac_1 <= 0.1 && frac_2 > 0.1) {
    offsets[category * 7 + 3] = idx + 1;
  } else if (frac_1 <= 0.2 && frac_2 > 0.2) {
    offsets[category * 7 + 4] = idx + 1;
  } else if (frac_1 <= 0.3 && frac_2 > 0.3) {
    offsets[category * 7 + 5] = idx + 1;
  } else if (frac_1 <= 0.4 && frac_2 > 0.4) {
    offsets[category * 7 + 6] = idx + 1;
  }
}

template <typename i_t, typename f_t>
void sort_subsections(raft::device_span<i_t> vars,
                      rmm::device_uvector<f_t>& random_vector,
                      rmm::device_uvector<i_t>& offsets,
                      const raft::handle_t* handle_ptr)
{
  size_t temp_storage_bytes = 0;
  rmm::device_uvector<std::byte> d_temp_storage(0, handle_ptr->get_stream());
  rmm::device_uvector<f_t> input_random_vec(random_vector, handle_ptr->get_stream());
  rmm::device_uvector<i_t> input_vars(vars.size(), handle_ptr->get_stream());
  raft::copy(input_vars.data(), vars.data(), vars.size(), handle_ptr->get_stream());
  hipcub::DeviceSegmentedSort::SortPairs(d_temp_storage.data(),
                                      temp_storage_bytes,
                                      input_random_vec.data(),
                                      random_vector.data(),
                                      input_vars.data(),
                                      vars.data(),
                                      vars.size(),
                                      n_subsections,
                                      offsets.data(),
                                      offsets.data() + 1,
                                      handle_ptr->get_stream());

  // Allocate temporary storage
  d_temp_storage.resize(temp_storage_bytes, handle_ptr->get_stream());

  // Run sorting operation
  hipcub::DeviceSegmentedSort::SortPairs(d_temp_storage.data(),
                                      temp_storage_bytes,
                                      input_random_vec.data(),
                                      random_vector.data(),
                                      input_vars.data(),
                                      vars.data(),
                                      vars.size(),
                                      n_subsections,
                                      offsets.data(),
                                      offsets.data() + 1,
                                      handle_ptr->get_stream());
  handle_ptr->sync_stream();
}

template <typename i_t, typename f_t>
__global__ void compute_implied_slack_consumption_per_var(
  typename problem_t<i_t, f_t>::view_t pb,
  raft::device_span<i_t> var_indices,
  raft::device_span<f_t> min_activity,
  raft::device_span<f_t> max_activity,
  raft::device_span<f_t> implied_var_slack_consumption,
  bool is_problem_ii,
  typename mip_solver_settings_t<i_t, f_t>::tolerances_t tols)
{
  i_t var_idx = var_indices[blockIdx.x];
  cuopt_assert(pb.is_integer_var(var_idx), "Variable must be integer!");
  i_t var_offset                       = pb.reverse_offsets[var_idx];
  i_t var_degree                       = pb.reverse_offsets[var_idx + 1] - var_offset;
  f_t th_var_implied_slack_consumption = 0.;
  f_t lb                               = pb.variable_lower_bounds[var_idx];
  f_t ub                               = pb.variable_upper_bounds[var_idx];
  for (i_t i = threadIdx.x; i < var_degree; i += blockDim.x) {
    auto a        = pb.reverse_coefficients[var_offset + i];
    auto cnst_idx = pb.reverse_constraints[var_offset + i];
    auto min_a    = min_activity[cnst_idx];
    auto max_a    = max_activity[cnst_idx];
    auto cnstr_ub = pb.constraint_upper_bounds[cnst_idx];
    auto cnstr_lb = pb.constraint_lower_bounds[cnst_idx];
    // don't consider constraints that are infeasible
    if ((min_a >= cnstr_ub + tols.absolute_tolerance) ||
        (max_a <= cnstr_lb - tols.absolute_tolerance)) {
      continue;
    }

    auto slack_min_act = cnstr_ub - min_a;
    auto slack_max_act = cnstr_lb - max_a;
#pragma unroll
    for (auto act : {slack_min_act, slack_max_act}) {
      f_t slack_consumption_ratio;
      if (is_problem_ii && abs(act) < tols.absolute_tolerance) {
        slack_consumption_ratio = 1000.;
      } else {
        slack_consumption_ratio = (a / act) * (a / act);
      }
      th_var_implied_slack_consumption += slack_consumption_ratio;
    }
  }
  __shared__ f_t shmem[raft::WarpSize];
  f_t block_var_implied_slack_consumption =
    raft::blockReduce(th_var_implied_slack_consumption, (char*)shmem);
  if (threadIdx.x == 0) {
    implied_var_slack_consumption[blockIdx.x] = block_var_implied_slack_consumption;
  }
}

// sort by the implied percent of slack consumption
// across all constraints, sum the square roots of implied slack consumption percent
template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::sort_by_implied_slack_consumption(solution_t<i_t, f_t>& sol,
                                                                    raft::device_span<i_t> vars,
                                                                    bool problem_ii)
{
  CUOPT_LOG_TRACE("Sorting vars by importance");
  rmm::device_uvector<f_t> implied_slack_consumption_per_var(vars.size(),
                                                             sol.handle_ptr->get_stream());
  const i_t block_dim = 128;
  auto min_activity   = selected_update ? make_span(multi_probe.upd_1.min_activity)
                                        : make_span(multi_probe.upd_0.min_activity);
  auto max_activity   = selected_update ? make_span(multi_probe.upd_1.max_activity)
                                        : make_span(multi_probe.upd_0.max_activity);
  compute_implied_slack_consumption_per_var<i_t, f_t>
    <<<vars.size(), block_dim, 0, sol.handle_ptr->get_stream()>>>(
      sol.problem_ptr->view(),
      vars,
      min_activity,
      max_activity,
      make_span(implied_slack_consumption_per_var),
      problem_ii,
      context.settings.get_tolerances());
  thrust::sort_by_key(sol.handle_ptr->get_thrust_policy(),
                      implied_slack_consumption_per_var.begin(),
                      implied_slack_consumption_per_var.end(),
                      vars.data(),
                      thrust::greater<f_t>{});
  sol.handle_ptr->sync_stream();
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::sort_by_interval_and_frac(solution_t<i_t, f_t>& sol,
                                                            raft::device_span<i_t> vars,
                                                            std::mt19937 rng)
{
  // we can't call this function when the problem is ii. it causes false offset computations
  // TODO add assert that the problem is not ii
  auto assgn = make_span(sol.assignment);
  thrust::stable_sort(sol.handle_ptr->get_thrust_policy(),
                      vars.begin(),
                      vars.end(),
                      [lb = sol.problem_ptr->variable_lower_bounds.data(),
                       ub = sol.problem_ptr->variable_upper_bounds.data(),
                       assgn] __device__(i_t v_idx_1, i_t v_idx_2) {
                        f_t bounds_interval_1 = ub[v_idx_1] - lb[v_idx_1];
                        f_t bounds_interval_2 = ub[v_idx_2] - lb[v_idx_2];
                        // if bounds interval are equal (binary and ternary) check fraction
                        // if both bounds intervals are greater than 2. then do fraction
                        if ((bounds_interval_1 == bounds_interval_2) ||
                            (bounds_interval_1 > 2 && bounds_interval_2 > 2)) {
                          f_t frac_1 = get_fractionality_of_val(assgn[v_idx_1]);
                          f_t frac_2 = get_fractionality_of_val(assgn[v_idx_2]);
                          return frac_1 < frac_2;
                        } else {
                          return bounds_interval_1 < bounds_interval_2;
                        }
                      });
  // now do the suffling, for that we need to assign some random values to rnd array
  // we will sort this rnd array and the vars in subsections, so that each subsection will be
  // shuffled in total we will have 3(binary, ternary and rest) x 7 intervals = 21 subsections.
  // first extract these subsections from the data
  rmm::device_uvector<i_t> subsection_offsets(size_of_subsections, sol.handle_ptr->get_stream());
  thrust::fill(
    sol.handle_ptr->get_thrust_policy(), subsection_offsets.begin(), subsection_offsets.end(), -1);
  subsection_offsets.set_element(0, 0, sol.handle_ptr->get_stream());
  subsection_offsets.set_element(n_subsections, vars.size(), sol.handle_ptr->get_stream());
  thrust::for_each(sol.handle_ptr->get_thrust_policy(),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator((i_t)vars.size() - 1),
                   [lb      = make_span(sol.problem_ptr->variable_lower_bounds),
                    ub      = make_span(sol.problem_ptr->variable_upper_bounds),
                    offsets = make_span(subsection_offsets),
                    vars,
                    assgn] __device__(i_t idx) {
                     i_t var_1             = vars[idx];
                     i_t var_2             = vars[idx + 1];
                     f_t bounds_interval_1 = ub[var_1] - lb[var_1];
                     f_t bounds_interval_2 = ub[var_2] - lb[var_2];
                     f_t frac_1            = get_fractionality_of_val(assgn[var_1]);
                     f_t frac_2            = get_fractionality_of_val(assgn[var_2]);
                     if (bounds_interval_1 == 1 && bounds_interval_2 == 1) {
                       i_t category = 0;
                       assign_offsets<i_t, f_t>(offsets, category, idx, frac_1, frac_2);
                     } else if (bounds_interval_1 == 1 && bounds_interval_2 == 2) {
                       offsets[7] = idx + 1;
                     } else if (bounds_interval_1 == 2 && bounds_interval_2 == 2) {
                       i_t category = 1;
                       assign_offsets<i_t, f_t>(offsets, category, idx, frac_1, frac_2);
                     } else if (bounds_interval_1 == 2 && bounds_interval_2 > 2) {
                       offsets[14] = idx + 1;
                     } else {
                       i_t category = 2;
                       assign_offsets<i_t, f_t>(offsets, category, idx, frac_1, frac_2);
                     }
                   });
  // if there are any empty sections fill their offsets as the previous offset
  thrust::for_each(sol.handle_ptr->get_thrust_policy(),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(1),
                   [offsets = subsection_offsets.data()] __device__(i_t idx) {
                     i_t last_existing_offset = 0;
                     for (i_t i = n_subsections; i > 0; --i) {
                       if (offsets[i] == -1) {
                         offsets[i] = last_existing_offset;
                       } else {
                         last_existing_offset = offsets[i];
                       }
                     }
                   });
  auto random_vector = get_random_uniform_vector<i_t, f_t>((i_t)vars.size(), rng);
  rmm::device_uvector<f_t> device_random_vector(random_vector.size(), sol.handle_ptr->get_stream());
  raft::copy(device_random_vector.data(),
             random_vector.data(),
             random_vector.size(),
             sol.handle_ptr->get_stream());
  sort_subsections<i_t, f_t>(vars, device_random_vector, subsection_offsets, sol.handle_ptr);
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::sort_by_frac(solution_t<i_t, f_t>& sol,
                                               raft::device_span<i_t> vars)
{
  auto assgn = make_span(sol.assignment);
  thrust::sort(sol.handle_ptr->get_thrust_policy(),
               vars.begin(),
               vars.end(),
               [assgn] __device__(i_t v_idx_1, i_t v_idx_2) {
                 f_t frac_1 = get_fractionality_of_val(assgn[v_idx_1]);
                 f_t frac_2 = get_fractionality_of_val(assgn[v_idx_2]);
                 return frac_1 < frac_2;
               });
}

template <typename i_t, typename f_t>
struct find_set_int_t {
  // This functor should be called only on integer variables
  f_t eps;
  raft::device_span<f_t> var_lb;
  raft::device_span<f_t> var_ub;
  raft::device_span<f_t> assignment;
  find_set_int_t(f_t eps_,
                 raft::device_span<f_t> lb_,
                 raft::device_span<f_t> ub_,
                 raft::device_span<f_t> assignment_)
    : eps(eps_), var_lb(lb_), var_ub(ub_), assignment(assignment_)
  {
  }

  HDI bool operator()(i_t idx)
  {
    auto var_val = assignment[idx];
    bool is_set  = is_integer<f_t>(var_val);
    return is_set;
  }
};

template <typename i_t, typename f_t>
struct find_unset_int_t {
  // This functor should be called only on integer variables
  f_t eps;
  raft::device_span<f_t> var_lb;
  raft::device_span<f_t> var_ub;
  raft::device_span<f_t> assignment;
  find_unset_int_t(f_t eps_,
                   raft::device_span<f_t> lb_,
                   raft::device_span<f_t> ub_,
                   raft::device_span<f_t> assignment_)
    : eps(eps_), var_lb(lb_), var_ub(ub_), assignment(assignment_)
  {
  }

  HDI bool operator()(i_t idx)
  {
    auto var_val = assignment[idx];
    bool is_set  = is_integer<f_t>(var_val);
    return !is_set;
  }
};

// TODO verify this logic
template <typename i_t, typename f_t>
__device__ bool round_val_on_singleton_and_crossing(
  f_t& assign, f_t v_lb, f_t v_ub, f_t o_lb, f_t o_ub)
{
  if (v_lb == v_ub) {
    assign = floor(v_lb + 0.5);
    return true;
  } else if (v_ub <= o_lb && v_lb <= o_ub) {
    assign = floor(v_lb + 0.5);
    return true;
  } else if (v_ub <= o_lb && v_lb >= o_ub) {
    if (!isfinite(o_lb)) {
      assign = ceil(o_ub - 0.5);
    } else if (!isfinite(o_ub)) {
      assign = floor(o_lb + 0.5);
    } else {
      assign = round((o_lb + o_ub) / 2);
    }
    return true;
  } else if (v_lb >= o_ub && v_ub >= o_lb) {
    assign = ceil(v_ub - 0.5);
    return true;
  }
  // if all cases fail
  else if (v_lb > v_ub) {
    if (!isfinite(o_lb)) {
      assign = ceil(o_ub - 0.5);
    } else if (!isfinite(o_ub)) {
      assign = floor(o_lb + 0.5);
    } else {
      assign = round((o_lb + o_ub) / 2);
    }
    return true;
  }
  return false;
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::collapse_crossing_bounds(problem_t<i_t, f_t>& problem,
                                                           problem_t<i_t, f_t>& orig_problem,
                                                           const raft::handle_t* handle_ptr)
{
  auto lb          = make_span(problem.variable_lower_bounds);
  auto ub          = make_span(problem.variable_upper_bounds);
  auto original_lb = make_span(orig_problem.variable_lower_bounds);
  auto original_ub = make_span(orig_problem.variable_upper_bounds);
  thrust::for_each(
    handle_ptr->get_thrust_policy(),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator((i_t)lb.size()),
    [lb,
     ub,
     original_lb,
     original_ub,
     variable_types = make_span(problem.variable_types),
     int_tol        = problem.tolerances.integrality_tolerance] __device__(i_t idx) {
      auto v_lb = lb[idx];
      auto v_ub = ub[idx];
      auto o_lb = original_lb[idx];
      auto o_ub = original_ub[idx];
      if (v_lb > v_ub) {
        f_t val_to_collapse;
        if (variable_types[idx] == var_t::INTEGER) {
          round_val_on_singleton_and_crossing<i_t, f_t>(val_to_collapse, v_lb, v_ub, o_lb, o_ub);
        } else {
          if (isfinite(o_lb) && isfinite(o_ub)) {
            val_to_collapse = (o_lb + o_ub) / 2;
          } else {
            val_to_collapse = isfinite(o_lb) ? o_lb : o_ub;
          }
        }

        cuopt_assert(o_lb - int_tol <= val_to_collapse && val_to_collapse <= o_ub + int_tol,
                     "Out of original bounds!");
        lb[idx] = val_to_collapse;
        ub[idx] = val_to_collapse;
      }
    });
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::set_bounds_on_fixed_vars(solution_t<i_t, f_t>& sol)
{
  auto assgn = make_span(sol.assignment);
  auto lb    = make_span(sol.problem_ptr->variable_lower_bounds);
  auto ub    = make_span(sol.problem_ptr->variable_upper_bounds);
  thrust::for_each(sol.handle_ptr->get_thrust_policy(),
                   sol.problem_ptr->integer_indices.begin(),
                   sol.problem_ptr->integer_indices.end(),
                   [pb = sol.problem_ptr->view(), assgn, lb, ub] __device__(i_t idx) {
                     auto var_val = assgn[idx];
                     if (pb.is_integer(var_val)) {
                       lb[idx] = var_val;
                       ub[idx] = var_val;
                     }
                   });
}

template <typename i_t, typename f_t>
struct is_bound_fixed_t {
  // This functor should be called only on integer variables
  f_t eps;
  raft::device_span<f_t> lb;
  raft::device_span<f_t> ub;
  raft::device_span<f_t> original_lb;
  raft::device_span<f_t> original_ub;
  raft::device_span<f_t> assignment;
  is_bound_fixed_t(f_t eps_,
                   raft::device_span<f_t> lb_,
                   raft::device_span<f_t> ub_,
                   raft::device_span<f_t> original_lb_,
                   raft::device_span<f_t> original_ub_,
                   raft::device_span<f_t> assignment_)
    : eps(eps_),
      lb(lb_),
      ub(ub_),
      original_lb(original_lb_),
      original_ub(original_ub_),
      assignment(assignment_)
  {
  }

  HDI bool operator()(i_t idx)
  {
    auto v_lb = lb[idx];
    auto v_ub = ub[idx];
    auto o_lb = original_lb[idx];
    auto o_ub = original_ub[idx];
    bool is_singleton =
      round_val_on_singleton_and_crossing<i_t, f_t>(assignment[idx], v_lb, v_ub, o_lb, o_ub);
    return is_singleton;
  }
};

template <typename i_t, typename f_t>
struct fix_bounds_t {
  f_t eps;
  raft::device_span<f_t> lb;
  raft::device_span<f_t> ub;
  raft::device_span<f_t> assign;

  fix_bounds_t(f_t eps_,
               raft::device_span<f_t> lb_,
               raft::device_span<f_t> ub_,
               raft::device_span<f_t> assign_)
    : eps(eps_), lb(lb_), ub(ub_), assign(assign_)
  {
  }

  HDI void operator()(i_t idx)
  {
    auto val = assign[idx];
    lb[idx]  = round(val) - eps;
    ub[idx]  = round(val) + eps;
  }
};

template <typename i_t, typename f_t>
struct greater_than_threshold_t {
  f_t threshold;
  raft::device_span<f_t> assignment;

  greater_than_threshold_t(f_t t, raft::device_span<f_t> assignment_)
    : threshold(t), assignment(assignment_)
  {
  }

  __host__ __device__ bool operator()(const i_t& x) const { return assignment[x] > threshold; }
};

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::copy_bounds(rmm::device_uvector<f_t>& output_lb,
                                              rmm::device_uvector<f_t>& output_ub,
                                              const rmm::device_uvector<f_t>& input_lb,
                                              const rmm::device_uvector<f_t>& input_ub,
                                              const raft::handle_t* handle_ptr)
{
  raft::copy(output_lb.data(), input_lb.data(), input_lb.size(), handle_ptr->get_stream());
  raft::copy(output_ub.data(), input_ub.data(), input_ub.size(), handle_ptr->get_stream());
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::copy_bounds(rmm::device_uvector<f_t>& output_lb,
                                              rmm::device_uvector<f_t>& output_ub,
                                              rmm::device_uvector<f_t>& output_assignment,
                                              const rmm::device_uvector<f_t>& input_lb,
                                              const rmm::device_uvector<f_t>& input_ub,
                                              const rmm::device_uvector<f_t>& input_assignment,
                                              const raft::handle_t* handle_ptr)
{
  copy_bounds(output_lb, output_ub, input_lb, input_ub, handle_ptr);
  raft::copy(output_assignment.data(),
             input_assignment.data(),
             input_assignment.size(),
             handle_ptr->get_stream());
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::save_bounds(solution_t<i_t, f_t>& sol)
{
  copy_bounds(lb_restore,
              ub_restore,
              assignment_restore,
              sol.problem_ptr->variable_lower_bounds,
              sol.problem_ptr->variable_upper_bounds,
              sol.assignment,
              sol.handle_ptr);
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::restore_bounds(solution_t<i_t, f_t>& sol)
{
  copy_bounds(sol.problem_ptr->variable_lower_bounds,
              sol.problem_ptr->variable_upper_bounds,
              sol.assignment,
              lb_restore,
              ub_restore,
              assignment_restore,
              sol.handle_ptr);
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::restore_original_bounds(solution_t<i_t, f_t>& sol,
                                                          solution_t<i_t, f_t>& orig_sol)
{
  copy_bounds(sol.problem_ptr->variable_lower_bounds,
              sol.problem_ptr->variable_upper_bounds,
              sol.assignment,
              orig_sol.problem_ptr->variable_lower_bounds,
              orig_sol.problem_ptr->variable_upper_bounds,
              orig_sol.assignment,
              orig_sol.handle_ptr);
}

template <typename i_t, typename f_t>
thrust::pair<f_t, f_t> constraint_prop_t<i_t, f_t>::generate_double_probing_pair(
  const solution_t<i_t, f_t>& sol,
  const solution_t<i_t, f_t>& orig_sol,
  i_t unset_var_idx,
  const std::optional<std::reference_wrapper<probing_config_t<i_t, f_t>>> probing_config,
  bool bulk_rounding)
{
  f_t first_probe, second_probe;
  if (probing_config.has_value()) {
    // for now get the first one
    auto [from_first, from_second] = probing_config.value().get().probing_values[unset_var_idx];
    std::mt19937 rng(cuopt::seed_generator::get_seed());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    f_t random_value  = dist(rng);
    f_t average_value = (from_first + from_second) / 2;
    if (random_value > 0.5) {
      average_value = ceil(average_value);
    } else {
      average_value = floor(average_value);
    }

    // if the first set of values are more represented use the second value
    if (probing_config.value().get().n_of_fixed_from_first >
        probing_config.value().get().n_of_fixed_from_second) {
      // if it is the same as less represented
      if (average_value == from_second) {
        first_probe  = from_second;
        second_probe = from_first;
      } else {
        first_probe  = average_value;
        second_probe = from_second;
      }
    } else {
      // if it is the same as less represented
      if (average_value == from_first) {
        first_probe  = from_first;
        second_probe = from_second;
      } else {
        first_probe  = average_value;
        second_probe = from_first;
      }
    }
  } else {
    std::tie(first_probe, std::ignore, second_probe) = probing_values(sol, orig_sol, unset_var_idx);
    // do another draw in bulk rounding, so the second vector is randomly drawn
    if (bulk_rounding) {
      std::tie(second_probe, std::ignore, std::ignore) =
        probing_values(sol, orig_sol, unset_var_idx);
    }
  }
  return thrust::make_pair(first_probe, second_probe);
}

template <typename i_t, typename f_t>
std::tuple<std::vector<i_t>, std::vector<f_t>, std::vector<f_t>>
constraint_prop_t<i_t, f_t>::generate_bulk_rounding_vector(
  const solution_t<i_t, f_t>& sol,
  const solution_t<i_t, f_t>& orig_sol,
  const std::vector<i_t>& host_vars_to_set,
  const std::optional<std::reference_wrapper<probing_config_t<i_t, f_t>>> probing_config)
{
  const f_t int_tol = orig_sol.problem_ptr->tolerances.integrality_tolerance;
  std::string log_str{"Setting var:\t"};
  std::tuple<std::vector<i_t>, std::vector<f_t>, std::vector<f_t>> var_probe_vals;
  std::get<0>(var_probe_vals).resize(host_vars_to_set.size());
  std::get<1>(var_probe_vals).resize(host_vars_to_set.size());
  std::get<2>(var_probe_vals).resize(host_vars_to_set.size());
  for (i_t i = 0; i < (i_t)host_vars_to_set.size(); ++i) {
    auto unset_var_idx = host_vars_to_set[i];
    f_t first_probe, second_probe;
    // if it is a bulk rounding do
    if (host_vars_to_set.size() > 1) {
      cuda::std::tie(first_probe, second_probe) =
        generate_double_probing_pair(sol, orig_sol, unset_var_idx, probing_config, true);
    } else {
      cuda::std::tie(first_probe, second_probe) =
        generate_double_probing_pair(sol, orig_sol, unset_var_idx, probing_config, false);
    }
    cuopt_assert(orig_sol.problem_ptr->is_integer(first_probe), "Probing value must be an integer");
    cuopt_assert(orig_sol.problem_ptr->is_integer(second_probe),
                 "Probing value must be an integer");
    f_t val_to_round = first_probe;
    // check probing cache if some implied bounds exists
    if (use_probing_cache &&
        bounds_update.probing_cache.contains(*sol.problem_ptr, unset_var_idx)) {
      // check if there are any conflicting bounds
      val_to_round = bounds_update.probing_cache.get_least_conflicting_rounding(*sol.problem_ptr,
                                                                                multi_probe.host_lb,
                                                                                multi_probe.host_ub,
                                                                                unset_var_idx,
                                                                                first_probe,
                                                                                second_probe,
                                                                                int_tol);
      if (val_to_round == second_probe) { second_probe = first_probe; }
    }
    cuopt_assert(orig_sol.problem_ptr->variable_lower_bounds.element(
                   unset_var_idx, sol.handle_ptr->get_stream()) <= val_to_round + int_tol &&
                   val_to_round - int_tol <= orig_sol.problem_ptr->variable_upper_bounds.element(
                                               unset_var_idx, sol.handle_ptr->get_stream()),
                 "Variable out of original bounds!");
    cuopt_assert(orig_sol.problem_ptr->variable_lower_bounds.element(
                   unset_var_idx, sol.handle_ptr->get_stream()) <= second_probe + int_tol &&
                   second_probe - int_tol <= orig_sol.problem_ptr->variable_upper_bounds.element(
                                               unset_var_idx, sol.handle_ptr->get_stream()),
                 "Variable out of original bounds!");
    std::get<0>(var_probe_vals)[i] = unset_var_idx;
    std::get<1>(var_probe_vals)[i] = val_to_round;
    std::get<2>(var_probe_vals)[i] = second_probe;
    log_str.append(std::to_string(unset_var_idx) + ", ");
  }
  CUOPT_LOG_TRACE("%s", log_str.c_str());
  return var_probe_vals;
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::update_host_assignment(const solution_t<i_t, f_t>& sol)
{
  raft::copy(curr_host_assignment.data(),
             sol.assignment.data(),
             sol.problem_ptr->n_variables,
             sol.handle_ptr->get_stream());
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::set_host_bounds(const solution_t<i_t, f_t>& sol)
{
  cuopt_assert(sol.problem_ptr->variable_lower_bounds.size() == multi_probe.host_lb.size(),
               "size of variable lower bound mismatch");
  raft::copy(multi_probe.host_lb.data(),
             sol.problem_ptr->variable_lower_bounds.data(),
             sol.problem_ptr->variable_lower_bounds.size(),
             sol.handle_ptr->get_stream());
  cuopt_assert(sol.problem_ptr->variable_upper_bounds.size() == multi_probe.host_ub.size(),
               "size of variable upper bound mismatch");
  raft::copy(multi_probe.host_ub.data(),
             sol.problem_ptr->variable_upper_bounds.data(),
             sol.problem_ptr->variable_upper_bounds.size(),
             sol.handle_ptr->get_stream());
}

template <typename i_t, typename f_t>
void constraint_prop_t<i_t, f_t>::restore_original_bounds_on_unfixed(
  problem_t<i_t, f_t>& problem,
  problem_t<i_t, f_t>& original_problem,
  const raft::handle_t* handle_ptr)
{
  thrust::for_each(handle_ptr->get_thrust_policy(),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(problem.n_variables),
                   [p_v = problem.view(), op_v = original_problem.view()] __device__(i_t var_idx) {
                     if (!p_v.integer_equal(p_v.variable_lower_bounds[var_idx],
                                            p_v.variable_upper_bounds[var_idx]) ||
                         !p_v.is_integer_var(var_idx)) {
                       p_v.variable_lower_bounds[var_idx] = op_v.variable_lower_bounds[var_idx];
                       p_v.variable_upper_bounds[var_idx] = op_v.variable_upper_bounds[var_idx];
                     }
                   });
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::run_repair_procedure(problem_t<i_t, f_t>& problem,
                                                       problem_t<i_t, f_t>& original_problem,
                                                       timer_t& timer,
                                                       const raft::handle_t* handle_ptr)
{
  // select the first probing value
  i_t select = 0;
  multi_probe.set_updated_bounds(problem, select, handle_ptr);
  bounds_update.copy_input_bounds(problem);
  repair_stats.repair_attempts++;
  f_t repair_start_time                = timer.remaining_time();
  i_t n_of_repairs_needed_for_feasible = 0;
  do {
    n_of_repairs_needed_for_feasible++;
    if (timer.check_time_limit()) {
      CUOPT_LOG_DEBUG("Time limit is reached in repair loop!");
      f_t repair_end_time = timer.remaining_time();
      repair_stats.total_time_spent_on_repair += repair_start_time - repair_end_time;
      return false;
    }
    repair_stats.total_repair_loops++;
    collapse_crossing_bounds(problem, original_problem, handle_ptr);
    bool bounds_repaired =
      bounds_repair.repair_problem(problem, original_problem, timer, handle_ptr);
    if (bounds_repaired) {
      repair_stats.intermediate_repair_success++;
      CUOPT_LOG_DEBUG("Bounds repair success, running bounds prop to verify feasibility!");
    }
    f_t bounds_prop_start_time = timer.remaining_time();
    // restore all bounds to the original bounds and run bounds prop on
    // it. note that the number of fixed vars will still be the same, as repair only shifts the vars
    restore_original_bounds_on_unfixed(problem, original_problem, handle_ptr);
    bounds_update.settings.iteration_limit = 100;
    bounds_update.settings.time_limit      = timer.remaining_time();
    auto term_crit                         = bounds_update.solve(problem);
    bounds_update.settings.iteration_limit = 50;
    if (timer.check_time_limit()) {
      CUOPT_LOG_DEBUG("Time limit is reached in repair loop!");
      f_t repair_end_time = timer.remaining_time();
      repair_stats.total_time_spent_on_repair += repair_start_time - repair_end_time;
      return false;
    }
    if (termination_criterion_t::NO_UPDATE != term_crit) {
      bounds_update.set_updated_bounds(problem);
    }

    f_t bounds_prop_end_time = timer.remaining_time();
    repair_stats.total_time_spent_bounds_prop_after_repair +=
      bounds_prop_start_time - bounds_prop_end_time;
  } while (bounds_update.infeas_constraints_count > 0);
  repair_stats.repair_success++;
  CUOPT_LOG_DEBUG("Repair success: n_of_repair_calls needed: %d", n_of_repairs_needed_for_feasible);
  f_t repair_end_time = timer.remaining_time();
  repair_stats.total_time_spent_on_repair += repair_start_time - repair_end_time;
  // test that bounds are really repaired and no ii cstr is present
  cuopt_assert(!is_problem_ii(problem), "Problem must not be ii after repair success");
  return true;
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::is_problem_ii(problem_t<i_t, f_t>& problem)
{
  bounds_update.calculate_activity_on_problem_bounds(problem);
  bounds_update.calculate_infeasible_redundant_constraints(problem);
  bool problem_ii = bounds_update.infeas_constraints_count > 0;
  return problem_ii;
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::find_integer(
  solution_t<i_t, f_t>& sol,
  solution_t<i_t, f_t>& orig_sol,
  f_t lp_run_time_after_feasible,
  timer_t& timer,
  std::optional<std::reference_wrapper<probing_config_t<i_t, f_t>>> probing_config)
{
  using crit_t             = termination_criterion_t;
  auto& unset_integer_vars = unset_vars;
  std::mt19937 rng(cuopt::seed_generator::get_seed());
  lb_restore.resize(sol.problem_ptr->n_variables, sol.handle_ptr->get_stream());
  ub_restore.resize(sol.problem_ptr->n_variables, sol.handle_ptr->get_stream());
  assignment_restore.resize(sol.problem_ptr->n_variables, sol.handle_ptr->get_stream());
  unset_integer_vars.resize(sol.problem_ptr->n_integer_vars, sol.handle_ptr->get_stream());
  curr_host_assignment.resize(sol.problem_ptr->n_variables);
  // round vals that are close enough
  bounds_update.settings.time_limit      = max_timer.remaining_time();
  bounds_update.settings.iteration_limit = 50;
  bounds_update.resize(*sol.problem_ptr);
  multi_probe.settings.iteration_limit = 50;
  multi_probe.settings.time_limit      = max_timer.remaining_time();
  multi_probe.resize(*sol.problem_ptr);
  if (max_timer.check_time_limit()) {
    CUOPT_LOG_DEBUG("Time limit is reached before bounds prop rounding!");
    sol.round_nearest();
    expand_device_copy(orig_sol.assignment, sol.assignment, sol.handle_ptr->get_stream());
    cuopt_func_call(orig_sol.test_variable_bounds());
    return orig_sol.compute_feasibility();
  }
  raft::copy(unset_integer_vars.data(),
             sol.problem_ptr->integer_indices.data(),
             sol.problem_ptr->n_integer_vars,
             sol.handle_ptr->get_stream());
  CUOPT_LOG_DEBUG("Bounds propagation rounding: unset vars %lu", unset_integer_vars.size());
  if (unset_integer_vars.size() == 0) {
    CUOPT_LOG_ERROR("No integer variables provided in the bounds prop rounding");
    expand_device_copy(orig_sol.assignment, sol.assignment, sol.handle_ptr->get_stream());
    cuopt_func_call(orig_sol.test_variable_bounds());
    return orig_sol.compute_feasibility();
  }
  // this is needed for the sort inside of the loop
  bool problem_ii = is_problem_ii(*sol.problem_ptr);
  // if the problem is ii, run the bounds prop in the beginning
  if (problem_ii) {
    bool bounds_repaired =
      bounds_repair.repair_problem(*sol.problem_ptr, *orig_sol.problem_ptr, timer, sol.handle_ptr);
    if (bounds_repaired) {
      CUOPT_LOG_DEBUG("Initial ii is repaired by bounds repair!");
    } else {
      auto term_crit = bounds_update.solve(*sol.problem_ptr);
      if (termination_criterion_t::NO_UPDATE != term_crit) {
        bounds_update.set_updated_bounds(*sol.problem_ptr);
      }
      rounding_ii = true;
    }
  }
  // do the sort if the problem is not ii. crossing bounds might cause some issues on the sort order
  else {
    // this is a sort to have initial shuffling, so that stable sort within will keep the order and
    // some randomness will be achieved
    sort_by_interval_and_frac(sol, make_span(unset_integer_vars), rng);
  }
  set_host_bounds(sol);
  size_t set_count               = 0;
  bool timeout_happened          = false;
  i_t n_failed_repair_iterations = 0;
  while (set_count < unset_integer_vars.size()) {
    CUOPT_LOG_TRACE("n_set_vars %d vars to set %lu", set_count, unset_integer_vars.size());
    update_host_assignment(sol);
    if (max_timer.check_time_limit()) {
      CUOPT_LOG_DEBUG("Second time limit is reached returning nearest rounding!");
      sol.round_nearest();
      timeout_happened = true;
      break;
    }
    if (!rounding_ii && timer.check_time_limit()) {
      CUOPT_LOG_DEBUG("First time limit is reached! Continuing without backtracking and repair!");
      rounding_ii = true;
      // this is to not try the repair procedure again
      n_failed_repair_iterations = std::numeric_limits<i_t>::max();
    }
    const i_t n_curr_unset = unset_integer_vars.size() - set_count;
    if (single_rounding_only) {
      bounds_prop_interval = 1;
    } else if (!recovery_mode || rounding_ii) {
      if (n_curr_unset > 36) {
        bounds_prop_interval = sqrt(n_curr_unset);
      } else {
        bounds_prop_interval = 1;
      }
    }
    i_t n_vars_to_set = recovery_mode ? 1 : bounds_prop_interval;
    // if we are not at the last stage or if we are in recovery mode, don't sort
    if (n_vars_to_set != 1) {
      sort_by_implied_slack_consumption(
        sol, make_span(unset_integer_vars, set_count, unset_integer_vars.size()), problem_ii);
    }
    std::vector<i_t> host_vars_to_set(n_vars_to_set);
    raft::copy(host_vars_to_set.data(),
               unset_integer_vars.data() + set_count,
               n_vars_to_set,
               sol.handle_ptr->get_stream());
    auto var_probe_vals =
      generate_bulk_rounding_vector(sol, orig_sol, host_vars_to_set, probing_config);
    probe(
      sol, orig_sol.problem_ptr, var_probe_vals, &set_count, unset_integer_vars, probing_config);
    if (!(n_failed_repair_iterations >= max_n_failed_repair_iterations) && rounding_ii &&
        !timeout_happened) {
      timer_t repair_timer{std::min(timer.remaining_time() / 5, timer.elapsed_time() / 3)};
      save_bounds(sol);
      // update bounds and run repair procedure
      bool bounds_repaired =
        run_repair_procedure(*sol.problem_ptr, *orig_sol.problem_ptr, repair_timer, sol.handle_ptr);
      if (!bounds_repaired) {
        restore_bounds(sol);
        n_failed_repair_iterations++;
      } else {
        CUOPT_LOG_DEBUG(
          "Bounds are repaired! Deactivating recovery mode in bounds prop. n_curr_unset %d  "
          "bounds_prop_interval %d",
          n_curr_unset,
          bounds_prop_interval);
        recovery_mode      = false;
        rounding_ii        = false;
        n_iter_in_recovery = 0;
        // during repair procedure some variables might be collapsed
        auto iter = thrust::stable_partition(
          sol.handle_ptr->get_thrust_policy(),
          unset_vars.begin() + set_count,
          unset_vars.end(),
          is_bound_fixed_t<i_t, f_t>{orig_sol.problem_ptr->tolerances.integrality_tolerance,
                                     make_span(sol.problem_ptr->variable_lower_bounds),
                                     make_span(sol.problem_ptr->variable_upper_bounds),
                                     make_span(orig_sol.problem_ptr->variable_lower_bounds),
                                     make_span(orig_sol.problem_ptr->variable_upper_bounds),
                                     make_span(sol.assignment)});
        i_t n_fixed_vars = (iter - (unset_vars.begin() + set_count));
        CUOPT_LOG_TRACE("After repair procedure, number of additional fixed vars %d", n_fixed_vars);
        set_count += n_fixed_vars;
      }
    }
    // we can keep normal bounds_update here because this is only activated after the  repair
    if (recovery_mode && multi_probe.infeas_constraints_count_0 > 0 &&
        multi_probe.infeas_constraints_count_1 > 0) {
      CUOPT_LOG_DEBUG("Problem is ii in constraint prop. n_curr_unset %d  bounds_prop_interval %d",
                      n_curr_unset,
                      bounds_prop_interval);
      rounding_ii   = true;
      recovery_mode = false;
    }
    if (recovery_mode && (++n_iter_in_recovery == bounds_prop_interval)) {
      CUOPT_LOG_DEBUG(
        "Deactivating recovery mode in bounds prop. n_curr_unset %d  bounds_prop_interval %d",
        n_curr_unset,
        bounds_prop_interval);
      recovery_mode      = false;
      n_iter_in_recovery = 0;
    }
    // we use this to utilize the caching
    // we update from the problem bounds and not the final bounds of bounds update
    // because we might be in a recovery mode where we want to continue with the bounds before bulk
    // which is the unchanged problem bounds
    multi_probe.update_host_bounds(sol.handle_ptr,
                                   make_span(sol.problem_ptr->variable_lower_bounds),
                                   make_span(sol.problem_ptr->variable_upper_bounds));
  }
  CUOPT_LOG_DEBUG(
    "Bounds propagation rounding end: ii constraint count first buffer %d, second buffer %d",
    multi_probe.infeas_constraints_count_0,
    multi_probe.infeas_constraints_count_1);
  cuopt_assert(sol.test_number_all_integer(), "All integers must be rounded");
  expand_device_copy(orig_sol.assignment, sol.assignment, sol.handle_ptr->get_stream());
  cuopt_func_call(orig_sol.test_variable_bounds());
  // if the constraint is not ii, run LP
  if ((multi_probe.infeas_constraints_count_0 == 0 ||
       multi_probe.infeas_constraints_count_1 == 0) &&
      !timeout_happened) {
    relaxed_lp_settings_t lp_settings;
    lp_settings.time_limit            = lp_run_time_after_feasible;
    lp_settings.tolerance             = orig_sol.problem_ptr->tolerances.absolute_tolerance;
    lp_settings.save_state            = false;
    lp_settings.return_first_feasible = true;
    run_lp_with_vars_fixed(*orig_sol.problem_ptr,
                           orig_sol,
                           orig_sol.problem_ptr->integer_indices,
                           lp_settings,
                           static_cast<bound_presolve_t<i_t, f_t>*>(nullptr));
  }
  bool res_feasible = orig_sol.compute_feasibility();
  orig_sol.handle_ptr->sync_stream();
  return res_feasible;
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::apply_round(
  solution_t<i_t, f_t>& sol,
  f_t lp_run_time_after_feasible,
  timer_t& timer,
  std::optional<std::reference_wrapper<probing_config_t<i_t, f_t>>> probing_config)
{
  raft::common::nvtx::range fun_scope("constraint prop round");

  // this is second timer that can continue but without recovery mode
  const f_t max_time_for_bounds_prop = 5.;
  max_timer                          = timer_t{max_time_for_bounds_prop};
  if (check_brute_force_rounding(sol)) { return true; }
  recovery_mode      = false;
  rounding_ii        = false;
  n_iter_in_recovery = 0;
  sol.compute_constraints();
  problem_t<i_t, f_t> p(*sol.problem_ptr);
  temp_sol.resize_copy(sol);
  temp_sol.problem_ptr       = &p;
  f_t bounds_prop_start_time = max_timer.remaining_time();
  cuopt_func_call(temp_sol.test_variable_bounds(false));
  bool sol_found = find_integer(temp_sol, sol, lp_run_time_after_feasible, timer, probing_config);
  f_t bounds_prop_end_time = max_timer.remaining_time();
  repair_stats.total_time_spent_on_bounds_prop += bounds_prop_start_time - bounds_prop_end_time;

  CUOPT_LOG_DEBUG(
    "repair_success %lu repair_attempts %lu intermediate_repair_success %lu total_repair_loops %lu "
    "total_time_spent_on_repair %f total_time_spent_bounds_prop_after_repair %f "
    "total_time_spent_on_bounds_prop %f",
    repair_stats.repair_success,
    repair_stats.repair_attempts,
    repair_stats.intermediate_repair_success,
    repair_stats.total_repair_loops,
    repair_stats.total_time_spent_on_repair,
    repair_stats.total_time_spent_bounds_prop_after_repair,
    repair_stats.total_time_spent_on_bounds_prop);
  if (!sol_found) {
    sol.compute_feasibility();
    return false;
  }
  return sol.compute_feasibility();
}

template <typename i_t, typename f_t>
std::tuple<f_t, f_t, f_t> constraint_prop_t<i_t, f_t>::probing_values(
  const solution_t<i_t, f_t>& sol, const solution_t<i_t, f_t>& orig_sol, i_t idx)
{
  auto v_lb    = multi_probe.host_lb[idx];
  auto v_ub    = multi_probe.host_ub[idx];
  auto var_val = curr_host_assignment[idx];

  const f_t int_tol  = sol.problem_ptr->tolerances.integrality_tolerance;
  auto eps           = int_tol;
  auto within_bounds = (v_lb - eps <= var_val) && (var_val <= v_ub + eps);
  // if it is a collapsed var, return immediately one of the bounds
  if (orig_sol.problem_ptr->integer_equal(v_lb, v_ub)) {
    return std::make_tuple(v_lb, var_val, v_lb);
  }
  if (within_bounds) {
    // the value might have been brought within the bounds when it was out of bounds
    f_t first_round_val = round_nearest(var_val, v_lb, v_ub, int_tol, rng);
    f_t second_round_val;
    auto v_f = std::floor(first_round_val - int_tol);
    auto v_c = std::ceil(first_round_val + int_tol);
    if (first_round_val - var_val >= 0) {
      second_round_val = v_f;
      bool floor_within_bounds =
        (v_lb - eps <= second_round_val) && (second_round_val <= v_ub + eps);
      if (!floor_within_bounds) { second_round_val = v_c; }
    } else {
      second_round_val = v_c;
      bool ceil_within_bounds =
        (v_lb - eps <= second_round_val) && (second_round_val <= v_ub + eps);
      if (!ceil_within_bounds) { second_round_val = v_f; }
    }

    cuopt_assert(v_lb <= first_round_val && first_round_val <= v_ub, "probing value out of bounds");
    cuopt_assert(v_lb <= second_round_val && second_round_val <= v_ub,
                 "probing value out of bounds");
    return std::make_tuple(first_round_val, var_val, second_round_val);
  } else {
    auto orig_v_lb =
      orig_sol.problem_ptr->variable_lower_bounds.element(idx, sol.handle_ptr->get_stream());
    auto orig_v_ub =
      orig_sol.problem_ptr->variable_upper_bounds.element(idx, sol.handle_ptr->get_stream());
    cuopt_assert(v_lb >= orig_v_lb, "Current lb should be greater than original lb");
    cuopt_assert(v_ub <= orig_v_ub, "Current ub should be smaller than original ub");
    v_lb = std::max(v_lb, orig_v_lb);
    v_ub = std::min(v_ub, orig_v_ub);
    // the bounds might cross, so correct them here
    if (v_lb > v_ub) {
      v_lb = orig_v_lb;
      v_ub = orig_v_lb;
    }
    auto v_f = std::floor(var_val);
    auto v_c = std::ceil(var_val);
    if (std::ceil(v_lb) == std::floor(v_ub)) {
      return std::make_tuple(std::ceil(v_lb), var_val, std::floor(v_ub));
    } else if (v_f < std::ceil(v_lb)) {
      v_f = std::ceil(v_lb);
      v_c = v_f + 1;
    } else if (v_c > std::floor(v_ub)) {
      v_c = std::floor(v_ub);
      v_f = v_c - 1;
    }
    cuopt_assert(orig_v_lb <= v_f && v_f <= orig_v_ub, "probing value out of bounds");
    cuopt_assert(orig_v_lb <= v_c && v_c <= orig_v_ub, "probing value out of bounds");
    return std::make_tuple(v_f, var_val, v_c);
  }
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::handle_fixed_vars(
  solution_t<i_t, f_t>& sol,
  problem_t<i_t, f_t>* original_problem,
  const std::tuple<std::vector<i_t>, std::vector<f_t>, std::vector<f_t>>& var_probe_vals,
  size_t* set_count_ptr,
  rmm::device_uvector<i_t>& unset_vars)
{
  auto set_count    = *set_count_ptr;
  const f_t int_tol = sol.problem_ptr->tolerances.integrality_tolerance;
  // which other variables were affected?
  auto iter = thrust::stable_partition(
    sol.handle_ptr->get_thrust_policy(),
    unset_vars.begin() + set_count,
    unset_vars.end(),
    is_bound_fixed_t<i_t, f_t>{int_tol,
                               make_span(sol.problem_ptr->variable_lower_bounds),
                               make_span(sol.problem_ptr->variable_upper_bounds),
                               make_span(original_problem->variable_lower_bounds),
                               make_span(original_problem->variable_upper_bounds),
                               make_span(sol.assignment)});
  i_t n_fixed_vars = (iter - (unset_vars.begin() + set_count));
  cuopt_assert(n_fixed_vars >= std::get<0>(var_probe_vals).size(),
               "Error in number of vars fixed!");
  set_count += n_fixed_vars;
  CUOPT_LOG_TRACE("Set var count increased from %d to %d", *set_count_ptr, set_count);
  *set_count_ptr = set_count;
  return multi_probe.infeas_constraints_count_0 == 0 || multi_probe.infeas_constraints_count_1 == 0;
}

template <typename i_t, typename f_t>
bool constraint_prop_t<i_t, f_t>::probe(
  solution_t<i_t, f_t>& sol,
  problem_t<i_t, f_t>* original_problem,
  const std::tuple<std::vector<i_t>, std::vector<f_t>, std::vector<f_t>>& var_probe_vals,
  size_t* set_count_ptr,
  rmm::device_uvector<i_t>& unset_vars,
  std::optional<std::reference_wrapper<probing_config_t<i_t, f_t>>> probing_config)
{
  const bool use_host_bounds      = true;
  multi_probe.settings.time_limit = max_timer.remaining_time();
  multi_probe.solve(*sol.problem_ptr, var_probe_vals, use_host_bounds);
  // if we are ii at this point, backtrack the number of variables we have set in this given
  // interval then start setting one by one
  // if we determined that the rounding is ii then don't do any recovery and finish ronuding
  // quickly
  bool first_bounds_update_ii  = multi_probe.infeas_constraints_count_0 > 0;
  bool second_bounds_update_ii = multi_probe.infeas_constraints_count_1 > 0;
  // if we are on single rounding mode, direcly mark as ii so we can run repair
  if (!rounding_ii && single_rounding_only && (first_bounds_update_ii && second_bounds_update_ii)) {
    rounding_ii = true;
    return false;
  }
  if (!recovery_mode && !rounding_ii && (first_bounds_update_ii && second_bounds_update_ii) &&
      bounds_prop_interval != 1) {
    CUOPT_LOG_DEBUG(
      "Activating recovery mode in bounds prop: bounds_prop_interval %d n_ii cstr %d %d",
      bounds_prop_interval,
      multi_probe.infeas_constraints_count_0,
      multi_probe.infeas_constraints_count_1);
    // do backtracking
    recovery_mode                          = true;
    multi_probe.infeas_constraints_count_0 = 0;
    multi_probe.infeas_constraints_count_1 = 0;
    n_iter_in_recovery                     = 0;
    return false;
  }
  selected_update = 0;
  if (first_bounds_update_ii) { selected_update = 1; }
  // if we are doing single rounding
  if (probing_config.has_value() && probing_config.value().get().use_balanced_probing) {
    cuopt_assert(std::get<0>(var_probe_vals).size() == 1,
                 "Balanced probing must be used with single rounding");
    i_t var_idx = std::get<0>(var_probe_vals)[0];
    f_t value_chosen =
      selected_update ? std::get<1>(var_probe_vals)[0] : std::get<2>(var_probe_vals)[0];
    if (value_chosen == probing_config.value().get().probing_values[var_idx].first) {
      probing_config.value().get().n_of_fixed_from_first++;
    } else {
      probing_config.value().get().n_of_fixed_from_second++;
    }
    CUOPT_LOG_TRACE("Balanced probing: n_of_fixed_from_first %d n_of_fixed_from_second %d",
                    probing_config.value().get().n_of_fixed_from_first,
                    probing_config.value().get().n_of_fixed_from_second);
  }
  multi_probe.set_updated_bounds(*sol.problem_ptr, selected_update, sol.handle_ptr);
  return handle_fixed_vars(sol, original_problem, var_probe_vals, set_count_ptr, unset_vars);
}

#if MIP_INSTANTIATE_FLOAT
template class constraint_prop_t<int, float>;
#endif

#if MIP_INSTANTIATE_DOUBLE
template class constraint_prop_t<int, double>;
#endif

}  // namespace cuopt::linear_programming::detail
