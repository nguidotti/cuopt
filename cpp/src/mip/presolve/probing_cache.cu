#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "probing_cache.cuh"

#include <mip/mip_constants.hpp>
#include <mip/presolve/multi_probe.cuh>
#include <mip/utils.cuh>

#include <omp.h>
#include <thrust/sort.h>
#include <utilities/copy_helpers.hpp>
#include <utilities/timer.hpp>

namespace cuopt::linear_programming::detail {

template <typename i_t, typename f_t>
i_t probing_cache_t<i_t, f_t>::check_number_of_conflicting_vars(
  const std::vector<f_t>& host_lb,
  const std::vector<f_t>& host_ub,
  const cache_entry_t<i_t, f_t>& cache_entry,
  f_t integrality_tolerance,
  const std::vector<i_t>& reverse_original_ids)
{
  i_t n_conflicting_var = 0;
  for (const auto& [var_idx, bound] : cache_entry.var_to_cached_bound_map) {
    i_t var_idx_in_current_problem = reverse_original_ids[var_idx];
    // -1 means that variable was fixed and doesn't exists in the current problem
    if (var_idx_in_current_problem == -1) { continue; }
    if (host_lb[var_idx_in_current_problem] - integrality_tolerance > bound.ub ||
        host_ub[var_idx_in_current_problem] < bound.lb - integrality_tolerance) {
      ++n_conflicting_var;
    }
  }
  return n_conflicting_var;
}

template <typename i_t, typename f_t>
void probing_cache_t<i_t, f_t>::update_bounds_with_selected(
  std::vector<f_t>& host_lb,
  std::vector<f_t>& host_ub,
  const cache_entry_t<i_t, f_t>& cache_entry,
  const std::vector<i_t>& reverse_original_ids)
{
  i_t n_bounds_updated = 0;
  for (const auto& [var_idx, bound] : cache_entry.var_to_cached_bound_map) {
    i_t var_idx_in_current_problem = reverse_original_ids[var_idx];
    // -1 means that variable was fixed and doesn't exists in the current problem
    if (var_idx_in_current_problem == -1) { continue; }
    if (host_lb[var_idx_in_current_problem] < bound.lb) {
      host_lb[var_idx_in_current_problem] = bound.lb;
      n_bounds_updated++;
    }
    if (host_ub[var_idx_in_current_problem] > bound.ub) {
      host_ub[var_idx_in_current_problem] = bound.ub;
      n_bounds_updated++;
    }
  }
}

template <typename i_t, typename f_t>
f_t probing_cache_t<i_t, f_t>::get_least_conflicting_rounding(problem_t<i_t, f_t>& problem,
                                                              std::vector<f_t>& host_lb,
                                                              std::vector<f_t>& host_ub,
                                                              i_t var_id_on_problem,
                                                              f_t first_probe,
                                                              f_t second_probe,
                                                              f_t integrality_tolerance)
{
  // get the var id where the probing cache was computed
  i_t var_id      = problem.original_ids[var_id_on_problem];
  auto& cache_row = probing_cache[var_id];

  i_t hit_interval_for_first_probe  = -1;
  i_t hit_interval_for_second_probe = -1;
  for (i_t i = 0; i < 2; ++i) {
    auto& cache_entry = cache_row[i];
    // if no implied bounds found go to next interval
    if (cache_entry.var_to_cached_bound_map.empty()) { continue; }
    cache_entry.val_interval.fill_cache_hits(
      i, first_probe, second_probe, hit_interval_for_first_probe, hit_interval_for_second_probe);
  }
  i_t n_conflicting_vars = 0;
  // first probe found some interval
  if (hit_interval_for_first_probe != -1) {
    n_conflicting_vars = check_number_of_conflicting_vars(host_lb,
                                                          host_ub,
                                                          cache_row[hit_interval_for_first_probe],
                                                          integrality_tolerance,
                                                          problem.reverse_original_ids);
    if (n_conflicting_vars == 0) {
      CUOPT_LOG_TRACE("No conflicting vars, returning first probe");
      update_bounds_with_selected(
        host_lb, host_ub, cache_row[hit_interval_for_first_probe], problem.reverse_original_ids);
      return first_probe;
    }
  }
  // if the interval is still -1, it means this probing doesn't have any implied bounds
  else {
    CUOPT_LOG_TRACE("No implied bounds on first probe, returning first probe");
    return first_probe;
  }
  CUOPT_LOG_TRACE("Conflicting vars %d found in first probing, searching least conflicting!",
                  n_conflicting_vars);
  // check for the other side, if it the interval includes second_probe return that, if not return
  // cutoff point second probe has a hit but it is not the same as first probe
  i_t other_interval_idx = 1 - hit_interval_for_first_probe;
  i_t n_conflicting_vars_other_probe =
    check_number_of_conflicting_vars(host_lb,
                                     host_ub,
                                     cache_row[other_interval_idx],
                                     integrality_tolerance,
                                     problem.reverse_original_ids);

  if (n_conflicting_vars_other_probe < n_conflicting_vars) {
    CUOPT_LOG_DEBUG(
      "For probing var %d with value %f better conflicting vars found %d in the other probing "
      "region (cache interval)!",
      var_id,
      first_probe,
      n_conflicting_vars_other_probe);
    update_bounds_with_selected(
      host_lb, host_ub, cache_row[other_interval_idx], problem.reverse_original_ids);
    if (other_interval_idx == hit_interval_for_second_probe) {
      CUOPT_LOG_DEBUG("Better value on second probe val %f", second_probe);
      return second_probe;
    } else {
      CUOPT_LOG_DEBUG("Better value on other interval cutoff %f",
                      cache_row[other_interval_idx].val_interval.val);
      return cache_row[other_interval_idx].val_interval.val;
    }
  }
  update_bounds_with_selected(
    host_lb, host_ub, cache_row[hit_interval_for_first_probe], problem.reverse_original_ids);
  return first_probe;
}

template <typename i_t, typename f_t>
bool probing_cache_t<i_t, f_t>::contains(problem_t<i_t, f_t>& problem, i_t var_id)
{
  return probing_cache.count(problem.original_ids[var_id]) > 0;
}

template <typename i_t, typename f_t>
void inline insert_current_probing_to_cache(i_t var_idx,
                                            const val_interval_t<i_t, f_t>& probe_val,
                                            bound_presolve_t<i_t, f_t>& bound_presolve,
                                            const std::vector<f_t>& original_lb,
                                            const std::vector<f_t>& original_ub,
                                            const std::vector<f_t>& modified_lb,
                                            const std::vector<f_t>& modified_ub,
                                            const std::vector<i_t>& h_integer_indices,
                                            std::atomic<size_t>& n_implied_singletons)
{
  f_t int_tol = bound_presolve.context.settings.tolerances.integrality_tolerance;

  cache_entry_t<i_t, f_t> cache_item;
  cache_item.val_interval = probe_val;
  for (auto impacted_var_idx : h_integer_indices) {
    if (original_lb[impacted_var_idx] != modified_lb[impacted_var_idx] ||
        original_ub[impacted_var_idx] != modified_ub[impacted_var_idx]) {
      if (integer_equal<f_t>(
            modified_lb[impacted_var_idx], modified_ub[impacted_var_idx], int_tol)) {
        ++n_implied_singletons;
      }
      cuopt_assert(modified_lb[impacted_var_idx] >= original_lb[impacted_var_idx],
                   "Lower bound must be greater than or equal to original lower bound");
      cuopt_assert(modified_ub[impacted_var_idx] <= original_ub[impacted_var_idx],
                   "Upper bound must be less than or equal to original upper bound");
      cached_bound_t<f_t> new_bound{modified_lb[impacted_var_idx], modified_ub[impacted_var_idx]};
      cache_item.var_to_cached_bound_map.insert({impacted_var_idx, new_bound});
    }
  }
  {
    std::lock_guard<std::mutex> lock(bound_presolve.probing_cache.probing_cache_mutex);
    if (!bound_presolve.probing_cache.probing_cache.count(var_idx) > 0) {
      std::array<cache_entry_t<i_t, f_t>, 2> entries_per_var;
      entries_per_var[0] = cache_item;
      bound_presolve.probing_cache.probing_cache.insert({var_idx, entries_per_var});
    } else {
      bound_presolve.probing_cache.probing_cache[var_idx][1] = cache_item;
    }
  }
}

template <typename i_t, typename f_t>
__global__ void compute_min_slack_per_var(typename problem_t<i_t, f_t>::view_t pb,
                                          raft::device_span<f_t> min_activity,
                                          raft::device_span<f_t> max_activity,
                                          raft::device_span<f_t> var_slack,
                                          raft::device_span<bool> different_coefficient,
                                          raft::device_span<f_t> max_excess_per_var,
                                          raft::device_span<i_t> max_n_violated_per_constraint)
{
  i_t var_idx           = pb.integer_indices[blockIdx.x];
  i_t var_offset        = pb.reverse_offsets[var_idx];
  i_t var_degree        = pb.reverse_offsets[var_idx + 1] - var_offset;
  f_t th_var_unit_slack = std::numeric_limits<f_t>::max();
  f_t lb                = pb.variable_lower_bounds[var_idx];
  f_t ub                = pb.variable_upper_bounds[var_idx];
  f_t first_coeff       = pb.reverse_coefficients[var_offset];
  bool different_coeff  = false;
  for (i_t i = threadIdx.x; i < var_degree; i += blockDim.x) {
    auto a = pb.reverse_coefficients[var_offset + i];
    if (std::signbit(a) != std::signbit(first_coeff)) { different_coeff = true; }
    auto cnst_idx = pb.reverse_constraints[var_offset + i];
    auto min_a    = min_activity[cnst_idx];
    auto max_a    = max_activity[cnst_idx];
    auto cnstr_ub = pb.constraint_upper_bounds[cnst_idx];
    auto cnstr_lb = pb.constraint_lower_bounds[cnst_idx];
    min_a -= (a < 0) ? a * ub : a * lb;
    auto delta_min_act = cnstr_ub - min_a;
    th_var_unit_slack  = min(th_var_unit_slack, (delta_min_act / a));
    max_a -= (a > 0) ? a * ub : a * lb;
    auto delta_max_act = cnstr_lb - max_a;
    th_var_unit_slack  = min(th_var_unit_slack, (delta_max_act / a));
    // if (var_idx == 0) {
    //   printf("\ncmp_min_slack cnst %d\n diff %f %f\n cnstr_ub %f min_a %f delta_min %f\n cnstr_lb
    //   %f max_a %f delta_max %f\n", cnst_idx,
    //       (a < 0) ? a * ub : a * lb,
    //       (a > 0) ? a * ub : a * lb,
    //       cnstr_ub, min_a, delta_min_act,
    //       cnstr_lb, max_a, delta_max_act);
    // }
  }
  __shared__ f_t shmem[raft::WarpSize];
  f_t block_var_unit_slack = raft::blockReduce(th_var_unit_slack, (char*)shmem, raft::min_op{});
  __syncthreads();
  i_t block_different_coeff = raft::blockReduce((i_t)different_coeff, (char*)shmem);
  if (threadIdx.x == 0) {
    var_slack[blockIdx.x]             = block_var_unit_slack;
    different_coefficient[blockIdx.x] = block_different_coeff > 0;
  }
  __syncthreads();
  // return vars that will have no implied bounds
  if (!different_coefficient[blockIdx.x]) { return; }
  // for each variable that appers with negated coeffs in different cosntraints
  // check whether flipping the var from lb to ub in constraints with positive coefficient
  // violates the constraint. we do it for 4 situation that can be inferred.
  i_t th_n_of_excess = 0;
  f_t th_max_excess  = 0.;
  for (i_t i = threadIdx.x; i < var_degree; i += blockDim.x) {
    auto a        = pb.reverse_coefficients[var_offset + i];
    auto cnst_idx = pb.reverse_constraints[var_offset + i];
    auto min_a    = min_activity[cnst_idx];
    auto max_a    = max_activity[cnst_idx];
    auto cnstr_ub = pb.constraint_upper_bounds[cnst_idx];
    auto cnstr_lb = pb.constraint_lower_bounds[cnst_idx];
    min_a -= (a < 0) ? a * ub : a * lb;
    f_t var_max_act = (a > 0) ? a * ub : a * lb;
    f_t excess      = max(0., min_a + var_max_act - cnstr_ub);
    if (excess > 0) {
      th_max_excess = max(th_max_excess, excess);
      th_n_of_excess++;
    }
    // now add max activity of this var to see the excess
    max_a -= (a > 0) ? a * ub : a * lb;
    f_t var_min_act = (a < 0) ? a * ub : a * lb;
    excess          = max(0., cnstr_lb - (max_a + var_min_act));
    if (excess > 0) {
      th_max_excess = max(th_max_excess, excess);
      th_n_of_excess++;
    }
  }
  f_t max_excess = raft::blockReduce(th_max_excess, (char*)shmem, raft::max_op{});
  __syncthreads();
  i_t total_excessed_cstr = raft::blockReduce(th_n_of_excess, (char*)shmem);
  if (threadIdx.x == 0) {
    max_excess_per_var[blockIdx.x]            = max_excess;
    max_n_violated_per_constraint[blockIdx.x] = total_excessed_cstr;
  }
}

// computes variables that appear in multiple constraints with different signs
// which means that min activity contribution in one constraint will not be valid in another
// constraint we will sort them by the violation rooted from the conflicting bounds. an example: lb:
// 0 ub: 5 cstr_1 coeff : -1  cstr_2 coeff: 1 min activity val in cstr_1 is 5 and 0 in cstr_2, they
// cannot happen at the same time we extract those variables and then sort it by the sum of
// excesses(or slack) in all constraints by setting to lb and ub
template <typename i_t, typename f_t>
inline std::vector<i_t> compute_prioritized_integer_indices(
  bound_presolve_t<i_t, f_t>& bound_presolve, problem_t<i_t, f_t>& problem)
{
  // sort the variables according to the min slack they have across constraints
  // we also need to consider the variable range
  // the priority is computed as the var_range * min_slack
  // min_slack is computed as var_range*coefficient/(b - min_act)
  rmm::device_uvector<f_t> min_slack_per_var(problem.n_integer_vars,
                                             problem.handle_ptr->get_stream());
  rmm::device_uvector<i_t> priority_indices(problem.integer_indices,
                                            problem.handle_ptr->get_stream());
  rmm::device_uvector<bool> different_coefficient(problem.n_integer_vars,
                                                  problem.handle_ptr->get_stream());
  rmm::device_uvector<f_t> max_excess_per_var(problem.n_integer_vars,
                                              problem.handle_ptr->get_stream());
  rmm::device_uvector<i_t> max_n_violated_per_constraint(problem.n_integer_vars,
                                                         problem.handle_ptr->get_stream());
  thrust::fill(problem.handle_ptr->get_thrust_policy(),
               min_slack_per_var.begin(),
               min_slack_per_var.end(),
               std::numeric_limits<f_t>::max());

  thrust::fill(problem.handle_ptr->get_thrust_policy(),
               max_excess_per_var.begin(),
               max_excess_per_var.end(),
               0);
  thrust::fill(problem.handle_ptr->get_thrust_policy(),
               max_n_violated_per_constraint.begin(),
               max_n_violated_per_constraint.end(),
               0);
  // compute min and max activity first
  bound_presolve.calculate_activity_on_problem_bounds(problem);
  bool res = bound_presolve.calculate_infeasible_redundant_constraints(problem);
  cuopt_assert(res, "The activity computation must be feasible during probing cache!");
  CUOPT_LOG_DEBUG("prioritized integer_indices n_integer_vars %d", problem.n_integer_vars);
  // compute the min var slack
  compute_min_slack_per_var<i_t, f_t>
    <<<problem.n_integer_vars, 128, 0, problem.handle_ptr->get_stream()>>>(
      problem.view(),
      make_span(bound_presolve.upd.min_activity),
      make_span(bound_presolve.upd.max_activity),
      make_span(min_slack_per_var),
      make_span(different_coefficient),
      make_span(max_excess_per_var),
      make_span(max_n_violated_per_constraint));
  auto iterator = thrust::make_zip_iterator(thrust::make_tuple(
    max_n_violated_per_constraint.begin(), max_excess_per_var.begin(), min_slack_per_var.begin()));
  // sort the vars
  thrust::sort_by_key(problem.handle_ptr->get_thrust_policy(),
                      iterator,
                      iterator + problem.n_integer_vars,
                      priority_indices.begin(),
                      [] __device__(auto tuple1, auto tuple2) {
                        // if both are zero, i.e. no excess, sort it by min slack
                        if (thrust::get<0>(tuple1) == 0 && thrust::get<0>(tuple2) == 0) {
                          return thrust::get<2>(tuple1) < thrust::get<2>(tuple2);
                        } else if (thrust::get<0>(tuple1) > thrust::get<0>(tuple2)) {
                          return true;
                        } else if (thrust::get<0>(tuple1) == thrust::get<0>(tuple2)) {
                          return thrust::get<1>(tuple1) > thrust::get<1>(tuple2);
                        }
                        return false;
                      });
  auto h_priority_indices = host_copy(priority_indices);
  problem.handle_ptr->sync_stream();
  return h_priority_indices;
}

template <typename i_t, typename f_t>
void compute_cache_for_var(i_t var_idx,
                           bound_presolve_t<i_t, f_t>& bound_presolve,
                           problem_t<i_t, f_t>& problem,
                           multi_probe_t<i_t, f_t>& multi_probe_presolve,
                           const std::vector<f_t>& h_var_lower_bounds,
                           const std::vector<f_t>& h_var_upper_bounds,
                           const std::vector<i_t>& h_integer_indices,
                           std::atomic<size_t>& n_of_implied_singletons,
                           std::atomic<size_t>& n_of_cached_probings,
                           i_t device_id)
{
  RAFT_CUDA_TRY(hipSetDevice(device_id));
  // test if we need per thread handle
  raft::handle_t handle{};
  std::vector<f_t> h_improved_lower_bounds(h_var_lower_bounds.size());
  std::vector<f_t> h_improved_upper_bounds(h_var_upper_bounds.size());
  std::pair<val_interval_t<i_t, f_t>, val_interval_t<i_t, f_t>> probe_vals;
  f_t lb = h_var_lower_bounds[var_idx];
  f_t ub = h_var_upper_bounds[var_idx];
  for (i_t i = 0; i < 2; ++i) {
    auto& probe_val = i == 0 ? probe_vals.first : probe_vals.second;
    // if binary, probe both values
    if (problem.integer_equal(ub - lb, 1.)) {
      probe_val.interval_type = interval_type_t::EQUALS;
      probe_val.val           = i == 0 ? lb : ub;
    }
    // if both sides are finite, probe on lower half and upper half
    else if (isfinite(lb) && isfinite(ub)) {
      probe_val.interval_type = i == 0 ? interval_type_t::LEQ : interval_type_t::GEQ;
      f_t middle              = floor((lb + ub) / 2);
      probe_val.val           = i == 0 ? middle : middle + 1;
    }
    // if only lower bound is finite, probe on lb and >lb
    else if (isfinite(lb)) {
      probe_val.interval_type = i == 0 ? interval_type_t::EQUALS : interval_type_t::GEQ;
      probe_val.val           = i == 0 ? lb : lb + 1;
    }
    // if only upper bound is finite, probe on ub and <ub
    else {
      probe_val.interval_type = i == 0 ? interval_type_t::EQUALS : interval_type_t::LEQ;
      probe_val.val           = i == 0 ? ub : ub - 1;
    }
  }
  std::tuple<i_t, std::pair<f_t, f_t>, std::pair<f_t, f_t>> var_interval_vals;
  std::get<0>(var_interval_vals) = var_idx;
  for (i_t i = 0; i < 2; ++i) {
    auto& probe_val = i == 0 ? probe_vals.first : probe_vals.second;
    // first(index 1) item of tuple is the first interval, the second is the second interval
    auto& bounds = i == 0 ? std::get<1>(var_interval_vals) : std::get<2>(var_interval_vals);
    // now solve bounds presolve for the value or the interval
    // if the type is equals, just set the value and solve the bounds presolve
    if (probe_val.interval_type == interval_type_t::EQUALS) {
      bounds.first  = probe_val.val;
      bounds.second = probe_val.val;
    }
    // if it is an interval change the variable bound and solve
    else {
      if (probe_val.interval_type == interval_type_t::LEQ) {
        bounds.first  = lb;
        bounds.second = probe_val.val;
      } else {
        bounds.first  = probe_val.val;
        bounds.second = ub;
      }
    }
  }
  auto bounds_presolve_result =
    multi_probe_presolve.solve_for_interval(problem, var_interval_vals, &handle);
  if (bounds_presolve_result != termination_criterion_t::NO_UPDATE) {
    CUOPT_LOG_TRACE("Adding cached bounds for var %d", var_idx);
  }
  for (i_t i = 0; i < 2; ++i) {
    // this only tracs the number of variables that have cached bounds
    n_of_cached_probings++;
    // save the impacted bounds
    if (bounds_presolve_result != termination_criterion_t::NO_UPDATE) {
      const auto& probe_val = i == 0 ? probe_vals.first : probe_vals.second;
      auto& d_lb = i == 0 ? multi_probe_presolve.upd_0.lb : multi_probe_presolve.upd_1.lb;
      auto& d_ub = i == 0 ? multi_probe_presolve.upd_0.ub : multi_probe_presolve.upd_1.ub;
      raft::copy(h_improved_lower_bounds.data(),
                 d_lb.data(),
                 h_improved_lower_bounds.size(),
                 handle.get_stream());
      raft::copy(h_improved_upper_bounds.data(),
                 d_ub.data(),
                 h_improved_upper_bounds.size(),
                 handle.get_stream());
      insert_current_probing_to_cache(var_idx,
                                      probe_val,
                                      bound_presolve,
                                      h_var_lower_bounds,
                                      h_var_upper_bounds,
                                      h_improved_lower_bounds,
                                      h_improved_upper_bounds,
                                      h_integer_indices,
                                      n_of_implied_singletons);
    }
  }
  handle.sync_stream();
}

template <typename i_t, typename f_t>
void compute_probing_cache(bound_presolve_t<i_t, f_t>& bound_presolve,
                           problem_t<i_t, f_t>& problem,
                           timer_t timer)
{
  // we dont want to compute the probing cache for all variables for time and computation resources
  auto priority_indices = compute_prioritized_integer_indices(bound_presolve, problem);
  CUOPT_LOG_DEBUG("Computing probing cache");
  auto h_integer_indices        = host_copy(problem.integer_indices);
  const auto h_var_upper_bounds = host_copy(problem.variable_upper_bounds);
  const auto h_var_lower_bounds = host_copy(problem.variable_lower_bounds);
  // TODO adjust the iteration limit depending on the total time limit and time it takes for single
  // var
  bound_presolve.settings.iteration_limit = 50;
  bound_presolve.settings.time_limit      = timer.remaining_time();

  // Set the number of threads
  const size_t max_threads = 10;
  omp_set_num_threads(max_threads);

  // Create a vector of multi_probe_t objects
  std::vector<multi_probe_t<i_t, f_t>> multi_probe_presolve_pool;

  // Initialize multi_probe_presolve_pool
  for (size_t i = 0; i < max_threads; i++) {
    multi_probe_presolve_pool.emplace_back(bound_presolve.context);
    multi_probe_presolve_pool[i].resize(problem);
    multi_probe_presolve_pool[i].compute_stats = false;
  }

  // Atomic variables for tracking progress
  std::atomic<size_t> n_of_implied_singletons(0);
  std::atomic<size_t> n_of_cached_probings(0);

// Main parallel loop
#pragma omp parallel
  {
#pragma omp for schedule(static, 4)
    for (auto var_idx : priority_indices) {
      if (timer.check_time_limit()) { continue; }

      int thread_idx = omp_get_thread_num();
      CUOPT_LOG_TRACE("Computing probing cache for var %d on thread %d", var_idx, thread_idx);

      auto& multi_probe_presolve = multi_probe_presolve_pool[thread_idx];

      compute_cache_for_var<i_t, f_t>(var_idx,
                                      bound_presolve,
                                      problem,
                                      multi_probe_presolve,
                                      h_var_lower_bounds,
                                      h_var_upper_bounds,
                                      h_integer_indices,
                                      n_of_implied_singletons,
                                      n_of_cached_probings,
                                      problem.handle_ptr->get_device());
    }
  }

  CUOPT_LOG_DEBUG("Total number of cached probings %lu number of implied singletons %lu",
                  n_of_cached_probings.load(),
                  n_of_implied_singletons.load());
  // restore the settings
  bound_presolve.settings = {};
}

#define INSTANTIATE(F_TYPE)                                                                        \
  template void compute_probing_cache<int, F_TYPE>(bound_presolve_t<int, F_TYPE> & bound_presolve, \
                                                   problem_t<int, F_TYPE> & problem,               \
                                                   timer_t timer);                                 \
  template class probing_cache_t<int, F_TYPE>;

#if MIP_INSTANTIATE_FLOAT
INSTANTIATE(float)
#endif

#if MIP_INSTANTIATE_DOUBLE
INSTANTIATE(double)
#endif

#undef INSTANTIATE

}  // namespace cuopt::linear_programming::detail
