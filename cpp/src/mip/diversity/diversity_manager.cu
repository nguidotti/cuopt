#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <mip/mip_constants.hpp>
#include <mip/problem/problem_helpers.cuh>
#include "diversity_manager.cuh"

#include <mip/presolve/probing_cache.cuh>
#include <mip/presolve/trivial_presolve.cuh>

#include <raft/common/nvtx.hpp>
#include <utilities/scope_guard.hpp>

#include "hip/hip_runtime_api.h"

namespace cuopt::linear_programming::detail {

constexpr int max_var_diff                    = 256;
constexpr size_t max_solutions                = 32;
constexpr double initial_infeasibility_weight = 1000.;
constexpr double default_time_limit           = 10.;
constexpr int initial_island_size             = 3;
constexpr int maximum_island_size             = 8;
constexpr bool use_avg_diversity              = false;

template <typename i_t, typename f_t>
diversity_manager_t<i_t, f_t>::diversity_manager_t(mip_solver_context_t<i_t, f_t>& context_)
  : problem_ptr(context.problem_ptr),
    context(context_),
    population("population",
               context,
               max_var_diff,
               max_solutions,
               initial_infeasibility_weight * context.problem_ptr->n_constraints),
    lp_optimal_solution(context.problem_ptr->n_variables,
                        context.problem_ptr->handle_ptr->get_stream()),
    ls(context, lp_optimal_solution),
    timer(default_time_limit),
    bound_prop_recombiner(context,
                          context.problem_ptr->n_variables,
                          ls.constraint_prop,
                          context.problem_ptr->handle_ptr),
    fp_recombiner(
      context, context.problem_ptr->n_variables, ls.fp, context.problem_ptr->handle_ptr),
    line_segment_recombiner(context,
                            context.problem_ptr->n_variables,
                            ls.line_segment_search,
                            context.problem_ptr->handle_ptr),
    rng(cuopt::seed_generator::get_seed()),
    stats(context.stats)
{
}

// There should be at least 3 solutions in the population
template <typename i_t, typename f_t>
bool diversity_manager_t<i_t, f_t>::regenerate_solutions()
{
  f_t time_limit     = 5;
  i_t counter        = 0;
  const i_t min_size = 2;
  while (population.current_size() <= min_size && (current_step == 0 || counter < 5)) {
    CUOPT_LOG_DEBUG("Trying to regenerate solution, pop size %d\n", population.current_size());
    time_limit = std::min(time_limit, timer.remaining_time());
    ls.fj.randomize_weights(problem_ptr->handle_ptr);
    population.add_solution(generate_solution(time_limit));
    if (timer.check_time_limit()) { return false; }
    // increase the time limit as we couldn't add a valid solution
    time_limit += 5;
    counter++;
  }
  ++current_step;
  // if there is at least two sols still return true
  return population.current_size() >= min_size;
}

// There should be at least 3 solutions in the population
template <typename i_t, typename f_t>
std::vector<solution_t<i_t, f_t>> diversity_manager_t<i_t, f_t>::generate_more_solutions()
{
  std::vector<solution_t<i_t, f_t>> solutions;
  timer_t total_time_to_generate = timer_t(timer.remaining_time() / 5.);
  f_t time_limit                 = std::min(60., total_time_to_generate.remaining_time());
  f_t ls_limit                   = std::min(5., timer.remaining_time() / 20.);
  const i_t n_sols_to_generate   = 2;
  for (i_t i = 0; i < n_sols_to_generate; ++i) {
    CUOPT_LOG_DEBUG("Trying to generate more solutions");
    time_limit = std::min(time_limit, timer.remaining_time());
    ls.fj.randomize_weights(problem_ptr->handle_ptr);
    auto sol = generate_solution(time_limit);
    population.run_solution_callbacks(sol);
    solutions.emplace_back(solution_t<i_t, f_t>(sol));
    if (total_time_to_generate.check_time_limit()) { return solutions; }
    timer_t timer(std::min(ls_limit, timer.remaining_time()));
    ls.run_local_search(sol, population.weights, timer);
    population.run_solution_callbacks(sol);
    solutions.emplace_back(std::move(sol));
    if (total_time_to_generate.check_time_limit()) { return solutions; }
  }
  return solutions;
}

template <typename i_t, typename f_t>
solution_t<i_t, f_t> diversity_manager_t<i_t, f_t>::generate_solution(f_t time_limit,
                                                                      bool random_start)
{
  solution_t<i_t, f_t> sol(*problem_ptr);
  sol.compute_feasibility();
  ls.generate_solution(sol, random_start, population.early_exit_primal_generation, time_limit);
  return sol;
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::generate_add_solution(
  std::vector<solution_t<i_t, f_t>>& initial_sol_vector, f_t time_limit, bool random_start)
{
  // TODO check weights here if they are all similar
  // do a local search than add it searched solution as well
  initial_sol_vector.emplace_back(generate_solution(time_limit, random_start));
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::average_fj_weights(i_t i)
{
  thrust::transform(problem_ptr->handle_ptr->get_thrust_policy(),
                    population.weights.cstr_weights.begin(),
                    population.weights.cstr_weights.end(),
                    ls.fj.cstr_weights.begin(),
                    population.weights.cstr_weights.begin(),
                    [i] __device__(f_t w1, f_t w2) { return (w1 * i + w2) / (i + 1); });
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::add_user_given_solution(
  std::vector<solution_t<i_t, f_t>>& initial_sol_vector)
{
  if (context.settings.has_initial_solution()) {
    solution_t<i_t, f_t> sol(*problem_ptr);
    auto& init_sol = context.settings.get_initial_solution();
    rmm::device_uvector<f_t> init_sol_assignment(init_sol, sol.handle_ptr->get_stream());
    if (problem_ptr->pre_process_assignment(init_sol_assignment)) {
      raft::copy(sol.assignment.data(),
                 init_sol_assignment.data(),
                 init_sol_assignment.size(),
                 sol.handle_ptr->get_stream());
      bool is_feasible = sol.compute_feasibility();
      cuopt_func_call(sol.test_variable_bounds(true));
      CUOPT_LOG_INFO("Adding initial solution success! feas %d objective %f excess %f",
                     is_feasible,
                     sol.get_objective(),
                     sol.get_total_excess());
      population.run_solution_callbacks(sol);
      initial_sol_vector.emplace_back(std::move(sol));
    } else {
      CUOPT_LOG_ERROR(
        "Error cannot add the provided initial solution! \
    Assignment size %lu \
    initial solution size %lu",
        sol.assignment.size(),
        init_sol.size());
    }
  }
}

// if 60% of the time, exit
// if 20% of the time finishes and we generate 5 solutions
template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::generate_initial_solutions()
{
  add_user_given_solution(initial_sol_vector);
  // allocate maximum of 40% of the time to the initial island generation
  // aim to generate at least 5 feasible solutions thus spending 8% of the time to generate a
  // solution if we can generate faster generate up to 10 sols
  const f_t generation_time_limit = 0.6 * timer.get_time_limit();
  const f_t max_island_gen_time   = 600;
  f_t total_island_gen_time       = std::min(generation_time_limit, max_island_gen_time);
  timer_t gen_timer(total_island_gen_time);
  f_t sol_time_limit = gen_timer.remaining_time();
  for (i_t i = 0; i < maximum_island_size; ++i) {
    if (check_b_b_preemption()) { return; }
    if (i + population.get_external_solution_size() >= 5) { break; }
    CUOPT_LOG_DEBUG("Generating sol %d", i);
    bool is_first_sol = (i == 0);
    if (i == 1) { sol_time_limit = gen_timer.remaining_time() / (initial_island_size - 1); }
    // in first iteration, definitely generate feasible
    if (is_first_sol) {
      sol_time_limit = gen_timer.remaining_time();
      ls.fj.reset_weights(problem_ptr->handle_ptr->get_stream());
    }
    // in other iterations(when there is at least one feasible)
    else {
      ls.fj.randomize_weights(problem_ptr->handle_ptr);
    }
    generate_add_solution(initial_sol_vector, sol_time_limit, !is_first_sol);
    if (is_first_sol && initial_sol_vector.back().get_feasible()) {
      CUOPT_LOG_DEBUG("First FP/FJ solution found at %f with objective %f",
                      timer.elapsed_time(),
                      initial_sol_vector.back().get_user_objective());
    }
    population.run_solution_callbacks(initial_sol_vector.back());
    // run ls on the generated solutions
    solution_t<i_t, f_t> searched_sol(initial_sol_vector.back());
    ls.run_local_search(searched_sol, population.weights, gen_timer);
    population.run_solution_callbacks(searched_sol);
    initial_sol_vector.emplace_back(std::move(searched_sol));
    average_fj_weights(i);
    // run ls on the solutions
    // if at least initial_island_size solutions are generated and time limit is reached
    if (i >= initial_island_size || gen_timer.check_time_limit()) { break; }
  }
  CUOPT_LOG_DEBUG("Initial unsearched solutions are generated!");
  i_t actual_island_size = initial_sol_vector.size();
  population.normalize_weights();
  // find diversity of the population
  population.find_diversity(initial_sol_vector, use_avg_diversity);
  population.add_solutions_from_vec(std::move(initial_sol_vector));
  population.update_qualities();
  CUOPT_LOG_DEBUG("Initial population generated, size %d var_threshold %d",
                  population.current_size(),
                  population.var_threshold);
  population.print();
  auto new_sol_vector = population.get_external_solutions();
  recombine_and_ls_with_all(new_sol_vector);
}

template <typename i_t, typename f_t>
bool diversity_manager_t<i_t, f_t>::run_presolve(f_t time_limit)
{
  CUOPT_LOG_INFO("Running presolve!");
  timer_t presolve_timer(time_limit);
  auto term_crit = ls.constraint_prop.bounds_update.solve(*problem_ptr);
  if (ls.constraint_prop.bounds_update.infeas_constraints_count > 0) {
    stats.presolve_time = timer.elapsed_time();
    return false;
  }
  if (termination_criterion_t::NO_UPDATE != term_crit) {
    ls.constraint_prop.bounds_update.set_updated_bounds(*problem_ptr);
    trivial_presolve(*problem_ptr);
    if (!problem_ptr->empty && !check_bounds_sanity(*problem_ptr)) { return false; }
  }
  if (!problem_ptr->empty) {
    // do the resizing no-matter what, bounds presolve might not change the bounds but initial
    // trivial presolve might have
    ls.constraint_prop.bounds_update.resize(*problem_ptr);
    ls.constraint_prop.conditional_bounds_update.update_constraint_bounds(
      *problem_ptr, ls.constraint_prop.bounds_update);
    if (!check_bounds_sanity(*problem_ptr)) { return false; }
  }
  stats.presolve_time = presolve_timer.elapsed_time();
  return true;
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::generate_quick_feasible_solution()
{
  raft::common::nvtx::range fun_scope("Generate Quick Feasible Solution");

  solution_t<i_t, f_t> solution(*problem_ptr);
  // min 1 second, max 10 seconds
  const f_t generate_fast_solution_time = std::min(10., std::max(1., timer.remaining_time() / 20.));
  timer_t sol_timer(generate_fast_solution_time);
  // do very short LP run to get somewhere close to the optimal point
  ls.generate_fast_solution(solution, sol_timer);
  if (solution.get_feasible()) {
    population.run_solution_callbacks(solution);
    initial_sol_vector.emplace_back(std::move(solution));
    problem_ptr->handle_ptr->sync_stream();
    solution_t<i_t, f_t> searched_sol(initial_sol_vector.back());
    ls.run_local_search(searched_sol, population.weights, sol_timer);
    population.run_solution_callbacks(searched_sol);
    initial_sol_vector.emplace_back(std::move(searched_sol));
    auto& feas_sol = initial_sol_vector.back().get_feasible()
                       ? initial_sol_vector.back()
                       : initial_sol_vector[initial_sol_vector.size() - 2];
    CUOPT_LOG_INFO("Generated fast solution in %f seconds with objective %f",
                   timer.elapsed_time(),
                   feas_sol.get_user_objective());
  }
  problem_ptr->handle_ptr->sync_stream();
}

template <typename i_t, typename f_t>
bool diversity_manager_t<i_t, f_t>::check_b_b_preemption()
{
  if (population.preempt_heuristic_solver_) {
    if (population.current_size() == 0) { population.allocate_solutions(); }
    auto new_sol_vector = population.get_external_solutions();
    population.add_solutions_from_vec(std::move(new_sol_vector));
    return true;
  }
  return false;
}

// returns the best feasible solution
template <typename i_t, typename f_t>
solution_t<i_t, f_t> diversity_manager_t<i_t, f_t>::run_solver()
{
  const f_t time_limit                = timer.remaining_time();
  constexpr f_t time_ratio_on_init_lp = 0.1;
  constexpr f_t max_time_on_lp        = 30;
  const f_t lp_time_limit = std::min(max_time_on_lp, time_limit * time_ratio_on_init_lp);

  // to automatically compute the solving time on scope exit
  auto timer_raii_guard =
    cuopt::scope_guard([&]() { stats.total_solve_time = timer.elapsed_time(); });

  // after every change to the problem, we should resize all the relevant vars
  // we need to encapsulate that to prevent repetitions
  lp_optimal_solution.resize(problem_ptr->n_variables, problem_ptr->handle_ptr->get_stream());
  ls.resize_vectors(*problem_ptr, problem_ptr->handle_ptr);
  ls.lb_constraint_prop.temp_problem.setup(*problem_ptr);
  ls.lb_constraint_prop.bounds_update.setup(ls.lb_constraint_prop.temp_problem);
  ls.constraint_prop.bounds_update.resize(*problem_ptr);
  problem_ptr->check_problem_representation(true);
  // test problem is not ii
  cuopt_func_call(
    ls.constraint_prop.bounds_update.calculate_activity_on_problem_bounds(*problem_ptr));
  cuopt_assert(
    ls.constraint_prop.bounds_update.calculate_infeasible_redundant_constraints(*problem_ptr),
    "The problem must not be ii");
  population.initialize_population();
  if (check_b_b_preemption()) { return population.best_feasible(); }
  // before probing cache or LP, run FJ to generate initial primal feasible solution
  generate_quick_feasible_solution();
  constexpr f_t time_ratio_of_probing_cache = 0.10;
  constexpr f_t max_time_on_probing         = 60;
  f_t time_for_probing_cache =
    std::min(max_time_on_probing, time_limit * time_ratio_of_probing_cache);
  timer_t probing_timer{time_for_probing_cache};
  if (check_b_b_preemption()) { return population.best_feasible(); }
  compute_probing_cache(ls.constraint_prop.bounds_update, *problem_ptr, probing_timer);
  // careful, assign the correct probing cache
  ls.lb_constraint_prop.bounds_update.probing_cache.probing_cache =
    ls.constraint_prop.bounds_update.probing_cache.probing_cache;

  if (check_b_b_preemption()) { return population.best_feasible(); }
  lp_state_t<i_t, f_t>& lp_state = context.lp_state;
  // resize because some constructor might be called before the presolve
  lp_state.resize(*problem_ptr, problem_ptr->handle_ptr->get_stream());
  auto lp_result = get_relaxed_lp_solution(*problem_ptr,
                                           lp_optimal_solution,
                                           lp_state,
                                           context.settings.tolerances.absolute_tolerance,
                                           lp_time_limit,
                                           false,
                                           false);
  population.allocate_solutions();
  ls.lp_optimal_exists = true;
  if (lp_result.get_termination_status() == pdlp_termination_status_t::Optimal) {
    // get lp user objective and pass it to set_new_user_bound
    set_new_user_bound(lp_result.get_objective_value());
  } else if (lp_result.get_termination_status() == pdlp_termination_status_t::PrimalInfeasible) {
    // PDLP's infeasibility detection isn't an exact method and might be subject to false positives.
    // Issue a warning, and continue solving.
    CUOPT_LOG_WARN("PDLP detected primal infeasibility, problem might be infeasible!");
    ls.lp_optimal_exists = false;
  } else if (lp_result.get_termination_status() == pdlp_termination_status_t::DualInfeasible) {
    CUOPT_LOG_WARN("PDLP detected dual infeasibility, problem might be unbounded!");
    ls.lp_optimal_exists = false;
  } else if (lp_result.get_termination_status() == pdlp_termination_status_t::TimeLimit) {
    CUOPT_LOG_DEBUG(
      "Initial LP run exceeded time limit, continuing solver with partial LP result!");
    // note to developer, in debug mode the LP run might be too slow and it might cause PDLP not to
    // bring variables within the bounds
  }
  // in case the pdlp returned var boudns that are out of bounds
  clamp_within_var_bounds(lp_optimal_solution, problem_ptr, problem_ptr->handle_ptr);
  if (check_b_b_preemption()) { return population.best_feasible(); }
  // generate a population with 5 solutions(FP+FJ)
  generate_initial_solutions();
  if (timer.check_time_limit()) { return population.best_feasible(); }
  main_loop();
  return population.best_feasible();
};

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::diversity_step()
{
  // TODO when the solver is faster, increase this number
  constexpr i_t max_iterations_without_improvement = 15;
  bool improved                                    = true;
  while (improved) {
    int k    = max_iterations_without_improvement;
    improved = false;
    while (k-- > 0) {
      if (check_b_b_preemption()) { return; }
      auto new_sol_vector = population.get_external_solutions();
      recombine_and_ls_with_all(new_sol_vector);

      cuopt_assert(population.test_invariant(), "");
      if (population.current_size() < 2) {
        CUOPT_LOG_DEBUG("Population degenerated in diversity step");
        return;
      }
      if (timer.check_time_limit()) return;
      constexpr bool tournament = true;
      auto [sol1, sol2]         = population.get_two_random(tournament);
      cuopt_assert(population.test_invariant(), "");
      auto [lp_offspring, offspring] = recombine_and_local_search(sol1, sol2);
      i_t inserted_pos_1             = population.add_solution(std::move(lp_offspring));
      i_t inserted_pos_2             = population.add_solution(std::move(offspring));
      cuopt_assert(population.test_invariant(), "");
      if ((inserted_pos_1 != -1 && inserted_pos_1 <= 3) ||
          (inserted_pos_2 != -1 && inserted_pos_2 <= 3)) {
        improved = true;
        recombine_stats.print();
        break;
      }
    }
  }
  recombine_stats.print();
}

// TODO check if the new bound is actually better than the previous one.
// consider max problems too!
template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::set_new_user_bound(f_t new_bound)
{
  stats.solution_bound = new_bound;
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::recombine_and_ls_with_all(solution_t<i_t, f_t>& solution)
{
  raft::common::nvtx::range fun_scope("recombine_and_ls_with_all");
  auto population_vector = population.population_to_vector();
  for (auto& curr_sol : population_vector) {
    if (check_b_b_preemption()) { return; }
    if (curr_sol.get_feasible()) {
      auto [offspring, lp_offspring] = recombine_and_local_search(curr_sol, solution);
      i_t inserted_pos_1             = population.add_solution(std::move(lp_offspring));
      i_t inserted_pos_2             = population.add_solution(std::move(offspring));
      if (timer.check_time_limit()) { return; }
    }
  }
  population.add_solution(std::move(solution_t<i_t, f_t>(solution)));
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::recombine_and_ls_with_all(
  std::vector<solution_t<i_t, f_t>>& solutions)
{
  raft::common::nvtx::range fun_scope("recombine_and_ls_with_all");
  if (solutions.size() > 0) {
    CUOPT_LOG_INFO("Running recombiners on B&B solutions with size %lu", solutions.size());
    for (auto& sol : solutions) {
      population.add_solution(std::move(solution_t<i_t, f_t>(sol)));
      cuopt_func_call(sol.test_feasibility(true));
      solution_t<i_t, f_t> ls_solution(sol);
      ls.run_local_search(ls_solution, population.weights, timer);
      // TODO try if running LP with integers fixed makes it feasible
      if (ls_solution.get_feasible()) {
        CUOPT_LOG_DEBUG("External LS searched solution feasible, running recombiners!");
        recombine_and_ls_with_all(ls_solution);
      } else {
        CUOPT_LOG_DEBUG("External solution feasible, running recombiners!");
        recombine_and_ls_with_all(sol);
      }
    }
  }
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::main_loop()
{
  population.diversity_start_time = timer.elapsed_time();
  recombine_stats.reset();
  while (true) {
    if (check_b_b_preemption()) { break; }
    CUOPT_LOG_DEBUG("Running a new step");
    bool enough_solutions = regenerate_solutions();
    if (!enough_solutions) {
      // do a longer search on the best solution then exit
      auto best_sol = population.is_feasible() ? population.best_feasible() : population.best();
      ls.run_fj_until_timer(best_sol, population.weights, timer);
      population.add_solution(std::move(best_sol));
      CUOPT_LOG_WARN("Enough solutions couldn't be generated,exiting heuristics!");
      break;
    }
    if (timer.check_time_limit()) { break; }
    diversity_step();
    if (timer.check_time_limit()) { break; }
    population.halve_the_population();
    auto new_solutions      = generate_more_solutions();
    auto current_population = population.population_to_vector();
    population.clear();
    current_population.insert(current_population.end(),
                              std::make_move_iterator(new_solutions.begin()),
                              std::make_move_iterator(new_solutions.end()));
    population.find_diversity(current_population, use_avg_diversity);
    population.add_solutions_from_vec(std::move(current_population));
    // population.add_solutions_from_vec(std::move(new_solutions));
    // idea to try, we can average the weights of the new solutions
    population.update_weights();
    population.print();
    if (timer.check_time_limit()) { break; }
  }
  auto new_sol_vector = population.get_external_solutions();
  recombine_and_ls_with_all(new_sol_vector);
  population.print();
}

template <typename i_t, typename f_t>
std::pair<solution_t<i_t, f_t>, solution_t<i_t, f_t>>
diversity_manager_t<i_t, f_t>::recombine_and_local_search(solution_t<i_t, f_t>& sol1,
                                                          solution_t<i_t, f_t>& sol2)
{
  raft::common::nvtx::range fun_scope("recombine_and_local_search");
  CUOPT_LOG_DEBUG("Recombining sol cost:feas %f : %d and %f : %d",
                  sol1.get_quality(population.weights),
                  sol1.get_feasible(),
                  sol2.get_quality(population.weights),
                  sol2.get_feasible());
  // randomly choose among 3 recombiners
  auto [offspring, success] = recombine(sol1, sol2);
  if (!success) { return std::make_pair(solution_t<i_t, f_t>(sol1), solution_t<i_t, f_t>(sol2)); }
  cuopt_assert(population.test_invariant(), "");
  cuopt_func_call(offspring.test_variable_bounds(false));
  CUOPT_LOG_DEBUG("Recombiner offspring sol cost:feas %f : %d",
                  offspring.get_quality(population.weights),
                  offspring.get_feasible());
  cuopt_assert(offspring.test_number_all_integer(), "All must be integers before LS");
  bool feasibility_before = offspring.get_feasible();
  ls.run_local_search(offspring, population.weights, timer);
  cuopt_assert(offspring.test_number_all_integer(), "All must be integers after LS");
  cuopt_assert(population.test_invariant(), "");

  CUOPT_LOG_DEBUG("After LS offspring sol cost:feas %f : %d",
                  offspring.get_quality(population.weights),
                  offspring.get_feasible());
  offspring.compute_feasibility();
  cuopt_assert(population.test_invariant(), "");
  // run LP with the vars
  solution_t<i_t, f_t> lp_offspring(offspring);
  cuopt_assert(population.test_invariant(), "");
  cuopt_assert(lp_offspring.test_number_all_integer(), "All must be integers before LP");
  f_t lp_run_time = offspring.get_feasible() ? 3. : 1.;
  lp_run_time     = std::min(lp_run_time, timer.remaining_time());
  run_lp_with_vars_fixed(*lp_offspring.problem_ptr,
                         lp_offspring,
                         lp_offspring.problem_ptr->integer_indices,
                         context.settings.get_tolerances(),
                         context.lp_state,
                         lp_run_time);
  cuopt_assert(population.test_invariant(), "");
  cuopt_assert(lp_offspring.test_number_all_integer(), "All must be integers after LP");
  f_t lp_qual = lp_offspring.get_quality(population.weights);
  CUOPT_LOG_DEBUG("After LP offspring sol cost:feas %f : %d", lp_qual, lp_offspring.get_feasible());
  f_t offspring_qual = std::min(offspring.get_quality(population.weights), lp_qual);
  recombine_stats.update_improve_stats(
    offspring_qual, sol1.get_quality(population.weights), sol2.get_quality(population.weights));
  return std::make_pair(std::move(offspring), std::move(lp_offspring));
}

template <typename i_t, typename f_t>
std::pair<solution_t<i_t, f_t>, bool> diversity_manager_t<i_t, f_t>::recombine(
  solution_t<i_t, f_t>& a, solution_t<i_t, f_t>& b)
{
  i_t recombiner = std::uniform_int_distribution(0, recombiner_enum_t::SIZE - 1)(rng);
  recombine_stats.add_attempt((recombiner_enum_t)recombiner);
  if (recombiner == recombiner_enum_t::BOUND_PROP) {
    CUOPT_LOG_DEBUG("Running bound_prop recombiner");
    auto [sol, success] = bound_prop_recombiner.recombine(a, b);
    if (success) { recombine_stats.add_success(); }
    return std::make_pair(sol, success);
  } else if (recombiner == recombiner_enum_t::FP) {
    CUOPT_LOG_DEBUG("Running fp recombiner");
    auto [sol, success] = fp_recombiner.recombine(a, b);
    if (success) { recombine_stats.add_success(); }
    return std::make_pair(sol, success);
  } else {
    CUOPT_LOG_DEBUG("Running line segment recombiner");
    auto [sol, success] = line_segment_recombiner.recombine(a, b, population.weights);
    if (success) { recombine_stats.add_success(); }
    return std::make_pair(sol, success);
  }
}

#if MIP_INSTANTIATE_FLOAT
template class diversity_manager_t<int, float>;
#endif

#if MIP_INSTANTIATE_DOUBLE
template class diversity_manager_t<int, double>;
#endif

}  // namespace cuopt::linear_programming::detail
