#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <mip/mip_constants.hpp>
#include <mip/presolve/probing_cache.cuh>
#include <mip/presolve/trivial_presolve.cuh>
#include <mip/problem/problem_helpers.cuh>
#include "diversity_manager.cuh"

#include <utilities/scope_guard.hpp>

#include "hip/hip_runtime_api.h"

constexpr bool from_dir    = false;
constexpr bool fj_only_run = false;

namespace cuopt::linear_programming::detail {

size_t fp_recombiner_config_t::max_n_of_vars_from_other =
  fp_recombiner_config_t::initial_n_of_vars_from_other;
size_t ls_recombiner_config_t::max_n_of_vars_from_other =
  ls_recombiner_config_t::initial_n_of_vars_from_other;
size_t bp_recombiner_config_t::max_n_of_vars_from_other =
  bp_recombiner_config_t::initial_n_of_vars_from_other;
size_t sub_mip_recombiner_config_t::max_n_of_vars_from_other =
  sub_mip_recombiner_config_t::initial_n_of_vars_from_other;

template <typename i_t, typename f_t>
diversity_manager_t<i_t, f_t>::diversity_manager_t(mip_solver_context_t<i_t, f_t>& context_)
  : context(context_),
    problem_ptr(context.problem_ptr),
    diversity_config(),
    population("population",
               context,
               *this,
               diversity_config.max_var_diff,
               diversity_config.max_solutions,
               diversity_config.initial_infeasibility_weight * context.problem_ptr->n_constraints),
    lp_optimal_solution(context.problem_ptr->n_variables,
                        context.problem_ptr->handle_ptr->get_stream()),
    lp_dual_optimal_solution(context.problem_ptr->n_constraints,
                             context.problem_ptr->handle_ptr->get_stream()),
    ls(context, lp_optimal_solution),
    timer(diversity_config.default_time_limit),
    bound_prop_recombiner(context,
                          context.problem_ptr->n_variables,
                          ls.constraint_prop,
                          context.problem_ptr->handle_ptr),
    fp_recombiner(context,
                  context.problem_ptr->n_variables,
                  ls.fj,
                  ls.constraint_prop,
                  ls.line_segment_search,
                  lp_optimal_solution,
                  context.problem_ptr->handle_ptr),
    line_segment_recombiner(context,
                            context.problem_ptr->n_variables,
                            ls.line_segment_search,
                            context.problem_ptr->handle_ptr),
    sub_mip_recombiner(
      context, population, context.problem_ptr->n_variables, context.problem_ptr->handle_ptr),
    rng(cuopt::seed_generator::get_seed()),
    stats(context.stats),
    mab_recombiner(static_cast<int>(recombiner_enum_t::SIZE),
                   cuopt::seed_generator::get_seed(),
                   recombiner_alpha,
                   "recombiner"),
    mab_ls(mab_ls_config_t<i_t, f_t>::n_of_arms, cuopt::seed_generator::get_seed(), ls_alpha, "ls"),
    ls_hash_map(*context.problem_ptr)
{
  // Read configuration ID from environment variable
  int max_config = -1;
  // Read max configuration value from environment variable
  const char* env_max_config = std::getenv("CUOPT_MAX_CONFIG");
  if (env_max_config != nullptr) {
    try {
      max_config = std::stoi(env_max_config);
      CUOPT_LOG_INFO("Using maximum configuration value from environment: %d", max_config);
    } catch (const std::exception& e) {
      CUOPT_LOG_WARN("Failed to parse CUOPT_MAX_CONFIG environment variable: %s", e.what());
    }
  }
  if (max_config > 1) {
    int config_id             = -1;  // Default value
    const char* env_config_id = std::getenv("CUOPT_CONFIG_ID");
    if (env_config_id != nullptr) {
      try {
        config_id = std::stoi(env_config_id);
        CUOPT_LOG_INFO("Using configuration ID from environment: %d", config_id);
      } catch (const std::exception& e) {
        CUOPT_LOG_WARN("Failed to parse CUOPT_CONFIG_ID environment variable: %s", e.what());
      }
    }
  }
}

// this function is to specialize the local search with config from diversity manager
template <typename i_t, typename f_t>
bool diversity_manager_t<i_t, f_t>::run_local_search(solution_t<i_t, f_t>& solution,
                                                     const weight_t<i_t, f_t>& weights,
                                                     timer_t& timer,
                                                     ls_config_t<i_t, f_t>& ls_config)
{
  i_t ls_mab_option = mab_ls.select_mab_option();
  mab_ls_config_t<i_t, f_t>::get_local_search_and_lm_from_config(ls_mab_option, ls_config);
  ls_hash_map.insert(solution);
  constexpr i_t skip_solutions_threshold = 3;
  if (ls_hash_map.check_skip_solution(solution, skip_solutions_threshold)) { return false; }
  ls.run_local_search(solution, weights, timer, ls_config);
  return true;
}

// There should be at least 3 solutions in the population
template <typename i_t, typename f_t>
bool diversity_manager_t<i_t, f_t>::regenerate_solutions()
{
  f_t time_limit     = 5;
  i_t counter        = 0;
  const i_t min_size = 2;
  while (population.current_size() <= min_size && (current_step == 0 || counter < 5)) {
    CUOPT_LOG_DEBUG("Trying to regenerate solution, pop size %d\n", population.current_size());
    time_limit = std::min(time_limit, timer.remaining_time());
    ls.fj.randomize_weights(problem_ptr->handle_ptr);
    population.add_solution(generate_solution(time_limit));
    if (timer.check_time_limit()) { return false; }
    // increase the time limit as we couldn't add a valid solution
    time_limit += 5;
    counter++;
  }
  ++current_step;
  // if there is at least two sols still return true
  return population.current_size() >= min_size;
}

// There should be at least 3 solutions in the population
template <typename i_t, typename f_t>
std::vector<solution_t<i_t, f_t>> diversity_manager_t<i_t, f_t>::generate_more_solutions()
{
  std::vector<solution_t<i_t, f_t>> solutions;
  timer_t total_time_to_generate = timer_t(timer.remaining_time() / 5.);
  f_t time_limit                 = std::min(60., total_time_to_generate.remaining_time());
  f_t ls_limit                   = std::min(5., timer.remaining_time() / 20.);
  const i_t n_sols_to_generate   = 3;
  for (i_t i = 0; i < n_sols_to_generate; ++i) {
    CUOPT_LOG_DEBUG("Trying to generate more solutions");
    time_limit = std::min(time_limit, timer.remaining_time());
    ls.fj.randomize_weights(problem_ptr->handle_ptr);
    auto sol = generate_solution(time_limit);
    population.run_solution_callbacks(sol);
    solutions.emplace_back(solution_t<i_t, f_t>(sol));
    if (total_time_to_generate.check_time_limit()) { return solutions; }
    timer_t ls_timer(std::min(ls_limit, timer.remaining_time()));
    ls_config_t<i_t, f_t> ls_config;
    run_local_search(sol, population.weights, ls_timer, ls_config);
    population.run_solution_callbacks(sol);
    solutions.emplace_back(std::move(sol));
    if (total_time_to_generate.check_time_limit()) { return solutions; }
  }
  return solutions;
}

template <typename i_t, typename f_t>
solution_t<i_t, f_t> diversity_manager_t<i_t, f_t>::generate_solution(f_t time_limit,
                                                                      bool random_start)
{
  solution_t<i_t, f_t> sol(*problem_ptr);
  sol.compute_feasibility();
  ls.generate_solution(sol, random_start, &population, time_limit);
  return sol;
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::add_user_given_solutions(
  std::vector<solution_t<i_t, f_t>>& initial_sol_vector)
{
  for (const auto& init_sol : context.settings.initial_solutions) {
    solution_t<i_t, f_t> sol(*problem_ptr);
    rmm::device_uvector<f_t> init_sol_assignment(*init_sol, sol.handle_ptr->get_stream());
    if (problem_ptr->pre_process_assignment(init_sol_assignment)) {
      raft::copy(sol.assignment.data(),
                 init_sol_assignment.data(),
                 init_sol_assignment.size(),
                 sol.handle_ptr->get_stream());
      bool is_feasible = sol.compute_feasibility();
      cuopt_func_call(sol.test_variable_bounds(true));
      CUOPT_LOG_INFO("Adding initial solution success! feas %d objective %f excess %f",
                     is_feasible,
                     sol.get_user_objective(),
                     sol.get_total_excess());
      population.run_solution_callbacks(sol);
      initial_sol_vector.emplace_back(std::move(sol));
    } else {
      CUOPT_LOG_ERROR(
        "Error cannot add the provided initial solution! \
    Assignment size %lu \
    initial solution size %lu",
        sol.assignment.size(),
        init_sol_assignment.size());
    }
  }
}

template <typename i_t, typename f_t>
bool diversity_manager_t<i_t, f_t>::run_presolve(f_t time_limit)
{
  CUOPT_LOG_INFO("Running presolve!");
  timer_t presolve_timer(time_limit);
  auto term_crit = ls.constraint_prop.bounds_update.solve(*problem_ptr);
  if (ls.constraint_prop.bounds_update.infeas_constraints_count > 0) {
    stats.presolve_time = timer.elapsed_time();
    return false;
  }
  if (termination_criterion_t::NO_UPDATE != term_crit) {
    ls.constraint_prop.bounds_update.set_updated_bounds(*problem_ptr);
    trivial_presolve(*problem_ptr);
    if (!problem_ptr->empty && !check_bounds_sanity(*problem_ptr)) { return false; }
  }
  // May overconstrain if Papilo presolve has been run before
  if (!context.settings.presolve) {
    if (!problem_ptr->empty) {
      // do the resizing no-matter what, bounds presolve might not change the bounds but initial
      // trivial presolve might have
      ls.constraint_prop.bounds_update.resize(*problem_ptr);
      ls.constraint_prop.conditional_bounds_update.update_constraint_bounds(
        *problem_ptr, ls.constraint_prop.bounds_update);
      if (!check_bounds_sanity(*problem_ptr)) { return false; }
    }
  }
  stats.presolve_time = presolve_timer.elapsed_time();
  lp_optimal_solution.resize(problem_ptr->n_variables, problem_ptr->handle_ptr->get_stream());
  lp_dual_optimal_solution.resize(problem_ptr->n_constraints,
                                  problem_ptr->handle_ptr->get_stream());
  problem_ptr->handle_ptr->sync_stream();
  CUOPT_LOG_INFO("After trivial presolve: %d constraints, %d variables, objective offset %f.",
                 problem_ptr->n_constraints,
                 problem_ptr->n_variables,
                 problem_ptr->presolve_data.objective_offset);
  return true;
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::generate_quick_feasible_solution()
{
  solution_t<i_t, f_t> solution(*problem_ptr);
  // min 1 second, max 10 seconds
  const f_t generate_fast_solution_time =
    std::min(diversity_config.max_fast_sol_time, std::max(1., timer.remaining_time() / 20.));
  timer_t sol_timer(generate_fast_solution_time);
  // do very short LP run to get somewhere close to the optimal point
  ls.generate_fast_solution(solution, sol_timer);
  if (solution.get_feasible()) {
    population.run_solution_callbacks(solution);
    initial_sol_vector.emplace_back(std::move(solution));
    problem_ptr->handle_ptr->sync_stream();
    solution_t<i_t, f_t> searched_sol(initial_sol_vector.back());
    ls_config_t<i_t, f_t> ls_config;
    run_local_search(searched_sol, population.weights, sol_timer, ls_config);
    population.run_solution_callbacks(searched_sol);
    initial_sol_vector.emplace_back(std::move(searched_sol));
    auto& feas_sol = initial_sol_vector.back().get_feasible()
                       ? initial_sol_vector.back()
                       : initial_sol_vector[initial_sol_vector.size() - 2];
    CUOPT_LOG_INFO("Generated fast solution in %f seconds with objective %f",
                   timer.elapsed_time(),
                   feas_sol.get_user_objective());
  }
  problem_ptr->handle_ptr->sync_stream();
}

template <typename i_t, typename f_t>
bool diversity_manager_t<i_t, f_t>::check_b_b_preemption()
{
  if (population.preempt_heuristic_solver_.load()) {
    if (population.current_size() == 0) { population.allocate_solutions(); }
    auto new_sol_vector = population.get_external_solutions();
    population.add_solutions_from_vec(std::move(new_sol_vector));
    return true;
  }
  return false;
}

// returns the best feasible solution
template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::run_fj_alone(solution_t<i_t, f_t>& solution)
{
  CUOPT_LOG_INFO("Running FJ alone!");
  solution.round_nearest();
  ls.fj.settings.mode                   = fj_mode_t::EXIT_NON_IMPROVING;
  ls.fj.settings.n_of_minimums_for_exit = 20000 * 1000;
  ls.fj.settings.update_weights         = true;
  ls.fj.settings.feasibility_run        = false;
  ls.fj.settings.time_limit             = timer.remaining_time();
  ls.fj.solve(solution);
  CUOPT_LOG_INFO("FJ alone finished!");
}

// returns the best feasible solution
template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::run_fp_alone(solution_t<i_t, f_t>& solution)
{
  CUOPT_LOG_DEBUG("Running FP alone!");
  ls.run_fp(solution, timer, &population);
  CUOPT_LOG_DEBUG("FP alone finished!");
}

template <typename i_t, typename f_t>
struct ls_cpufj_raii_guard_t {
  ls_cpufj_raii_guard_t(local_search_t<i_t, f_t>& ls) : ls(ls) {}
  ~ls_cpufj_raii_guard_t() { ls.stop_cpufj_scratch_threads(); }
  local_search_t<i_t, f_t>& ls;
};

// returns the best feasible solution
template <typename i_t, typename f_t>
solution_t<i_t, f_t> diversity_manager_t<i_t, f_t>::run_solver()
{
  population.timer     = timer;
  const f_t time_limit = timer.remaining_time();
  const f_t lp_time_limit =
    std::min(diversity_config.max_time_on_lp, time_limit * diversity_config.time_ratio_on_init_lp);
  // to automatically compute the solving time on scope exit
  auto timer_raii_guard =
    cuopt::scope_guard([&]() { stats.total_solve_time = timer.elapsed_time(); });
  // after every change to the problem, we should resize all the relevant vars
  // we need to encapsulate that to prevent repetitions

  ls.resize_vectors(*problem_ptr, problem_ptr->handle_ptr);
  ls.constraint_prop.bounds_update.resize(*problem_ptr);
  problem_ptr->check_problem_representation(true);
  // have the structure ready for reusing later
  problem_ptr->compute_integer_fixed_problem();

  // test problem is not ii
  cuopt_func_call(
    ls.constraint_prop.bounds_update.calculate_activity_on_problem_bounds(*problem_ptr));
  cuopt_assert(
    ls.constraint_prop.bounds_update.calculate_infeasible_redundant_constraints(*problem_ptr),
    "The problem must not be ii");
  population.initialize_population();
  if (check_b_b_preemption()) { return population.best_feasible(); }

  // Run CPUFJ early to find quick initial solutions
  population.allocate_solutions();
  ls_cpufj_raii_guard_t ls_cpufj_raii_guard(ls);  // RAII to stop cpufj threads on solve stop
  ls.start_cpufj_scratch_threads(population);

  // before probing cache or LP, run FJ to generate initial primal feasible solution
  // TODO: commenting this out decreases the gap on trento1.mps dramatically. figure out why?
  // if (!from_dir && !fj_only_run) { generate_quick_feasible_solution(); }
  const f_t time_ratio_of_probing_cache = diversity_config.time_ratio_of_probing_cache;
  const f_t max_time_on_probing         = diversity_config.max_time_on_probing;
  f_t time_for_probing_cache =
    std::min(max_time_on_probing, time_limit * time_ratio_of_probing_cache);
  timer_t probing_timer{time_for_probing_cache};
  if (check_b_b_preemption()) { return population.best_feasible(); }
  if (!fj_only_run) {
    compute_probing_cache(ls.constraint_prop.bounds_update, *problem_ptr, probing_timer);
  }

  if (check_b_b_preemption()) { return population.best_feasible(); }
  lp_state_t<i_t, f_t>& lp_state = problem_ptr->lp_state;
  // resize because some constructor might be called before the presolve
  lp_state.resize(*problem_ptr, problem_ptr->handle_ptr->get_stream());
  bool bb_thread_solution_exists = simplex_solution_exists.load();
  if (bb_thread_solution_exists) {
    ls.lp_optimal_exists = true;
  } else if (!fj_only_run) {
    relaxed_lp_settings_t lp_settings;
    lp_settings.time_limit            = lp_time_limit;
    lp_settings.tolerance             = context.settings.tolerances.absolute_tolerance;
    lp_settings.return_first_feasible = false;
    lp_settings.save_state            = true;
    lp_settings.concurrent_halt       = &global_concurrent_halt;
    lp_settings.has_initial_primal    = false;
    rmm::device_uvector<f_t> lp_optimal_solution_copy(lp_optimal_solution.size(),
                                                      problem_ptr->handle_ptr->get_stream());
    auto lp_result =
      get_relaxed_lp_solution(*problem_ptr, lp_optimal_solution_copy, lp_state, lp_settings);
    {
      std::lock_guard<std::mutex> guard(relaxed_solution_mutex);
      if (!simplex_solution_exists.load()) {
        raft::copy(lp_optimal_solution.data(),
                   lp_optimal_solution_copy.data(),
                   lp_optimal_solution.size(),
                   problem_ptr->handle_ptr->get_stream());
      } else {
        // copy the lp state
        raft::copy(lp_state.prev_primal.data(),
                   lp_optimal_solution.data(),
                   lp_optimal_solution.size(),
                   problem_ptr->handle_ptr->get_stream());
        raft::copy(lp_state.prev_dual.data(),
                   lp_dual_optimal_solution.data(),
                   lp_dual_optimal_solution.size(),
                   problem_ptr->handle_ptr->get_stream());
      }
    }
    problem_ptr->handle_ptr->sync_stream();
    ls.lp_optimal_exists = true;
    if (lp_result.get_termination_status() == pdlp_termination_status_t::Optimal) {
      set_new_user_bound(lp_result.get_objective_value());
    } else if (lp_result.get_termination_status() == pdlp_termination_status_t::PrimalInfeasible) {
      CUOPT_LOG_ERROR("Problem is primal infeasible, continuing anyway!");
      ls.lp_optimal_exists = false;
    } else if (lp_result.get_termination_status() == pdlp_termination_status_t::DualInfeasible) {
      CUOPT_LOG_ERROR("PDLP detected dual infeasibility, continuing anyway!");
      ls.lp_optimal_exists = false;
    } else if (lp_result.get_termination_status() == pdlp_termination_status_t::TimeLimit) {
      CUOPT_LOG_DEBUG(
        "Initial LP run exceeded time limit, continuing solver with partial LP result!");
      // note to developer, in debug mode the LP run might be too slow and it might cause PDLP not
      // to bring variables within the bounds
    }
    // in case the pdlp returned var boudns that are out of bounds
    clamp_within_var_bounds(lp_optimal_solution, problem_ptr, problem_ptr->handle_ptr);
    ls.start_cpufj_lptopt_scratch_threads(population);
  }

  population.add_solutions_from_vec(std::move(initial_sol_vector));

  if (check_b_b_preemption()) { return population.best_feasible(); }

  if (context.settings.benchmark_info_ptr != nullptr) {
    context.settings.benchmark_info_ptr->objective_of_initial_population =
      population.best_feasible().get_user_objective();
  }

  if (fj_only_run) {
    solution_t<i_t, f_t> sol(*problem_ptr);
    run_fj_alone(sol);
    return sol;
  }

  auto sol = generate_solution(timer.remaining_time(), false);
  population.add_solution(std::move(solution_t<i_t, f_t>(sol)));
  if (timer.check_time_limit()) {
    auto new_sol_vector = population.get_external_solutions();
    population.add_solutions_from_vec(std::move(new_sol_vector));
    return population.best_feasible();
  }
  run_fp_alone(sol);
  population.update_weights();

  if (timer.check_time_limit()) {
    auto new_sol_vector = population.get_external_solutions();
    population.add_solutions_from_vec(std::move(new_sol_vector));
    return population.best_feasible();
  }
  main_loop();

  return population.best_feasible();
};

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::diversity_step()
{
  // TODO when the solver is faster, increase this number
  const i_t max_iterations_without_improvement =
    diversity_config.max_iterations_without_improvement;
  bool improved = true;
  while (improved) {
    int k    = max_iterations_without_improvement;
    improved = false;
    while (k-- > 0) {
      if (check_b_b_preemption()) { return; }
      auto new_sol_vector = population.get_external_solutions();
      recombine_and_ls_with_all(new_sol_vector);
      population.adjust_weights_according_to_best_feasible();
      cuopt_assert(population.test_invariant(), "");
      if (population.current_size() < 2) {
        CUOPT_LOG_DEBUG("Population degenerated in diversity step");
        return;
      }
      if (timer.check_time_limit()) return;
      constexpr bool tournament = true;
      auto [sol1, sol2]         = population.get_two_random(tournament);
      cuopt_assert(population.test_invariant(), "");
      auto [lp_offspring, offspring] = recombine_and_local_search(sol1, sol2);
      i_t inserted_pos_1             = population.add_solution(std::move(lp_offspring));
      i_t inserted_pos_2             = population.add_solution(std::move(offspring));
      cuopt_assert(population.test_invariant(), "");
      if ((inserted_pos_1 != -1 && inserted_pos_1 <= 3) ||
          (inserted_pos_2 != -1 && inserted_pos_2 <= 3)) {
        improved = true;
        recombine_stats.print();
        break;
      }
    }
  }
  recombine_stats.print();
}

// TODO check if the new bound is actually better than the previous one.
// consider max problems too!
template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::set_new_user_bound(f_t new_bound)
{
  stats.solution_bound = new_bound;
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::recombine_and_ls_with_all(solution_t<i_t, f_t>& solution,
                                                              bool add_only_feasible)
{
  raft::common::nvtx::range fun_scope("recombine_and_ls_with_all");
  if (population.population_hash_map.check_skip_solution(solution, 1)) { return; }
  auto population_vector = population.population_to_vector();
  for (auto& curr_sol : population_vector) {
    for (const auto recombiner_type : recombiner_types) {
      if (check_b_b_preemption()) { return; }
      if (curr_sol.get_feasible()) {
        auto [offspring, lp_offspring] =
          recombine_and_local_search(curr_sol, solution, recombiner_type);
        if (!add_only_feasible || lp_offspring.get_feasible()) {
          population.add_solution(std::move(lp_offspring));
        }
        if (!add_only_feasible || offspring.get_feasible()) {
          population.add_solution(std::move(offspring));
        }
        if (timer.check_time_limit()) { return; }
      }
    }
  }
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::recombine_and_ls_with_all(
  std::vector<solution_t<i_t, f_t>>& solutions, bool add_only_feasible)
{
  raft::common::nvtx::range fun_scope("recombine_and_ls_with_all");
  if (solutions.size() > 0) {
    CUOPT_LOG_DEBUG("Running recombiners on B&B solutions with size %lu", solutions.size());
    // add all solutions because time limit might have been consumed and we might have exited before
    for (auto& sol : solutions) {
      cuopt_func_call(sol.test_feasibility(true));
      population.add_solution(std::move(solution_t<i_t, f_t>(sol)));
    }
    for (auto& sol : solutions) {
      if (timer.check_time_limit()) { return; }
      solution_t<i_t, f_t> ls_solution(sol);
      ls_config_t<i_t, f_t> ls_config;
      run_local_search(ls_solution, population.weights, timer, ls_config);
      if (timer.check_time_limit()) { return; }
      // TODO try if running LP with integers fixed makes it feasible
      if (ls_solution.get_feasible()) {
        CUOPT_LOG_DEBUG("External LS searched solution feasible, running recombiners!");
        recombine_and_ls_with_all(ls_solution, add_only_feasible);
      } else {
        CUOPT_LOG_DEBUG("External solution feasible, running recombiners!");
        recombine_and_ls_with_all(sol, add_only_feasible);
      }
    }
  }
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::main_loop()
{
  population.start_threshold_adjustment();
  recombine_stats.reset();
  population.print();
  while (true) {
    if (check_b_b_preemption()) { break; }
    CUOPT_LOG_DEBUG("Running a new step");
    bool enough_solutions = regenerate_solutions();
    if (!enough_solutions) {
      // do a longer search on the best solution then exit
      auto best_sol = population.is_feasible() ? population.best_feasible() : population.best();
      ls.run_fj_until_timer(best_sol, population.weights, timer);
      population.add_solution(std::move(best_sol));
      CUOPT_LOG_WARN("Enough solutions couldn't be generated,exiting heuristics!");
      break;
    }
    if (timer.check_time_limit()) { break; }
    diversity_step();
    if (timer.check_time_limit()) { break; }

    if (diversity_config.halve_population) {
      population.adjust_threshold(timer);
      i_t prev_threshold = population.var_threshold;
      population.halve_the_population();
      auto new_solutions      = generate_more_solutions();
      auto current_population = population.population_to_vector();
      population.clear();
      current_population.insert(current_population.end(),
                                std::make_move_iterator(new_solutions.begin()),
                                std::make_move_iterator(new_solutions.end()));
      population.find_diversity(current_population, diversity_config.use_avg_diversity);
      // if the threshold is lower than the threshold we progress with time
      // set it to the higher threshold
      population.add_solutions_from_vec(std::move(current_population));
    } else {
      // increase the threshold/decrease the diversity
      population.adjust_threshold(timer);
    }
    // idea to try, we can average the weights of the new solutions
    population.update_weights();
    population.print();
    if (timer.check_time_limit()) { break; }
  }
  auto new_sol_vector = population.get_external_solutions();
  recombine_and_ls_with_all(new_sol_vector);
  population.print();
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::check_better_than_both(solution_t<i_t, f_t>& offspring,
                                                           solution_t<i_t, f_t>& sol1,
                                                           solution_t<i_t, f_t>& sol2)
{
  bool better_than_both = false;
  if (sol1.get_feasible() && sol2.get_feasible()) {
    better_than_both = offspring.get_objective() <
                       (std::min(sol1.get_objective(), sol2.get_objective()) - OBJECTIVE_EPSILON);
  } else if (sol1.get_feasible()) {
    better_than_both = offspring.get_objective() < (sol1.get_objective() - OBJECTIVE_EPSILON);
  } else if (sol2.get_feasible()) {
    better_than_both = offspring.get_objective() < (sol2.get_objective() - OBJECTIVE_EPSILON);
  } else {
    better_than_both = offspring.get_feasible();
  }
  if (offspring.get_feasible() && better_than_both) {
    context.settings.benchmark_info_ptr->last_improvement_after_recombination =
      timer.elapsed_time();
  }
}

template <typename i_t, typename f_t>
std::pair<solution_t<i_t, f_t>, solution_t<i_t, f_t>>
diversity_manager_t<i_t, f_t>::recombine_and_local_search(solution_t<i_t, f_t>& sol1,
                                                          solution_t<i_t, f_t>& sol2,
                                                          recombiner_enum_t recombiner_type)
{
  raft::common::nvtx::range fun_scope("recombine_and_local_search");
  CUOPT_LOG_DEBUG("Recombining sol cost:feas %f : %d and %f : %d",
                  sol1.get_quality(population.weights),
                  sol1.get_feasible(),
                  sol2.get_quality(population.weights),
                  sol2.get_feasible());
  double best_objective_of_parents  = std::min(sol1.get_objective(), sol2.get_objective());
  bool at_least_one_parent_feasible = sol1.get_feasible() || sol2.get_feasible();
  // randomly choose among 3 recombiners
  auto [offspring, success] = recombine(sol1, sol2, recombiner_type);
  if (!success) {
    // add the attempt
    mab_recombiner.add_mab_reward(static_cast<int>(recombine_stats.get_last_attempt()),
                                  std::numeric_limits<double>::lowest(),
                                  std::numeric_limits<double>::lowest(),
                                  std::numeric_limits<double>::max(),
                                  recombiner_work_normalized_reward_t(0.0));
    return std::make_pair(solution_t<i_t, f_t>(sol1), solution_t<i_t, f_t>(sol2));
  }
  cuopt_assert(population.test_invariant(), "");
  cuopt_func_call(offspring.test_variable_bounds(false));
  CUOPT_LOG_DEBUG("Recombiner offspring sol cost:feas %f : %d",
                  offspring.get_quality(population.weights),
                  offspring.get_feasible());
  cuopt_assert(offspring.test_number_all_integer(), "All must be integers before LS");
  bool feasibility_before = offspring.get_feasible();
  ls_config_t<i_t, f_t> ls_config;
  ls_config.best_objective_of_parents    = best_objective_of_parents;
  ls_config.at_least_one_parent_feasible = at_least_one_parent_feasible;
  success = this->run_local_search(offspring, population.weights, timer, ls_config);
  if (!success) {
    // add the attempt
    mab_recombiner.add_mab_reward(static_cast<int>(recombine_stats.get_last_attempt()),
                                  std::numeric_limits<double>::lowest(),
                                  std::numeric_limits<double>::lowest(),
                                  std::numeric_limits<double>::max(),
                                  recombiner_work_normalized_reward_t(0.0));
    return std::make_pair(solution_t<i_t, f_t>(sol1), solution_t<i_t, f_t>(sol2));
  }
  cuopt_assert(offspring.test_number_all_integer(), "All must be integers after LS");
  cuopt_assert(population.test_invariant(), "");
  offspring.compute_feasibility();
  CUOPT_LOG_DEBUG("After LS offspring sol cost:feas %f : %d",
                  offspring.get_quality(population.weights),
                  offspring.get_feasible());
  cuopt_assert(population.test_invariant(), "");
  // run LP with the vars
  solution_t<i_t, f_t> lp_offspring(offspring);
  cuopt_assert(population.test_invariant(), "");
  cuopt_assert(lp_offspring.test_number_all_integer(), "All must be integers before LP");
  f_t lp_run_time = offspring.get_feasible() ? diversity_config.lp_run_time_if_feasible
                                             : diversity_config.lp_run_time_if_infeasible;
  lp_run_time     = std::min(lp_run_time, timer.remaining_time());
  relaxed_lp_settings_t lp_settings;
  lp_settings.time_limit              = lp_run_time;
  lp_settings.tolerance               = context.settings.tolerances.absolute_tolerance;
  lp_settings.return_first_feasible   = false;
  lp_settings.save_state              = true;
  lp_settings.per_constraint_residual = true;
  run_lp_with_vars_fixed(*lp_offspring.problem_ptr,
                         lp_offspring,
                         lp_offspring.problem_ptr->integer_indices,
                         lp_settings,
                         &ls.constraint_prop.bounds_update,
                         true /* check fixed assignment is feasible */,
                         true /* use integer fixed problem */);
  cuopt_assert(population.test_invariant(), "");
  cuopt_assert(lp_offspring.test_number_all_integer(), "All must be integers after LP");
  f_t lp_qual = lp_offspring.get_quality(population.weights);
  CUOPT_LOG_DEBUG("After LP offspring sol cost:feas %f : %d", lp_qual, lp_offspring.get_feasible());
  f_t offspring_qual = std::min(offspring.get_quality(population.weights), lp_qual);
  recombine_stats.update_improve_stats(
    offspring_qual, sol1.get_quality(population.weights), sol2.get_quality(population.weights));
  f_t best_quality_of_parents =
    std::min(sol1.get_quality(population.weights), sol2.get_quality(population.weights));
  mab_recombiner.add_mab_reward(
    static_cast<int>(recombine_stats.get_last_attempt()),
    best_quality_of_parents,
    population.best().get_quality(population.weights),
    offspring_qual,
    recombiner_work_normalized_reward_t(recombine_stats.get_last_recombiner_time()));
  mab_ls.add_mab_reward(mab_ls_config_t<i_t, f_t>::last_ls_mab_option,
                        best_quality_of_parents,
                        population.best_feasible().get_quality(population.weights),
                        offspring_qual,
                        ls_work_normalized_reward_t(mab_ls_config_t<i_t, f_t>::last_lm_config));
  if (context.settings.benchmark_info_ptr != nullptr) {
    check_better_than_both(offspring, sol1, sol2);
    check_better_than_both(lp_offspring, sol1, sol2);
  }
  return std::make_pair(std::move(offspring), std::move(lp_offspring));
}

template <typename i_t, typename f_t>
std::pair<solution_t<i_t, f_t>, bool> diversity_manager_t<i_t, f_t>::recombine(
  solution_t<i_t, f_t>& a, solution_t<i_t, f_t>& b, recombiner_enum_t recombiner_type)
{
  recombiner_enum_t recombiner;
  if (run_only_ls_recombiner) {
    recombiner = recombiner_enum_t::LINE_SEGMENT;
  } else if (run_only_bp_recombiner) {
    recombiner = recombiner_enum_t::BOUND_PROP;
  } else if (run_only_fp_recombiner) {
    recombiner = recombiner_enum_t::FP;
  } else if (run_only_sub_mip_recombiner) {
    recombiner = recombiner_enum_t::SUB_MIP;
  } else {
    // only run the given recombiner unless it is defult
    if (recombiner_type == recombiner_enum_t::SIZE) {
      recombiner = static_cast<recombiner_enum_t>(mab_recombiner.select_mab_option());
    } else {
      recombiner = recombiner_type;
    }
  }
  recombine_stats.add_attempt((recombiner_enum_t)recombiner);
  recombine_stats.start_recombiner_time();
  // Refactored code using a switch statement
  switch (recombiner) {
    case recombiner_enum_t::BOUND_PROP: {
      auto [sol, success] = bound_prop_recombiner.recombine(a, b, population.weights);
      recombine_stats.stop_recombiner_time();
      if (success) { recombine_stats.add_success(); }
      return std::make_pair(sol, success);
    }
    case recombiner_enum_t::FP: {
      auto [sol, success] = fp_recombiner.recombine(a, b, population.weights);
      recombine_stats.stop_recombiner_time();
      if (success) { recombine_stats.add_success(); }
      return std::make_pair(sol, success);
    }
    case recombiner_enum_t::LINE_SEGMENT: {
      auto [sol, success] = line_segment_recombiner.recombine(a, b, population.weights);
      recombine_stats.stop_recombiner_time();
      if (success) { recombine_stats.add_success(); }
      return std::make_pair(sol, success);
    }
    case recombiner_enum_t::SUB_MIP: {
      auto [sol, success] = sub_mip_recombiner.recombine(a, b, population.weights);
      recombine_stats.stop_recombiner_time();
      if (success) { recombine_stats.add_success(); }
      return std::make_pair(sol, success);
    }
    case recombiner_enum_t::SIZE: {
      CUOPT_LOG_ERROR("Invalid or unhandled recombiner type: %d", recombiner);
      return std::make_pair(solution_t<i_t, f_t>(a), false);
    }
  }
  CUOPT_LOG_ERROR("Invalid or unhandled recombiner type: %d", recombiner);
  return std::make_pair(solution_t<i_t, f_t>(a), false);
}

template <typename i_t, typename f_t>
void diversity_manager_t<i_t, f_t>::set_simplex_solution(const std::vector<f_t>& solution,
                                                         const std::vector<f_t>& dual_solution,
                                                         f_t objective)
{
  CUOPT_LOG_DEBUG("Setting simplex solution with objective %f", objective);
  using sol_t = solution_t<i_t, f_t>;
  hipSetDevice(context.handle_ptr->get_device());
  context.handle_ptr->sync_stream();
  cuopt_func_call(sol_t new_sol(*problem_ptr));
  cuopt_assert(new_sol.assignment.size() == solution.size(), "Assignment size mismatch");
  cuopt_assert(problem_ptr->n_constraints == dual_solution.size(), "Dual assignment size mismatch");
  cuopt_func_call(new_sol.copy_new_assignment(solution));
  cuopt_func_call(new_sol.compute_feasibility());
  cuopt_assert(integer_equal(new_sol.get_user_objective(), objective, 1e-3), "Objective mismatch");
  std::lock_guard<std::mutex> lock(relaxed_solution_mutex);
  simplex_solution_exists = true;
  global_concurrent_halt  = 1;
  // it is safe to use lp_optimal_solution while executing the copy operation
  // the operations are ordered as long as they are on the same stream
  raft::copy(
    lp_optimal_solution.data(), solution.data(), solution.size(), context.handle_ptr->get_stream());
  raft::copy(lp_dual_optimal_solution.data(),
             dual_solution.data(),
             dual_solution.size(),
             context.handle_ptr->get_stream());
  set_new_user_bound(objective);
  context.handle_ptr->sync_stream();
}

#if MIP_INSTANTIATE_FLOAT
template class diversity_manager_t<int, float>;
#endif

#if MIP_INSTANTIATE_DOUBLE
template class diversity_manager_t<int, double>;
#endif

}  // namespace cuopt::linear_programming::detail
