/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuopt/error.hpp>

#include <linear_programming/cusparse_view.hpp>
#include <linear_programming/utils.cuh>
#include <mip/mip_constants.hpp>

#include <raft/sparse/detail/cusparse_macros.h>
#include <raft/sparse/detail/cusparse_wrappers.h>
#include <raft/sparse/linalg/transpose.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <dlfcn.h>

namespace cuopt::linear_programming::detail {

#define CUDA_VER_12_4_UP (CUDART_VERSION >= 12040)

#if CUDA_VER_12_4_UP
struct dynamic_load_runtime {
  static void* get_cusparse_runtime_handle()
  {
    auto close_cudart = [](void* handle) { ::dlclose(handle); };
    auto open_cudart  = []() {
      ::dlerror();
      int major_version;
      RAFT_CUSPARSE_TRY(cusparseGetProperty(libraryPropertyType_t::MAJOR_VERSION, &major_version));
      const std::string libname_ver_o = "libcusparse.so." + std::to_string(major_version) + ".0";
      const std::string libname_ver   = "libcusparse.so." + std::to_string(major_version);
      const std::string libname       = "libcusparse.so";

      auto ptr = ::dlopen(libname_ver_o.c_str(), RTLD_LAZY);
      if (!ptr) { ptr = ::dlopen(libname_ver.c_str(), RTLD_LAZY); }
      if (!ptr) { ptr = ::dlopen(libname.c_str(), RTLD_LAZY); }
      if (ptr) { return ptr; }

      EXE_CUOPT_FAIL("Unable to dlopen cusparse");
    };
    static std::unique_ptr<void, decltype(close_cudart)> cudart_handle{open_cudart(), close_cudart};
    return cudart_handle.get();
  }

  template <typename... Args>
  using function_sig = std::add_pointer_t<hipsparseStatus_t(Args...)>;

  template <typename signature>
  static std::optional<signature> function(const char* func_name)
  {
    auto* runtime = get_cusparse_runtime_handle();
    auto* handle  = ::dlsym(runtime, func_name);
    if (!handle) { return std::nullopt; }
    auto* function_ptr = reinterpret_cast<signature>(handle);
    return std::optional<signature>(function_ptr);
  }
};

template <typename... Args>
using cusparse_sig = dynamic_load_runtime::function_sig<Args...>;

using cusparseSpMV_preprocess_sig = cusparse_sig<hipsparseHandle_t,
                                                 hipsparseOperation_t,
                                                 const void*,
                                                 hipsparseConstSpMatDescr_t,
                                                 hipsparseConstDnVecDescr_t,
                                                 const void*,
                                                 hipsparseDnVecDescr_t,
                                                 hipDataType,
                                                 hipsparseSpMVAlg_t,
                                                 void*>;

// This is tmp until it's added to raft
template <
  typename T,
  typename std::enable_if_t<std::is_same_v<T, float> || std::is_same_v<T, double>>* = nullptr>
void my_cusparsespmv_preprocess(hipsparseHandle_t handle,
                                hipsparseOperation_t opA,
                                const T* alpha,
                                hipsparseConstSpMatDescr_t matA,
                                hipsparseConstDnVecDescr_t vecX,
                                const T* beta,
                                hipsparseDnVecDescr_t vecY,
                                hipsparseSpMVAlg_t alg,
                                void* externalBuffer,
                                hipStream_t stream)
{
  auto constexpr float_type = []() constexpr {
    if constexpr (std::is_same_v<T, float>) {
      return HIP_R_32F;
    } else if constexpr (std::is_same_v<T, double>) {
      return HIP_R_64F;
    }
  }();

  // There can be a missmatch between compiled CUDA version and the runtime CUDA version
  // Since cusparse is only available post >= 12.4 we need to use dlsym to make sure the symbol is
  // present at runtime
  static const auto func =
    dynamic_load_runtime::function<cusparseSpMV_preprocess_sig>("hipsparseSpMV_preprocess");
  if (func.has_value()) {
    RAFT_CUSPARSE_TRY(hipsparseSetStream(handle, stream));
    RAFT_CUSPARSE_TRY(
      (*func)(handle, opA, alpha, matA, vecX, beta, vecY, float_type, alg, externalBuffer));
  }
}
#endif

// TODO add proper checking
#if CUDA_VER_12_4_UP
template <
  typename T,
  typename std::enable_if_t<std::is_same_v<T, float> || std::is_same_v<T, double>>* = nullptr>
hipsparseStatus_t my_cusparsespmm_preprocess(hipsparseHandle_t handle,
                                         hipsparseOperation_t opA,
                                         hipsparseOperation_t opB,
                                         const T* alpha,
                                         const hipsparseSpMatDescr_t matA,
                                         const hipsparseDnMatDescr_t matB,
                                         const T* beta,
                                         const hipsparseDnMatDescr_t matC,
                                         hipsparseSpMMAlg_t alg,
                                         void* externalBuffer,
                                         hipStream_t stream)
{
  auto constexpr float_type = []() constexpr {
    if constexpr (std::is_same_v<T, float>) {
      return HIP_R_32F;
    } else if constexpr (std::is_same_v<T, double>) {
      return HIP_R_64F;
    }
  }();
  CUSPARSE_CHECK(hipsparseSetStream(handle, stream));
  return hipsparseSpMM_preprocess(
    handle, opA, opB, alpha, matA, matB, beta, matC, float_type, alg, externalBuffer);
}
#endif

// This cstr is used in pdhg
// A_T is owned by the scaled problem
// It was already transposed in the scaled_problem version
template <typename i_t, typename f_t>
cusparse_view_t<i_t, f_t>::cusparse_view_t(
  raft::handle_t const* handle_ptr,
  const problem_t<i_t, f_t>& op_problem_scaled,
  saddle_point_state_t<i_t, f_t>& current_saddle_point_state,
  rmm::device_uvector<f_t>& _tmp_primal,
  rmm::device_uvector<f_t>& _batch_tmp_primals,
  rmm::device_uvector<f_t>& _tmp_dual,
  rmm::device_uvector<f_t>& _potential_next_dual_solution)
  : handle_ptr_(handle_ptr),
    A{},
    A_T{},
    c{},
    primal_solution{},
    dual_solution{},
    primal_gradient{},
    dual_gradient{},
    current_AtY{},
    next_AtY{},
    potential_next_dual_solution{},
    tmp_primal{},
    tmp_dual{},
    A_T_{op_problem_scaled.reverse_coefficients},
    A_T_offsets_{op_problem_scaled.reverse_offsets},
    A_T_indices_{op_problem_scaled.reverse_constraints},
    buffer_non_transpose{0, handle_ptr->get_stream()},
    buffer_transpose{0, handle_ptr->get_stream()},
    buffer_transpose_batch{0, handle_ptr->get_stream()},
    buffer_non_transpose_batch{0, handle_ptr->get_stream()},
    A_{op_problem_scaled.coefficients},
    A_offsets_{op_problem_scaled.offsets},
    A_indices_{op_problem_scaled.variables}
{
  raft::common::nvtx::range fun_scope("Initializing cuSparse view");

#ifdef PDLP_DEBUG_MODE
  RAFT_CUDA_TRY(hipDeviceSynchronize());
  std::cout << "PDHG cusparse view init" << std::endl;
#endif

  // setup cusparse view
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatecsr(
    &A,
    op_problem_scaled.n_constraints,
    op_problem_scaled.n_variables,
    op_problem_scaled.nnz,
    const_cast<i_t*>(op_problem_scaled.offsets.data()),
    const_cast<i_t*>(op_problem_scaled.variables.data()),
    const_cast<f_t*>(op_problem_scaled.coefficients.data())));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatecsr(&A_T,
                                                            op_problem_scaled.n_variables,
                                                            op_problem_scaled.n_constraints,
                                                            op_problem_scaled.nnz,
                                                            const_cast<i_t*>(A_T_offsets_.data()),
                                                            const_cast<i_t*>(A_T_indices_.data()),
                                                            const_cast<f_t*>(A_T_.data())));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &c,
    op_problem_scaled.n_variables,
    const_cast<f_t*>(op_problem_scaled.objective_coefficients.data())));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &primal_solution,
    op_problem_scaled.n_variables,
    current_saddle_point_state.get_primal_solution().data()));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &dual_solution,
    op_problem_scaled.n_constraints,
    current_saddle_point_state.get_dual_solution().data()));

  if (true) {
    RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednmat(
      &batch_dual_solutions,
      op_problem_scaled.n_constraints,
      (0 + 1)/*@@*/,
      (0 + 1)/*@@*/,
      current_saddle_point_state.batch_dual_solutions_.data(),
      HIPSPARSE_ORDER_ROW));
    RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednmat(
      &batch_current_AtYs,
      op_problem_scaled.n_variables,
      (0 + 1)/*@@*/,
      (0 + 1)/*@@*/,
      current_saddle_point_state.batch_current_AtYs_.data(),
      HIPSPARSE_ORDER_ROW));
    RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednmat(
      &batch_tmp_primals,
      op_problem_scaled.n_variables,
      (0 + 1)/*@@*/,
      (0 + 1)/*@@*/,
      _batch_tmp_primals.data(),
      HIPSPARSE_ORDER_ROW));
    RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednmat(
      &batch_dual_gradients,
      op_problem_scaled.n_constraints,
      (0 + 1)/*@@*/,
      (0 + 1)/*@@*/,
      current_saddle_point_state.batch_dual_gradients_.data(),
      HIPSPARSE_ORDER_ROW));
  }

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &primal_gradient,
    op_problem_scaled.n_variables,
    current_saddle_point_state.get_primal_gradient().data()));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &dual_gradient,
    op_problem_scaled.n_constraints,
    current_saddle_point_state.get_dual_gradient().data()));

  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsecreatednvec(&current_AtY,
                                              op_problem_scaled.n_variables,
                                              current_saddle_point_state.get_current_AtY().data()));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &next_AtY, op_problem_scaled.n_variables, current_saddle_point_state.get_next_AtY().data()));

  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsecreatednvec(&potential_next_dual_solution,
                                              op_problem_scaled.n_constraints,
                                              _potential_next_dual_solution.data()));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &tmp_primal, op_problem_scaled.n_variables, _tmp_primal.data()));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &tmp_dual, op_problem_scaled.n_constraints, _tmp_dual.data()));

  const rmm::device_scalar<f_t> alpha{1, handle_ptr->get_stream()};
  const rmm::device_scalar<f_t> beta{1, handle_ptr->get_stream()};
  size_t buffer_size_non_transpose = 0;
  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsespmv_buffersize(handle_ptr_->get_cusparse_handle(),
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  alpha.data(),
                                                  A,
                                                  c,
                                                  beta.data(),
                                                  dual_solution,
                                                  HIPSPARSE_SPMV_CSR_ALG2,
                                                  &buffer_size_non_transpose,
                                                  handle_ptr->get_stream()));
  buffer_non_transpose.resize(buffer_size_non_transpose, handle_ptr->get_stream());

  size_t buffer_size_transpose = 0;
  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsespmv_buffersize(handle_ptr_->get_cusparse_handle(),
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  alpha.data(),
                                                  A_T,
                                                  dual_solution,
                                                  beta.data(),
                                                  c,
                                                  HIPSPARSE_SPMV_CSR_ALG2,
                                                  &buffer_size_transpose,
                                                  handle_ptr->get_stream()));

  buffer_transpose.resize(buffer_size_transpose, handle_ptr->get_stream());

  if (true) {
    size_t buffer_size_transpose_batch = 0;
    RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsespmm_bufferSize(handle_ptr_->get_cusparse_handle(),
                                                                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                                   alpha.data(),
                                                                   A_T,
                                                                   batch_dual_solutions,
                                                                   beta.data(),
                                                                   batch_current_AtYs,
                                                                   HIPSPARSE_SPMM_CSR_ALG3,
                                                                   &buffer_size_transpose_batch,
                                                                   handle_ptr->get_stream()));
    buffer_transpose_batch.resize(buffer_size_transpose_batch, handle_ptr->get_stream());
    size_t buffer_size_non_transpose_batch = 0;
    RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsespmm_bufferSize(handle_ptr_->get_cusparse_handle(),
                                                                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                                   alpha.data(),
                                                                   A,
                                                                   batch_tmp_primals,
                                                                   beta.data(),
                                                                   batch_dual_gradients,
                                                                   HIPSPARSE_SPMM_CSR_ALG3,
                                                                   &buffer_size_non_transpose_batch,
                                                                   handle_ptr->get_stream()));
    buffer_non_transpose_batch.resize(buffer_size_non_transpose_batch, handle_ptr->get_stream());
  }

#if CUDA_VER_12_4_UP
  my_cusparsespmv_preprocess(handle_ptr_->get_cusparse_handle(),
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             alpha.data(),
                             A,
                             c,
                             beta.data(),
                             dual_solution,
                             HIPSPARSE_SPMV_CSR_ALG2,
                             buffer_non_transpose.data(),
                             handle_ptr->get_stream());

  my_cusparsespmv_preprocess(handle_ptr_->get_cusparse_handle(),
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             alpha.data(),
                             A_T,
                             dual_solution,
                             beta.data(),
                             c,
                             HIPSPARSE_SPMV_CSR_ALG2,
                             buffer_transpose.data(),
                             handle_ptr->get_stream());

  my_cusparsespmm_preprocess(handle_ptr_->get_cusparse_handle(),
                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          alpha.data(),
                          A_T,
                          batch_dual_solutions,
                          beta.data(), batch_current_AtYs, HIPSPARSE_SPMM_CSR_ALG3, buffer_transpose_batch.data(), handle_ptr->get_stream());

  my_cusparsespmm_preprocess(handle_ptr_->get_cusparse_handle(),
                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          alpha.data(),
                          A,
                          batch_tmp_primals,
                          beta.data(), batch_dual_gradients, HIPSPARSE_SPMM_CSR_ALG3, buffer_non_transpose_batch.data(), handle_ptr->get_stream());
#endif
}

// Used by pdlp object for current and average termination condition
// A_T is owned by the problem object and is transposed by the problem
template <typename i_t, typename f_t>
cusparse_view_t<i_t, f_t>::cusparse_view_t(raft::handle_t const* handle_ptr,
                                           const problem_t<i_t, f_t>& op_problem,
                                           rmm::device_uvector<f_t>& _primal_solution,
                                           rmm::device_uvector<f_t>& _dual_solution,
                                           rmm::device_uvector<f_t>& _tmp_primal,
                                           rmm::device_uvector<f_t>& _tmp_dual,
                                           const rmm::device_uvector<f_t>& _A_T,
                                           const rmm::device_uvector<i_t>& _A_T_offsets,
                                           const rmm::device_uvector<i_t>& _A_T_indices)
  : handle_ptr_(handle_ptr),
    A{},
    A_T{},
    c{},
    primal_solution{},
    dual_solution{},
    primal_gradient{},
    dual_gradient{},
    tmp_primal{},
    tmp_dual{},
    A_T_{_A_T},
    A_T_offsets_{_A_T_offsets},
    A_T_indices_{_A_T_indices},
    buffer_non_transpose{0, handle_ptr->get_stream()},
    buffer_transpose{0, handle_ptr->get_stream()},
    buffer_transpose_batch{0, handle_ptr->get_stream()},
    buffer_non_transpose_batch{0, handle_ptr->get_stream()},
    A_{op_problem.coefficients},
    A_offsets_{op_problem.offsets},
    A_indices_{op_problem.variables}
{
#ifdef PDLP_DEBUG_MODE
  RAFT_CUDA_TRY(hipDeviceSynchronize());
  std::cout << "PDLP cusparse view init" << std::endl;
#endif

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsesetpointermode(
    handle_ptr_->get_cusparse_handle(), HIPSPARSE_POINTER_MODE_DEVICE, handle_ptr->get_stream()));

  // setup cusparse view
  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsecreatecsr(&A,
                                            op_problem.n_constraints,
                                            op_problem.n_variables,
                                            op_problem.nnz,
                                            const_cast<i_t*>(op_problem.offsets.data()),
                                            const_cast<i_t*>(op_problem.variables.data()),
                                            const_cast<f_t*>(op_problem.coefficients.data())));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatecsr(&A_T,
                                                            op_problem.n_variables,
                                                            op_problem.n_constraints,
                                                            op_problem.nnz,
                                                            const_cast<i_t*>(A_T_offsets_.data()),
                                                            const_cast<i_t*>(A_T_indices_.data()),
                                                            const_cast<f_t*>(A_T_.data())));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &c, op_problem.n_variables, const_cast<f_t*>(op_problem.objective_coefficients.data())));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &primal_solution, op_problem.n_variables, _primal_solution.data()));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &dual_solution, op_problem.n_constraints, _dual_solution.data()));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &tmp_primal, op_problem.n_variables, _tmp_primal.data()));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &tmp_dual, op_problem.n_constraints, _tmp_dual.data()));

  const rmm::device_scalar<f_t> alpha{1, handle_ptr->get_stream()};
  const rmm::device_scalar<f_t> beta{1, handle_ptr->get_stream()};
  size_t buffer_size_non_transpose = 0;
  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsespmv_buffersize(handle_ptr_->get_cusparse_handle(),
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  alpha.data(),
                                                  A,
                                                  c,
                                                  beta.data(),
                                                  dual_solution,
                                                  HIPSPARSE_SPMV_CSR_ALG2,
                                                  &buffer_size_non_transpose,
                                                  handle_ptr->get_stream()));
  buffer_non_transpose.resize(buffer_size_non_transpose, handle_ptr->get_stream());

  size_t buffer_size_transpose = 0;
  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsespmv_buffersize(handle_ptr_->get_cusparse_handle(),
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  alpha.data(),
                                                  A_T,
                                                  dual_solution,
                                                  beta.data(),
                                                  c,
                                                  HIPSPARSE_SPMV_CSR_ALG2,
                                                  &buffer_size_transpose,
                                                  handle_ptr->get_stream()));

  buffer_transpose.resize(buffer_size_transpose, handle_ptr->get_stream());

#if CUDA_VER_12_4_UP
  my_cusparsespmv_preprocess(handle_ptr_->get_cusparse_handle(),
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             alpha.data(),
                             A,
                             c,
                             beta.data(),
                             dual_solution,
                             HIPSPARSE_SPMV_CSR_ALG2,
                             buffer_non_transpose.data(),
                             handle_ptr->get_stream());

  my_cusparsespmv_preprocess(handle_ptr_->get_cusparse_handle(),
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             alpha.data(),
                             A_T,
                             dual_solution,
                             beta.data(),
                             c,
                             HIPSPARSE_SPMV_CSR_ALG2,
                             buffer_transpose.data(),
                             handle_ptr->get_stream());
#endif
}

// Constructor used 3 times in restart strategy for trust region restart
template <typename i_t, typename f_t>
cusparse_view_t<i_t, f_t>::cusparse_view_t(
  raft::handle_t const* handle_ptr,
  const problem_t<i_t, f_t>& op_problem,  // Just used for the sizes
  const cusparse_view_t<i_t, f_t>& existing_cusparse_view,
  f_t* _primal_solution,
  f_t* _dual_solution,
  f_t* _primal_gradient,
  f_t* _dual_gradient)
  : handle_ptr_(handle_ptr),
    c(existing_cusparse_view.c),
    primal_solution{},
    dual_solution{},
    primal_gradient{},
    dual_gradient{},
    tmp_primal(existing_cusparse_view.tmp_primal),
    tmp_dual(existing_cusparse_view.tmp_dual),
    buffer_non_transpose{0, handle_ptr->get_stream()},
    buffer_transpose{0, handle_ptr->get_stream()},
    buffer_transpose_batch{0, handle_ptr->get_stream()},
    buffer_non_transpose_batch{0, handle_ptr->get_stream()},
    A_T_{existing_cusparse_view.A_T_},                  // Need to be init but not used
    A_T_offsets_{existing_cusparse_view.A_T_offsets_},  // Need to be init but not used
    A_T_indices_{existing_cusparse_view.A_T_indices_},  // Need to be init but not used
    A_{existing_cusparse_view.A_},
    A_offsets_{existing_cusparse_view.A_offsets_},
    A_indices_{existing_cusparse_view.A_indices_}
{
#ifdef PDLP_DEBUG_MODE
  RAFT_CUDA_TRY(hipDeviceSynchronize());
  std::cout << "Restart Strategy cusparse view init" << std::endl;
#endif

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsesetpointermode(
    handle_ptr_->get_cusparse_handle(), HIPSPARSE_POINTER_MODE_DEVICE, handle_ptr->get_stream()));

  // Need to reinstanciate the cuSparse views
  // Copying them from the existing cuSparse view is a bad practice and creates segfault post
  // CUDA 12.4 Using the saved pointer of the existing cusparse view to make sure we capture the
  // correct pointer
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatecsr(&A,
                                                            op_problem.n_constraints,
                                                            op_problem.n_variables,
                                                            op_problem.nnz,
                                                            const_cast<i_t*>(A_offsets_.data()),
                                                            const_cast<i_t*>(A_indices_.data()),
                                                            const_cast<f_t*>(A_.data())));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatecsr(
    &A_T,
    op_problem.n_variables,
    op_problem.n_constraints,
    op_problem.nnz,
    const_cast<i_t*>(existing_cusparse_view.A_T_offsets_.data()),
    const_cast<i_t*>(existing_cusparse_view.A_T_indices_.data()),
    const_cast<f_t*>(existing_cusparse_view.A_T_.data())));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &primal_solution, op_problem.n_variables, _primal_solution));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &dual_solution, op_problem.n_constraints, _dual_solution));

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &primal_gradient, op_problem.n_variables, _primal_gradient));
  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsecreatednvec(
    &dual_gradient, op_problem.n_constraints, _dual_gradient));

  const rmm::device_scalar<f_t> alpha{1, handle_ptr->get_stream()};
  const rmm::device_scalar<f_t> beta{1, handle_ptr->get_stream()};
  size_t buffer_size_non_transpose = 0;
  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsespmv_buffersize(handle_ptr_->get_cusparse_handle(),
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  alpha.data(),
                                                  A,
                                                  c,
                                                  beta.data(),
                                                  dual_solution,
                                                  HIPSPARSE_SPMV_CSR_ALG2,
                                                  &buffer_size_non_transpose,
                                                  handle_ptr->get_stream()));
  buffer_non_transpose.resize(buffer_size_non_transpose, handle_ptr->get_stream());

  size_t buffer_size_transpose = 0;
  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsespmv_buffersize(handle_ptr_->get_cusparse_handle(),
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  alpha.data(),
                                                  A_T,
                                                  dual_solution,
                                                  beta.data(),
                                                  c,
                                                  HIPSPARSE_SPMV_CSR_ALG2,
                                                  &buffer_size_transpose,
                                                  handle_ptr->get_stream()));

  buffer_transpose.resize(buffer_size_transpose, handle_ptr->get_stream());

#if CUDA_VER_12_4_UP
  my_cusparsespmv_preprocess(handle_ptr_->get_cusparse_handle(),
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             alpha.data(),
                             A,
                             c,
                             beta.data(),
                             dual_solution,
                             HIPSPARSE_SPMV_CSR_ALG2,
                             buffer_non_transpose.data(),
                             handle_ptr->get_stream());

  my_cusparsespmv_preprocess(handle_ptr_->get_cusparse_handle(),
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             alpha.data(),
                             A_T,
                             dual_solution,
                             beta.data(),
                             c,
                             HIPSPARSE_SPMV_CSR_ALG2,
                             buffer_transpose.data(),
                             handle_ptr->get_stream());
#endif
}

// Empty constructor used in kkt restart to save memory
template <typename i_t, typename f_t>
cusparse_view_t<i_t, f_t>::cusparse_view_t(
  raft::handle_t const* handle_ptr,
  const rmm::device_uvector<f_t>& dummy_float,  // Empty just to init the const&
  const rmm::device_uvector<i_t>& dummy_int     // Empty just to init the const&
  )
  : handle_ptr_(handle_ptr),
    buffer_non_transpose{0, handle_ptr->get_stream()},
    buffer_transpose{0, handle_ptr->get_stream()},
    buffer_transpose_batch{0, handle_ptr->get_stream()},
    buffer_non_transpose_batch{0, handle_ptr->get_stream()},
    A_T_(dummy_float),
    A_T_offsets_(dummy_int),
    A_T_indices_(dummy_int),
    A_(dummy_float),
    A_offsets_(dummy_int),
    A_indices_(dummy_int)
{
}

#if MIP_INSTANTIATE_FLOAT
template class cusparse_view_t<int, float>;
#endif
#if MIP_INSTANTIATE_DOUBLE
template class cusparse_view_t<int, double>;
#endif

}  // namespace cuopt::linear_programming::detail
