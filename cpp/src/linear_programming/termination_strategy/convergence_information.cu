#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2025 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <linear_programming/pdlp_constants.hpp>
#include <linear_programming/termination_strategy/convergence_information.hpp>
#include <linear_programming/utils.cuh>
#include <mip/mip_constants.hpp>

#include <cuopt/linear_programming/pdlp/solver_settings.hpp>

#include <raft/sparse/detail/cusparse_wrappers.h>
#include <raft/common/nvtx.hpp>
#include <raft/linalg/binary_op.cuh>
#include <raft/linalg/detail/cublas_wrappers.hpp>
#include <raft/linalg/eltwise.cuh>
#include <raft/linalg/ternary_op.cuh>
#include <raft/util/cuda_utils.cuh>

#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>

#include <hipcub/hipcub.hpp>

namespace cuopt::linear_programming::detail {
template <typename i_t, typename f_t>
convergence_information_t<i_t, f_t>::convergence_information_t(
  raft::handle_t const* handle_ptr,
  problem_t<i_t, f_t>& op_problem,
  cusparse_view_t<i_t, f_t>& cusparse_view,
  i_t primal_size,
  i_t dual_size)
  : handle_ptr_(handle_ptr),
    stream_view_(handle_ptr_->get_stream()),
    primal_size_h_(primal_size),
    dual_size_h_(dual_size),
    problem_ptr(&op_problem),
    op_problem_cusparse_view_(cusparse_view),
    l2_norm_primal_linear_objective_{0.0, stream_view_},
    l2_norm_primal_right_hand_side_{0.0, stream_view_},
    primal_objective_{0.0, stream_view_},
    dual_objective_{0.0, stream_view_},
    reduced_cost_dual_objective_{0.0, stream_view_},
    l2_primal_residual_{0.0, stream_view_},
    l2_dual_residual_{0.0, stream_view_},
    linf_primal_residual_{0.0, stream_view_},
    linf_dual_residual_{0.0, stream_view_},
    nb_violated_constraints_{0, stream_view_},
    gap_{0.0, stream_view_},
    abs_objective_{0.0, stream_view_},
    l2_primal_variable_{0.0, stream_view_},
    l2_dual_variable_{0.0, stream_view_},
    primal_residual_{static_cast<size_t>(dual_size_h_), stream_view_},
    dual_residual_{static_cast<size_t>(primal_size_h_), stream_view_},
    reduced_cost_{static_cast<size_t>(primal_size_h_), stream_view_},
    bound_value_{static_cast<size_t>(std::max(primal_size_h_, dual_size_h_)), stream_view_},
    reusable_device_scalar_value_1_{1.0, stream_view_},
    reusable_device_scalar_value_0_{0.0, stream_view_},
    reusable_device_scalar_value_neg_1_{-1.0, stream_view_}
{
  combine_constraint_bounds(
    *problem_ptr,
    primal_residual_);  // primal_residual_ will contain abs max of bounds when
                        // finite, otherwise 0 //just reused allocated mem here

  // constant throughout solving, so precompute
  my_l2_norm<i_t, f_t>(
    problem_ptr->objective_coefficients, l2_norm_primal_linear_objective_, handle_ptr_);
  my_l2_norm<i_t, f_t>(primal_residual_, l2_norm_primal_right_hand_side_, handle_ptr_);

  void* d_temp_storage        = NULL;
  size_t temp_storage_bytes_1 = 0;
  hipcub::DeviceReduce::Sum(d_temp_storage,
                         temp_storage_bytes_1,
                         bound_value_.begin(),
                         dual_objective_.data(),
                         dual_size_h_,
                         stream_view_);

  size_t temp_storage_bytes_2 = 0;
  hipcub::DeviceReduce::Sum(d_temp_storage,
                         temp_storage_bytes_2,
                         bound_value_.begin(),
                         reduced_cost_dual_objective_.data(),
                         primal_size_h_,
                         stream_view_);

  size_of_buffer_       = std::max({temp_storage_bytes_1, temp_storage_bytes_2});
  this->rmm_tmp_buffer_ = rmm::device_buffer{size_of_buffer_, stream_view_};

  RAFT_CUDA_TRY(hipMemsetAsync(
    primal_residual_.data(), 0.0, sizeof(f_t) * primal_residual_.size(), stream_view_));
  RAFT_CUDA_TRY(
    hipMemsetAsync(dual_residual_.data(), 0.0, sizeof(f_t) * dual_residual_.size(), stream_view_));
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::set_relative_dual_tolerance_factor(
  f_t dual_tolerance_factor)
{
  l2_norm_primal_linear_objective_.set_value_async(dual_tolerance_factor, stream_view_);
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::set_relative_primal_tolerance_factor(
  f_t primal_tolerance_factor)
{
  l2_norm_primal_right_hand_side_.set_value_async(primal_tolerance_factor, stream_view_);
}

template <typename i_t, typename f_t>
f_t convergence_information_t<i_t, f_t>::get_relative_dual_tolerance_factor() const
{
  return l2_norm_primal_linear_objective_.value(stream_view_);
}

template <typename i_t, typename f_t>
f_t convergence_information_t<i_t, f_t>::get_relative_primal_tolerance_factor() const
{
  return l2_norm_primal_right_hand_side_.value(stream_view_);
}

template <typename i_t, typename f_t>
__global__ void compute_remaining_stats_kernel(
  typename convergence_information_t<i_t, f_t>::view_t convergence_information_view)
{
  if (threadIdx.x + blockIdx.x * blockDim.x > 0) { return; }

  *convergence_information_view.gap = raft::abs(*convergence_information_view.primal_objective -
                                                *convergence_information_view.dual_objective);
  *convergence_information_view.abs_objective =
    raft::abs(*convergence_information_view.primal_objective) +
    raft::abs(*convergence_information_view.dual_objective);
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::compute_convergence_information(
  pdhg_solver_t<i_t, f_t>& current_pdhg_solver,
  rmm::device_uvector<f_t>& primal_iterate,
  rmm::device_uvector<f_t>& dual_iterate,
  const rmm::device_uvector<f_t>& combined_bounds,
  const rmm::device_uvector<f_t>& objective_coefficients,
  const pdlp_solver_settings_t<i_t, f_t>& settings)
{
  raft::common::nvtx::range fun_scope("compute_convergence_information");

  compute_primal_residual(op_problem_cusparse_view_, current_pdhg_solver.get_dual_tmp_resource());
  compute_primal_objective(primal_iterate);
  my_l2_norm<i_t, f_t>(primal_residual_, l2_primal_residual_, handle_ptr_);
  // If per_constraint_residual is false we still need to perform the l2 since it's used in kkt
  if (settings.per_constraint_residual) {
    // Compute the linf of (residual_i - rel * b_i)
    thrust::device_ptr<f_t> result_ptr(linf_primal_residual_.data());
    const f_t neutral = f_t(0.0);

    if (settings.save_best_primal_so_far) {
      const i_t zero_int = 0;
      nb_violated_constraints_.set_value_async(zero_int, handle_ptr_->get_stream());
      *result_ptr = thrust::transform_reduce(
        handle_ptr_->get_thrust_policy(),
        thrust::make_zip_iterator(primal_residual_.cbegin(), combined_bounds.cbegin()),
        thrust::make_zip_iterator(primal_residual_.cend(), combined_bounds.cend()),
        relative_residual_t<i_t, f_t>{settings.tolerances.relative_primal_tolerance},
        neutral,
        thrust::maximum<f_t>());
    } else {
      *result_ptr = thrust::transform_reduce(
        handle_ptr_->get_thrust_policy(),
        thrust::make_zip_iterator(primal_residual_.cbegin(), combined_bounds.cbegin()),
        thrust::make_zip_iterator(primal_residual_.cend(), combined_bounds.cend()),
        relative_residual_t<i_t, f_t>{settings.tolerances.relative_primal_tolerance},
        neutral,
        thrust::maximum<f_t>());
    }
  }
  my_l2_norm<i_t, f_t>(primal_iterate, l2_primal_variable_, handle_ptr_);

  compute_dual_residual(
    op_problem_cusparse_view_, current_pdhg_solver.get_primal_tmp_resource(), primal_iterate);
  compute_dual_objective(dual_iterate);
  my_l2_norm<i_t, f_t>(dual_residual_, l2_dual_residual_, handle_ptr_);
  // If per_constraint_residual is false we still need to perform the l2 since it's used in kkt
  if (settings.per_constraint_residual) {
    // Compute the linf of (residual_i - rel * c_i)
    thrust::device_ptr<f_t> result_ptr(linf_dual_residual_.data());
    const f_t neutral = f_t(0.0);

    *result_ptr = thrust::transform_reduce(
      handle_ptr_->get_thrust_policy(),
      thrust::make_zip_iterator(dual_residual_.cbegin(), objective_coefficients.cbegin()),
      thrust::make_zip_iterator(dual_residual_.cend(), objective_coefficients.cend()),
      relative_residual_t<i_t, f_t>{settings.tolerances.relative_dual_tolerance},
      neutral,
      thrust::maximum<f_t>());
  }
  my_l2_norm<i_t, f_t>(dual_iterate, l2_dual_variable_, handle_ptr_);

  compute_remaining_stats_kernel<i_t, f_t><<<1, 1, 0, stream_view_>>>(this->view());
  RAFT_CUDA_TRY(hipPeekAtLastError());

  //  cleanup for next termination evaluation
  RAFT_CUDA_TRY(hipMemsetAsync(
    primal_residual_.data(), 0.0, sizeof(f_t) * primal_residual_.size(), stream_view_));
  RAFT_CUDA_TRY(
    hipMemsetAsync(dual_residual_.data(), 0.0, sizeof(f_t) * dual_residual_.size(), stream_view_));
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::compute_primal_residual(
  cusparse_view_t<i_t, f_t>& cusparse_view, rmm::device_uvector<f_t>& tmp_dual)
{
  raft::common::nvtx::range fun_scope("compute_primal_residual");

  // primal_product
  RAFT_CUSPARSE_TRY(
    raft::sparse::detail::cusparsespmv(handle_ptr_->get_cusparse_handle(),
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       reusable_device_scalar_value_1_.data(),
                                       cusparse_view.A,
                                       cusparse_view.primal_solution,
                                       reusable_device_scalar_value_0_.data(),
                                       cusparse_view.tmp_dual,
                                       HIPSPARSE_SPMV_CSR_ALG2,
                                       (f_t*)cusparse_view.buffer_non_transpose.data(),
                                       stream_view_));

  // The constraint bound violations for the first part of the residual
  raft::linalg::ternaryOp<f_t, violation<f_t>>(primal_residual_.data(),
                                               tmp_dual.data(),
                                               problem_ptr->constraint_lower_bounds.data(),
                                               problem_ptr->constraint_upper_bounds.data(),
                                               dual_size_h_,
                                               violation<f_t>(),
                                               stream_view_);
}

template <typename i_t, typename f_t>
__global__ void apply_objective_scaling_and_offset(f_t* objective,
                                                   f_t objective_scaling_factor,
                                                   f_t objective_offset)
{
  if (threadIdx.x + blockIdx.x * blockDim.x > 0) { return; }

  *objective = objective_scaling_factor * (*objective + objective_offset);
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::compute_primal_objective(
  rmm::device_uvector<f_t>& primal_solution)
{
  raft::common::nvtx::range fun_scope("compute_primal_objective");

  RAFT_CUBLAS_TRY(raft::linalg::detail::cublasdot(handle_ptr_->get_cublas_handle(),
                                                  (int)primal_size_h_,
                                                  primal_solution.data(),
                                                  primal_stride,
                                                  problem_ptr->objective_coefficients.data(),
                                                  primal_stride,
                                                  primal_objective_.data(),
                                                  stream_view_));

  // primal_objective = 1 * (primal_objective + 0) = primal_objective
  if (problem_ptr->presolve_data.objective_scaling_factor != 1 ||
      problem_ptr->presolve_data.objective_offset != 0) {
    apply_objective_scaling_and_offset<i_t, f_t>
      <<<1, 1, 0, stream_view_>>>(primal_objective_.data(),
                                  problem_ptr->presolve_data.objective_scaling_factor,
                                  problem_ptr->presolve_data.objective_offset);
    RAFT_CUDA_TRY(hipPeekAtLastError());
  }
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::compute_dual_residual(
  cusparse_view_t<i_t, f_t>& cusparse_view,
  rmm::device_uvector<f_t>& tmp_primal,
  rmm::device_uvector<f_t>& primal_solution)
{
  raft::common::nvtx::range fun_scope("compute_dual_residual");
  // compute objective product (Q*x) if QP

  // gradient is recomputed with the dual solution that has been computed since the gradient was
  // last computed
  //  c-K^Ty -> copy c to gradient first
  raft::copy(
    tmp_primal.data(), problem_ptr->objective_coefficients.data(), primal_size_h_, stream_view_);

  RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsespmv(handle_ptr_->get_cusparse_handle(),
                                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                       reusable_device_scalar_value_neg_1_.data(),
                                                       cusparse_view.A_T,
                                                       cusparse_view.dual_solution,
                                                       reusable_device_scalar_value_1_.data(),
                                                       cusparse_view.tmp_primal,
                                                       HIPSPARSE_SPMV_CSR_ALG2,
                                                       (f_t*)cusparse_view.buffer_transpose.data(),
                                                       stream_view_));

  compute_reduced_cost_from_primal_gradient(tmp_primal, primal_solution);

  // primal_gradient - reduced_costs
  raft::linalg::eltwiseSub(dual_residual_.data(),
                           tmp_primal.data(),  // primal_gradient
                           reduced_cost_.data(),
                           primal_size_h_,
                           stream_view_);
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::compute_dual_objective(
  rmm::device_uvector<f_t>& dual_solution)
{
  raft::common::nvtx::range fun_scope("compute_dual_objective");

  // for QP would need to add + problem.objective_constant - 0.5 * objective_product' *
  // primal_solution (iteration_stats.jl:186)

  // the value of y term in the objective of the dual problem, see[]
  //  (l^c)^T[y]_+ − (u^c)^T[y]_− in the dual objective

  raft::linalg::ternaryOp(bound_value_.data(),
                          dual_solution.data(),
                          problem_ptr->constraint_lower_bounds.data(),
                          problem_ptr->constraint_upper_bounds.data(),
                          dual_size_h_,
                          bound_value_reduced_cost_product<f_t>(),
                          stream_view_);

  hipcub::DeviceReduce::Sum(rmm_tmp_buffer_.data(),
                         size_of_buffer_,
                         bound_value_.begin(),
                         dual_objective_.data(),
                         dual_size_h_,
                         stream_view_);

  compute_reduced_costs_dual_objective_contribution();

  raft::linalg::eltwiseAdd(dual_objective_.data(),
                           dual_objective_.data(),
                           reduced_cost_dual_objective_.data(),
                           1,
                           stream_view_);

  // dual_objective = 1 * (dual_objective + 0) = dual_objective
  if (problem_ptr->presolve_data.objective_scaling_factor != 1 ||
      problem_ptr->presolve_data.objective_offset != 0) {
    apply_objective_scaling_and_offset<i_t, f_t>
      <<<1, 1, 0, stream_view_>>>(dual_objective_.data(),
                                  problem_ptr->presolve_data.objective_scaling_factor,
                                  problem_ptr->presolve_data.objective_offset);
    RAFT_CUDA_TRY(hipPeekAtLastError());
  }
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::compute_reduced_cost_from_primal_gradient(
  const rmm::device_uvector<f_t>& primal_gradient, const rmm::device_uvector<f_t>& primal_solution)
{
  raft::common::nvtx::range fun_scope("compute_reduced_cost_from_primal_gradient");

  raft::linalg::ternaryOp(bound_value_.data(),
                          primal_gradient.data(),
                          problem_ptr->variable_lower_bounds.data(),
                          problem_ptr->variable_upper_bounds.data(),
                          primal_size_h_,
                          bound_value_gradient<f_t>(),
                          stream_view_);

  if (pdlp_hyper_params::handle_some_primal_gradients_on_finite_bounds_as_residuals) {
    raft::linalg::ternaryOp(reduced_cost_.data(),
                            primal_solution.data(),
                            bound_value_.data(),
                            primal_gradient.data(),
                            primal_size_h_,
                            copy_gradient_if_should_be_reduced_cost<f_t>(),
                            stream_view_);
  } else {
    raft::linalg::binaryOp(reduced_cost_.data(),
                           bound_value_.data(),
                           primal_gradient.data(),
                           primal_size_h_,
                           copy_gradient_if_finite_bounds<f_t>(),
                           stream_view_);
  }
}

template <typename i_t, typename f_t>
void convergence_information_t<i_t, f_t>::compute_reduced_costs_dual_objective_contribution()
{
  raft::common::nvtx::range fun_scope("compute_reduced_costs_dual_objective_contribution");

  // if reduced cost is positive -> lower bound, negative -> upper bounds, 0 -> 0
  // if bound_val is not finite let element be -inf, otherwise bound_value*reduced_cost
  raft::linalg::ternaryOp(bound_value_.data(),
                          reduced_cost_.data(),
                          problem_ptr->variable_lower_bounds.data(),
                          problem_ptr->variable_upper_bounds.data(),
                          primal_size_h_,
                          bound_value_reduced_cost_product<f_t>(),
                          stream_view_);

  // sum over bound_value*reduced_cost, but should be -inf if any element is -inf
  hipcub::DeviceReduce::Sum(rmm_tmp_buffer_.data(),
                         size_of_buffer_,
                         bound_value_.begin(),
                         reduced_cost_dual_objective_.data(),
                         primal_size_h_,
                         stream_view_);
}

template <typename i_t, typename f_t>
rmm::device_uvector<f_t>& convergence_information_t<i_t, f_t>::get_reduced_cost()
{
  return reduced_cost_;
}

template <typename i_t, typename f_t>
const rmm::device_scalar<f_t>& convergence_information_t<i_t, f_t>::get_l2_primal_residual() const
{
  return l2_primal_residual_;
}

template <typename i_t, typename f_t>
const rmm::device_scalar<f_t>& convergence_information_t<i_t, f_t>::get_primal_objective() const
{
  return primal_objective_;
}

template <typename i_t, typename f_t>
const rmm::device_scalar<f_t>& convergence_information_t<i_t, f_t>::get_dual_objective() const
{
  return dual_objective_;
}

template <typename i_t, typename f_t>
const rmm::device_scalar<f_t>& convergence_information_t<i_t, f_t>::get_l2_dual_residual() const
{
  return l2_dual_residual_;
}

template <typename i_t, typename f_t>
const rmm::device_scalar<f_t>&
convergence_information_t<i_t, f_t>::get_relative_linf_primal_residual() const
{
  return linf_primal_residual_;
}

template <typename i_t, typename f_t>
const rmm::device_scalar<f_t>&
convergence_information_t<i_t, f_t>::get_relative_linf_dual_residual() const
{
  return linf_dual_residual_;
}

template <typename i_t, typename f_t>
const rmm::device_scalar<f_t>& convergence_information_t<i_t, f_t>::get_gap() const
{
  return gap_;
}

template <typename i_t, typename f_t>
f_t convergence_information_t<i_t, f_t>::get_relative_gap_value() const
{
  return gap_.value(stream_view_) / (f_t(1.0) + abs_objective_.value(stream_view_));
}

template <typename i_t, typename f_t>
f_t convergence_information_t<i_t, f_t>::get_relative_l2_primal_residual_value() const
{
  return l2_primal_residual_.value(stream_view_) /
         (f_t(1.0) + l2_norm_primal_right_hand_side_.value(stream_view_));
}

template <typename i_t, typename f_t>
f_t convergence_information_t<i_t, f_t>::get_relative_l2_dual_residual_value() const
{
  return l2_dual_residual_.value(stream_view_) /
         (f_t(1.0) + l2_norm_primal_linear_objective_.value(stream_view_));
}

template <typename i_t, typename f_t>
typename convergence_information_t<i_t, f_t>::view_t convergence_information_t<i_t, f_t>::view()
{
  convergence_information_t<i_t, f_t>::view_t v;
  v.primal_size = primal_size_h_;
  v.dual_size   = dual_size_h_;

  v.l2_norm_primal_linear_objective = l2_norm_primal_linear_objective_.data();
  v.l2_norm_primal_right_hand_side  = l2_norm_primal_right_hand_side_.data();

  v.primal_objective               = primal_objective_.data();
  v.dual_objective                 = dual_objective_.data();
  v.l2_primal_residual             = l2_primal_residual_.data();
  v.l2_dual_residual               = l2_dual_residual_.data();
  v.relative_l_inf_primal_residual = linf_primal_residual_.data();
  v.relative_l_inf_dual_residual   = linf_dual_residual_.data();

  v.gap           = gap_.data();
  v.abs_objective = abs_objective_.data();

  v.l2_primal_variable = l2_primal_variable_.data();
  v.l2_dual_variable   = l2_dual_variable_.data();

  v.primal_residual = primal_residual_.data();
  v.dual_residual   = dual_residual_.data();
  v.reduced_cost    = reduced_cost_.data();
  v.bound_value     = bound_value_.data();

  return v;
}

template <typename i_t, typename f_t>
typename convergence_information_t<i_t, f_t>::primal_quality_adapter_t
convergence_information_t<i_t, f_t>::to_primal_quality_adapter(
  bool is_primal_feasible) const noexcept
{
  return {is_primal_feasible,
          nb_violated_constraints_.value(stream_view_),
          l2_primal_residual_.value(stream_view_),
          primal_objective_.value(stream_view_)};
}

#if MIP_INSTANTIATE_FLOAT
template class convergence_information_t<int, float>;

template __global__ void compute_remaining_stats_kernel<int, float>(
  typename convergence_information_t<int, float>::view_t convergence_information_view);
#endif

#if MIP_INSTANTIATE_DOUBLE
template class convergence_information_t<int, double>;

template __global__ void compute_remaining_stats_kernel<int, double>(
  typename convergence_information_t<int, double>::view_t convergence_information_view);
#endif

}  // namespace cuopt::linear_programming::detail
