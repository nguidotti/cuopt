#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuopt/error.hpp>

#include <linear_programming/pdlp_constants.hpp>
#include <linear_programming/restart_strategy/weighted_average_solution.hpp>
#include <linear_programming/utils.cuh>
#include <mip/mip_constants.hpp>

#include <raft/linalg/binary_op.cuh>
#include <raft/linalg/divide.cuh>

namespace cuopt::linear_programming::detail {
template <typename i_t, typename f_t>
weighted_average_solution_t<i_t, f_t>::weighted_average_solution_t(raft::handle_t const* handle_ptr,
                                                                   i_t primal_size,
                                                                   i_t dual_size)
  : handle_ptr_(handle_ptr),
    stream_view_(handle_ptr_->get_stream()),
    primal_size_h_(primal_size),
    dual_size_h_(dual_size),
    sum_primal_solutions_{static_cast<size_t>(primal_size_h_), stream_view_},
    sum_dual_solutions_{static_cast<size_t>(dual_size_h_), stream_view_},
    sum_primal_solution_weights_{0.0, stream_view_},
    sum_dual_solution_weights_{0.0, stream_view_},
    iterations_since_last_restart_{0},
    graph(stream_view_)
{
  RAFT_CUDA_TRY(
    hipMemsetAsync(sum_primal_solutions_.data(), 0.0, sizeof(f_t) * primal_size_h_, stream_view_));
  RAFT_CUDA_TRY(
    hipMemsetAsync(sum_dual_solutions_.data(), 0.0, sizeof(f_t) * dual_size_h_, stream_view_));
}

template <typename i_t, typename f_t>
void weighted_average_solution_t<i_t, f_t>::reset_weighted_average_solution()
{
  RAFT_CUDA_TRY(
    hipMemsetAsync(sum_primal_solutions_.data(), 0.0, sizeof(f_t) * primal_size_h_, stream_view_));
  RAFT_CUDA_TRY(
    hipMemsetAsync(sum_dual_solutions_.data(), 0.0, sizeof(f_t) * dual_size_h_, stream_view_));
  sum_primal_solution_weights_.set_value_to_zero_async(stream_view_);
  sum_dual_solution_weights_.set_value_to_zero_async(stream_view_);
  iterations_since_last_restart_ = 0;
}

template <typename f_t>
__global__ void add_weight_sums(const f_t* primal_weight,
                                const f_t* dual_weight,
                                f_t* sum_primal_solution_weights,
                                f_t* sum_dual_solution_weights)
{
  *sum_primal_solution_weights += *primal_weight;
  *sum_dual_solution_weights += *dual_weight;
}

template <typename i_t, typename f_t>
void weighted_average_solution_t<i_t, f_t>::add_current_solution_to_weighted_average_solution(
  const f_t* primal_solution,
  const f_t* dual_solution,
  const rmm::device_uvector<f_t>& weight,
  i_t total_pdlp_iterations)
{
  // primalavg += primal_sol*weight     -- weight is just set to be step_size for the new solution
  // (same for primal and dual although julia repo makes it seem as though these should/could be
  // different)

  // TODO: handle batch mode

  if (!graph.is_initialized(total_pdlp_iterations)) {
    graph.start_capture(total_pdlp_iterations);

    hipcub::DeviceTransform::Transform(
      cuda::std::make_tuple(sum_primal_solutions_.data(), primal_solution),
      sum_primal_solutions_.data(),
      primal_size_h_,
      a_add_scalar_times_b<f_t>(weight.data()),
      stream_view_);

    hipcub::DeviceTransform::Transform(
      cuda::std::make_tuple(sum_dual_solutions_.data(), dual_solution),
      sum_dual_solutions_.data(),
      dual_size_h_,
      a_add_scalar_times_b<f_t>(weight.data()),
      stream_view_);

    // update weight sums and count (add weight and +1 respectively)
    add_weight_sums<<<1, 1, 0, stream_view_>>>(weight.data(),
                                               weight.data(),
                                               sum_primal_solution_weights_.data(),
                                               sum_dual_solution_weights_.data());

    graph.end_capture(total_pdlp_iterations);
  }
  graph.launch(total_pdlp_iterations);

  iterations_since_last_restart_ += 1;
}

template <typename i_t, typename f_t>
void weighted_average_solution_t<i_t, f_t>::compute_averages(rmm::device_uvector<f_t>& avg_primal,
                                                             rmm::device_uvector<f_t>& avg_dual)
{
  // no iterations have added to the sum, so avg is all zero vector
  if (!iterations_since_last_restart_) {
    RAFT_CUDA_TRY(
      hipMemsetAsync(avg_primal.data(), f_t(0.0), sizeof(f_t) * primal_size_h_, stream_view_));
    RAFT_CUDA_TRY(
      hipMemsetAsync(avg_dual.data(), f_t(0.0), sizeof(f_t) * dual_size_h_, stream_view_));
    return;
  }

  // return weight sums to host to fit API call
  f_t sum_primal_solution_weights_h = sum_primal_solution_weights_.value(stream_view_);
  f_t sum_dual_solution_weights_h   = sum_dual_solution_weights_.value(stream_view_);

  RAFT_CUDA_TRY(hipStreamSynchronize(stream_view_));

  // compute sum_primal_solutions/primal_size
  raft::linalg::divideScalar(avg_primal.data(),
                             sum_primal_solutions_.data(),
                             sum_primal_solution_weights_h,
                             primal_size_h_,
                             stream_view_);
  raft::linalg::divideScalar(avg_dual.data(),
                             sum_dual_solutions_.data(),
                             sum_dual_solution_weights_h,
                             dual_size_h_,
                             stream_view_);
}

template <typename i_t, typename f_t>
i_t weighted_average_solution_t<i_t, f_t>::get_iterations_since_last_restart() const
{
  return iterations_since_last_restart_;
}

#if MIP_INSTANTIATE_FLOAT
template __global__ void add_weight_sums<float>(const float* primal_weight,
                                                const float* dual_weight,
                                                float* sum_primal_solution_weights,
                                                float* sum_dual_solution_weights);

template class weighted_average_solution_t<int, float>;
#endif

#if MIP_INSTANTIATE_DOUBLE
template __global__ void add_weight_sums<double>(const double* primal_weight,
                                                 const double* dual_weight,
                                                 double* sum_primal_solution_weights,
                                                 double* sum_dual_solution_weights);

template class weighted_average_solution_t<int, double>;
#endif

}  // namespace cuopt::linear_programming::detail
